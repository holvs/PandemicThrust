
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "PandemicSim.h"

int main()
{
	PandemicSim sim;

	sim.setupSim();
	sim.runToCompletion();

	return 0;
}
