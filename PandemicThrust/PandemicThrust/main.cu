#include "stdafx.h"

#include "PandemicSim.h"

//#include "indirect.h"
#include "resource_logging.h"

#if CUDA_PROFILER_ENABLE == 1
#include "hip/hip_runtime_api.h"
#endif

int main()
{
	if(CUDA_PROFILER_ENABLE)
		hipProfilerStart();

	logging_pollMemUsage_doSetup(POLL_MEMORY_USAGE, OUTPUT_FILES_IN_PARENTDIR);

	PandemicSim sim;
	sim.setupSim();
	sim.runToCompletion();

	logging_pollMemoryUsage_done();

	if(CUDA_PROFILER_ENABLE)
		hipProfilerStop();

//	hipDeviceReset();

	return 0;
}
