#include "hip/hip_runtime.h"
#include "stdafx.h"

#include "simParameters.h"
#include "profiler.h"

#include "PandemicSim.h"
#include "thrust_functors.h"





#pragma region settings

//Simulation profiling master control - low performance overhead
#define PROFILE_SIMULATION 1

#define CONSOLE_OUTPUT 1

//controls master logging - everything except for profiler
#define GLOBAL_LOGGING 1
#define SANITY_CHECK 1

#define print_infected_info 0
#define log_infected_info GLOBAL_LOGGING

#define print_location_info 0
#define log_location_info 0

#define print_contact_kernel_setup 0
#define log_contact_kernel_setup GLOBAL_LOGGING

#define dump_contact_kernel_random_data 0

#define print_contacts 0
#define log_contacts GLOBAL_LOGGING
#define DOUBLECHECK_CONTACTS 0

#define print_actions 0
#define log_actions GLOBAL_LOGGING

#define print_actions_filtered 0
#define log_actions_filtered GLOBAL_LOGGING

#define log_people_info GLOBAL_LOGGING

//low overhead
#define debug_log_function_calls 0

#pragma endregion settings

int cuda_blocks = 32;
int cuda_threads = 32;



FILE * fDebug;

int SEED_HOST[SEED_LENGTH];
__device__ __constant__ int SEED_DEVICE[SEED_LENGTH];

__device__ __constant__ int business_type_count[NUM_BUSINESS_TYPES];				//stores number of each type of business
__device__ __constant__ int business_type_count_offset[NUM_BUSINESS_TYPES];			//stores location number of first business of this type
__device__ __constant__ float weekday_errand_pdf[NUM_BUSINESS_TYPES];				//stores PDF for weekday errand destinations
__device__ __constant__ float weekend_errand_pdf[NUM_BUSINESS_TYPES];				//stores PDF for weekend errand destinations
__device__ __constant__ float infectiousness_profile[CULMINATION_PERIOD];			//stores viral shedding profiles
__device__ __constant__ int weekend_errand_contact_assignments[6][3];				//stores number of 

//__device__ __constant__ float infectiousness_profiles_all[6][CULMINATION_PERIOD];

#define STRAIN_COUNT 2
__device__ __constant__ float BASE_REPRODUCTION_DEVICE[STRAIN_COUNT];
float BASE_REPRODUCTION_HOST[STRAIN_COUNT];


#define BASE_R_PANDEMIC_DEVICE BASE_REPRODUCTION_DEVICE[0]
#define BASE_R_SEASONAL_DEVICE BASE_REPRODUCTION_DEVICE[1]
#define BASE_R_PANDEMIC_HOST BASE_REPRODUCTION_HOST[0]
#define BASE_R_SEASONAL_HOST BASE_REPRODUCTION_HOST[1]

#define UNSIGNED_MAX (unsigned int) -1

float workplace_type_pdf[NUM_BUSINESS_TYPES];
int h_workplace_type_offset[NUM_BUSINESS_TYPES];
int h_workplace_type_counts[NUM_BUSINESS_TYPES];

float h_weekday_errand_pdf[NUM_BUSINESS_TYPES];
float h_weekend_errand_pdf[NUM_BUSINESS_TYPES];
float h_infectiousness_profile[CULMINATION_PERIOD];
int h_weekend_errand_contact_assignments[6][3];

float h_infectiousness_profile_all[6][CULMINATION_PERIOD];

#define CHILD_DATA_ROWS 5
float child_CDF[CHILD_DATA_ROWS];
int child_wp_types[CHILD_DATA_ROWS];

#define HH_TABLE_ROWS 9
int hh_adult_count[HH_TABLE_ROWS];
int hh_child_count[HH_TABLE_ROWS];
float hh_type_cdf[HH_TABLE_ROWS];


#define FIRST_WEEKDAY_ERRAND_ROW 9
#define FIRST_WEEKEND_ERRAND_ROW 9


PandemicSim::PandemicSim() 
{
	logging_openOutputStreams();

	if(PROFILE_SIMULATION)
		profiler.initStack();

	setup_loadParameters();

	if(debug_log_function_calls)
		debug_print("parameters loaded");

}


PandemicSim::~PandemicSim(void)
{
	logging_closeOutputStreams();
}

void PandemicSim::setupSim()
{
	
	if(PROFILE_SIMULATION)
	{
		profiler.beginFunction(-1,"setupSim");
	}

	//moved to constructor for batching
	//	open_debug_streams();
	//	setupLoadParameters();

	srand(SEED_HOST[0]);					//seed host RNG
	rand_offset = 0;				//set global rand counter to 0

	current_day = -1;

	debug_print("beginning setup");



	if(debug_log_function_calls)
		debug_print("setting up households");
	
	//setup households
	setup_generateHouseholds();	//generates according to PDFs
	setup_sizeGlobalArrays(); // only after households are generated

	if(log_people_info)
		dump_people_info();

	printf("%d people, %d households, %d workplaces\n",number_people, number_households, number_workplaces);

	setup_buildFixedLocations();	//household and workplace
	setup_initialInfected();


	//copy everything down to the GPU
	setup_pushDeviceData();

	if(PROFILE_SIMULATION)
	{
		profiler.endFunction(-1, number_people);
	}

	if(debug_log_function_calls)
		debug_print("simulation setup complete");
}



void PandemicSim::logging_openOutputStreams()
{
	if(log_infected_info)
	{
		fInfected = fopen("../debug_infected.csv", "w");
		fprintf(fInfected, "current_day, i, idx, status_p, day_p, gen_p, status_s, day_s, gen_s\n");
	}

	if(log_location_info)
	{
		fLocationInfo = fopen("../debug_location_info.csv","w");
		fprintf(fLocationInfo, "current_day, hour_index, i, offset, count, max_contacts\n");
	}

	if(log_contacts)
	{
		fContacts = fopen("../debug_contacts.csv", "w");
		fprintf(fContacts, "current_day, i, contact_type, infector_idx, victim_idx, infector_loc, victim_loc, infector_found, victim_found\n");
	}

	if(log_contact_kernel_setup)
	{
		fContactsKernelSetup = fopen("../debug_contacts_kernel_setup.csv", "w");
		fprintf(fContactsKernelSetup, "current_day,hour,i,infector_idx,loc,loc_offset,loc_count,contacts_desired,output_offset\n");
	}


	if(log_actions)
	{
		fActions = fopen("../debug_actions.csv", "w");
		fprintf(fActions, "current_day, i, type, infector, infector_status_p, infector_status_s, victim, action_gen_p, action_gen_s, y_p, thresh_p, infects_p, y_s, thresh_s, infects_s\n");
	}

	if(log_actions_filtered)
	{
		fActionsFiltered = fopen("../debug_filtered_actions.csv", "w");
		fprintf(fActionsFiltered, "current_day, i, type, victim, victim_status_p, victim_gen_p, victim_status_s, victim_gen_s\n");
	}


	fDebug = fopen("../debug.txt", "w");

	
}


void PandemicSim::setup_loadParameters()
{

	//load 4 seeds from file
	FILE * fSeed = fopen("seed.txt","r");
	if(fSeed == NULL)
	{
		debug_print("failed to open seed file");
		perror("Error opening seed file");
		exit(1);
	}
	for(int i = 0; i < SEED_LENGTH; i++)
	{
		fscanf(fSeed, "%d", &(SEED_HOST[i]));
	}
	fclose(fSeed);

	//if printing seeds is desired for debug, etc
	if(0)
	{
		printf("seeds:\t");
		for(int i = 0; i < SEED_LENGTH; i++)
			if(i < SEED_LENGTH - 1)
				printf("%d\t",SEED_HOST[i]);
			else
				printf("%d\n",SEED_HOST[i]);
	}

	//read constants file 
	FILE * fConstants = fopen("constants.csv","r");	//open file
	if(fSeed == NULL)
	{
		debug_print("failed to open constants file");
		perror("Error opening constants file");
		exit(1);
	}

	//get a line buffer
#define LINEBUFF_SIZE 512
	char line[LINEBUFF_SIZE];	

	fgets(line, LINEBUFF_SIZE, fConstants);	//read the first line into the buffer to skip it
	fscanf(fConstants,"%*[^,]%*c");	//skip the first column of the table
	fscanf(fConstants, "%d%*c", &MAX_DAYS);
	fscanf(fConstants, "%f%*c", &BASE_R_PANDEMIC_HOST);
	fscanf(fConstants, "%f%*c", &BASE_R_SEASONAL_HOST);
	fscanf(fConstants, "%d%*c", &INITIAL_INFECTED_PANDEMIC);
	fscanf(fConstants, "%d%*c", &INITIAL_INFECTED_SEASONAL);
	fscanf(fConstants, "%f%*c", &sim_scaling_factor);
	fscanf(fConstants, "%f", &asymp_factor);
	fclose(fConstants);

	number_households = 100000;
	number_workplaces = 1300;

	printf("max days: %d\nr_p: %f\nr_s: %f\ninitial_pandemic: %d\ninitial_seasonal: %d\nnumber_households: %d\n",
		MAX_DAYS,
		BASE_R_PANDEMIC_HOST,
		BASE_R_SEASONAL_HOST,
		INITIAL_INFECTED_PANDEMIC,
		INITIAL_INFECTED_SEASONAL,
		number_households);

	//read other parameter sets
	//hard coded these for time currently since we have no other sets

	//cdf for child age
	child_CDF[0] = 0.24f;
	child_CDF[1] = 0.47f;
	child_CDF[2] = 0.72f;
	child_CDF[3] = 0.85f;
	child_CDF[4] = 1.0f;

	//what workplace type children get for this age
	child_wp_types[0] = 3;
	child_wp_types[1] = 4;
	child_wp_types[2] = 5;
	child_wp_types[3] = 6;
	child_wp_types[4] = 7;

	//workplace PDF for adults
	workplace_type_pdf[0] = 0.06586f;
	workplace_type_pdf[1] = 0.05802f;
	workplace_type_pdf[2] = 0.30227f;
	workplace_type_pdf[3] = 0.0048f;
	workplace_type_pdf[4] = 0.00997f;
	workplace_type_pdf[5] = 0.203f;
	workplace_type_pdf[6] = 0.09736f;
	workplace_type_pdf[7] = 0.10598f;
	workplace_type_pdf[8] = 0.00681f;
	workplace_type_pdf[9] = 0.02599f;
	workplace_type_pdf[10] = 0.f;
	workplace_type_pdf[11] = 0.08749f;
	workplace_type_pdf[12] = 0.03181f;
	workplace_type_pdf[13] = 0.00064f;

	//number of each type of workplace
	h_workplace_type_counts[0] = 100;
	h_workplace_type_counts[1] = 700;
	h_workplace_type_counts[2] = 240;
	h_workplace_type_counts[3] = 30;
	h_workplace_type_counts[4] = 10;
	h_workplace_type_counts[5] = 20;
	h_workplace_type_counts[6] = 10;
	h_workplace_type_counts[7] = 10;
	h_workplace_type_counts[8] = 30;
	h_workplace_type_counts[9] = 50;
	h_workplace_type_counts[10] = 0;
	h_workplace_type_counts[11] = 30;
	h_workplace_type_counts[12] = 40;
	h_workplace_type_counts[13] = 10;

	//calculate the offset of each workplace type
	thrust::exclusive_scan(
		h_workplace_type_counts,
		h_workplace_type_counts + NUM_BUSINESS_TYPES,
		h_workplace_type_offset);			


	//pdf for weekday errand location generation
	//most entries are 0.0
	thrust::fill(h_weekday_errand_pdf, h_weekday_errand_pdf + NUM_BUSINESS_TYPES, 0.0);
	h_weekday_errand_pdf[9] = 0.61919f;
	h_weekday_errand_pdf[11] = 0.27812f;
	h_weekday_errand_pdf[12] = 0.06601f;
	h_weekday_errand_pdf[13] = 0.03668f;

	//pdf for weekend errand location generation
	//most entries are 0.0
	thrust::fill(h_weekend_errand_pdf, h_weekend_errand_pdf + NUM_BUSINESS_TYPES, 0.0f);
	h_weekend_errand_pdf[9] = 0.51493f;
	h_weekend_errand_pdf[11] = 0.25586f;
	h_weekend_errand_pdf[12] = 0.1162f;
	h_weekend_errand_pdf[13] = 0.113f;


	//how many adults in each household type
	hh_adult_count[0] = 1;
	hh_adult_count[1] = 1;
	hh_adult_count[2] = 2;
	hh_adult_count[3] = 1;
	hh_adult_count[4] = 2;
	hh_adult_count[5] = 1;
	hh_adult_count[6] = 2;
	hh_adult_count[7] = 1;
	hh_adult_count[8] = 2;

	//how many children in each household type
	hh_child_count[0] = 0;
	hh_child_count[1] = 1;
	hh_child_count[2] = 0;
	hh_child_count[3] = 2;
	hh_child_count[4] = 1;
	hh_child_count[5] = 3;
	hh_child_count[6] = 2;
	hh_child_count[7] = 4;
	hh_child_count[8] = 3;

	//the PDF of each household type
	hh_type_cdf[0] = 0.279f;
	hh_type_cdf[1] = 0.319f;
	hh_type_cdf[2] = 0.628f;
	hh_type_cdf[3] = 0.671f;
	hh_type_cdf[4] = 0.8f;
	hh_type_cdf[5] = 0.812f;
	hh_type_cdf[6] = 0.939f;
	hh_type_cdf[7] = 0.944f;
	hh_type_cdf[8] = 1.0f;

	//store all permutations of contact assignments
	h_weekend_errand_contact_assignments[0][0] = 2;
	h_weekend_errand_contact_assignments[0][1] = 0;
	h_weekend_errand_contact_assignments[0][2] = 0;

	h_weekend_errand_contact_assignments[1][0] = 0;
	h_weekend_errand_contact_assignments[1][1] = 2;
	h_weekend_errand_contact_assignments[1][2] = 0;

	h_weekend_errand_contact_assignments[2][0] = 0;
	h_weekend_errand_contact_assignments[2][1] = 0;
	h_weekend_errand_contact_assignments[2][2] = 2;

	h_weekend_errand_contact_assignments[3][0] = 1;
	h_weekend_errand_contact_assignments[3][1] = 1;
	h_weekend_errand_contact_assignments[3][2] = 0;

	h_weekend_errand_contact_assignments[4][0] = 1;
	h_weekend_errand_contact_assignments[4][1] = 0;
	h_weekend_errand_contact_assignments[4][2] = 1;

	h_weekend_errand_contact_assignments[5][0] = 0;
	h_weekend_errand_contact_assignments[5][1] = 1;
	h_weekend_errand_contact_assignments[5][2] = 1;


	//load lognorm1 as default profile - others will be used later
	h_infectiousness_profile[0] = 0.002533572f;
	h_infectiousness_profile[1] = 0.348252834f;
	h_infectiousness_profile[2] = 0.498210218f;
	h_infectiousness_profile[3] = 0.130145145f;
	h_infectiousness_profile[4] = 0.018421298f;
	h_infectiousness_profile[5] = 0.002158374f;
	h_infectiousness_profile[6] = 0.000245489f;
	h_infectiousness_profile[7] = 2.88922E-05f;
	h_infectiousness_profile[8] = 3.61113E-06f;
	h_infectiousness_profile[9] = 4.83901E-07f;

#pragma region profiles

	//gamma1
	h_infectiousness_profile_all[0][0] = 0.007339835f;
	h_infectiousness_profile_all[0][1] = 0.332600216f;
	h_infectiousness_profile_all[0][2] = 0.501192066f;
	h_infectiousness_profile_all[0][3] = 0.142183447f;
	h_infectiousness_profile_all[0][4] = 0.015675154f;
	h_infectiousness_profile_all[0][5] = 0.000967407f;
	h_infectiousness_profile_all[0][6] = 4.055E-05f;
	h_infectiousness_profile_all[0][7] = 1.29105E-06f;
	h_infectiousness_profile_all[0][8] = 3.34836E-08f;
	h_infectiousness_profile_all[0][9] = 7.41011E-10f;

	//lognorm1
	h_infectiousness_profile_all[1][0] = 0.002533572f;
	h_infectiousness_profile_all[1][1] = 0.348252834f;
	h_infectiousness_profile_all[1][2] = 0.498210218f;
	h_infectiousness_profile_all[1][3] = 0.130145145f;
	h_infectiousness_profile_all[1][4] = 0.018421298f;
	h_infectiousness_profile_all[1][5] = 0.002158374f;
	h_infectiousness_profile_all[1][6] = 0.000245489f;
	h_infectiousness_profile_all[1][7] = 2.88922E-05f;
	h_infectiousness_profile_all[1][8] = 3.61113E-06f;
	h_infectiousness_profile_all[1][9] = 4.83901E-07f;


	//weib1
	h_infectiousness_profile_all[2][0] = 0.05927385f;
	h_infectiousness_profile_all[2][1] = 0.314171688f;
	h_infectiousness_profile_all[2][2] = 0.411588802f;
	h_infectiousness_profile_all[2][3] = 0.187010054f;
	h_infectiousness_profile_all[2][4] = 0.026934715f;
	h_infectiousness_profile_all[2][5] = 0.001013098f;
	h_infectiousness_profile_all[2][6] = 7.78449E-06f;
	h_infectiousness_profile_all[2][7] = 9.29441E-09f;
	h_infectiousness_profile_all[2][8] = 1.29796E-12f;
	h_infectiousness_profile_all[2][9] = 0;

	//gamma2
	h_infectiousness_profile_all[3][0] = 0.04687299f;
	h_infectiousness_profile_all[3][1] = 0.248505983f;
	h_infectiousness_profile_all[3][2] = 0.30307952f;
	h_infectiousness_profile_all[3][3] = 0.211008627f;
	h_infectiousness_profile_all[3][4] = 0.11087006f;
	h_infectiousness_profile_all[3][5] = 0.049241932f;
	h_infectiousness_profile_all[3][6] = 0.019562658f;
	h_infectiousness_profile_all[3][7] = 0.007179076f;
	h_infectiousness_profile_all[3][8] = 0.002482875f;
	h_infectiousness_profile_all[3][9] = 0.000820094f;

	//lognorm2
	h_infectiousness_profile_all[4][0] = 0.028667712f;
	h_infectiousness_profile_all[4][1] = 0.283445338f;
	h_infectiousness_profile_all[4][2] = 0.319240133f;
	h_infectiousness_profile_all[4][3] = 0.190123057f;
	h_infectiousness_profile_all[4][4] = 0.093989959f;
	h_infectiousness_profile_all[4][5] = 0.044155659f;
	h_infectiousness_profile_all[4][6] = 0.020682822f;
	h_infectiousness_profile_all[4][7] = 0.009841839f;
	h_infectiousness_profile_all[4][8] = 0.00479234f;
	h_infectiousness_profile_all[4][9] = 0.002393665f;

	//weib2
	h_infectiousness_profile_all[5][0] = 0.087866042f;
	h_infectiousness_profile_all[5][1] = 0.223005225f;
	h_infectiousness_profile_all[5][2] = 0.258992749f;
	h_infectiousness_profile_all[5][3] = 0.208637267f;
	h_infectiousness_profile_all[5][4] = 0.127489076f;
	h_infectiousness_profile_all[5][5] = 0.061148649f;
	h_infectiousness_profile_all[5][6] = 0.023406737f;
	h_infectiousness_profile_all[5][7] = 0.007216643f;
	h_infectiousness_profile_all[5][8] = 0.001802145f;
	h_infectiousness_profile_all[5][9] = 0.00036581f;

	for(int i = 3; i < 6; i++)
		for(int j = 0; i < CULMINATION_PERIOD; i++)
			h_infectiousness_profile_all[i][j] *= asymp_factor;

#pragma endregion profiles
}

//push various things to device constant memory
void PandemicSim::setup_pushDeviceData()
{
	//workplace location data
	hipMemcpyToSymbol(HIP_SYMBOL(
		business_type_count),
		h_workplace_type_counts,
		sizeof(int) * NUM_BUSINESS_TYPES);
	hipMemcpyToSymbol(HIP_SYMBOL(
		business_type_count_offset),
		h_workplace_type_offset,
		sizeof(int) * NUM_BUSINESS_TYPES);

	//weekday+weekend errand PDFs
	hipMemcpyToSymbol(HIP_SYMBOL(
		weekday_errand_pdf),
		h_weekday_errand_pdf,
		sizeof(float) * NUM_BUSINESS_TYPES);
	hipMemcpyToSymbol(HIP_SYMBOL(
		weekend_errand_pdf),
		h_weekend_errand_pdf,
		sizeof(float) * NUM_BUSINESS_TYPES);

	//viral shedding profile
	hipMemcpyToSymbol(HIP_SYMBOL(
		infectiousness_profile),
		h_infectiousness_profile,
		sizeof(float) * CULMINATION_PERIOD);

	//reproduction numbers
	hipMemcpyToSymbol(HIP_SYMBOL(
		BASE_REPRODUCTION_DEVICE),
		BASE_REPRODUCTION_HOST,
		sizeof(float) * STRAIN_COUNT);

	//weekend errand contacts_desired assignments
	hipMemcpyToSymbol(HIP_SYMBOL(
		weekend_errand_contact_assignments),
		h_weekend_errand_contact_assignments,
		sizeof(int) * 6 * 3);

	//seeds
	hipMemcpyToSymbol(HIP_SYMBOL(
		SEED_DEVICE),
		SEED_HOST,
		sizeof(int) * SEED_LENGTH);

	hipDeviceSynchronize();
}

//Sets up people's households and workplaces according to the probability functions
void PandemicSim::setup_generateHouseholds()
{
	//actual expected value: 2.5
	int expected_people = 3 * number_households;

	//stores household and workplace data for all people
	thrust::host_vector<int> h_people_hh;
	thrust::host_vector<int> h_people_wp;
	h_people_hh.reserve(expected_people);
	h_people_wp.reserve(expected_people);

	//stores the list of adults and children for weekday errands/afterschool
	thrust::host_vector<int> h_adult_indexes;
	thrust::host_vector<int> h_child_indexes;
	h_adult_indexes.reserve(expected_people);
	h_child_indexes.reserve(expected_people);

	//count number of people
	number_people = 0;

	for(int hh = 0; hh < number_households; hh++)
	{
		//fish out the type of household from CDF
		float y = (float) rand() / RAND_MAX;
		int hh_type = 0;
		while(y > hh_type_cdf[hh_type] && hh_type < HH_TABLE_ROWS - 1)
			hh_type++;

		//generate the adults for this household
		for(int i = 0; i < hh_adult_count[hh_type]; i++)
		{
			//assign adult workplace
			int wp = setup_assignWorkplace();
			h_people_wp.push_back(wp);

			//assign household
			h_people_hh.push_back(hh);

			//store as adult
			h_adult_indexes.push_back(number_people);

			number_people++;
		}

		//generate the children for this household
		for(int i = 0; i < hh_child_count[hh_type]; i++)
		{
			//assign school
			int wp = setup_assignSchool();
			h_people_wp.push_back(wp);

			//assign household
			h_people_hh.push_back(hh);

			//store as child
			h_child_indexes.push_back(number_people);

			number_people++;
		}
	}

	//trim arrays to data size, and transfer them to GPU
	h_people_wp.shrink_to_fit();
	people_workplaces = h_people_wp;

	h_people_hh.shrink_to_fit();
	people_households = h_people_hh;

	h_adult_indexes.shrink_to_fit();
	people_adult_indexes = h_adult_indexes;

	h_child_indexes.shrink_to_fit();
	people_child_indexes = h_child_indexes;

	number_adults = h_adult_indexes.size();
	number_children = h_child_indexes.size();

	printf("%d households, %d adults, %d children, %d total\n",
		number_households, number_adults, number_children, number_people);

	//setting up status array will be handled in setupSim()
}

int PandemicSim::setup_assignWorkplace()
{
	//fish out workplace type
	float y = (float) rand() / RAND_MAX;
	int row = 0;
	while(workplace_type_pdf[row] < y && row < NUM_BUSINESS_TYPES - 1)
	{
		y -= workplace_type_pdf[row];
		row++;
	}

	//of this workplace type, which number is this?
	float frac = y / workplace_type_pdf[row];
	int ret = frac * h_workplace_type_counts[row];  //truncate to int

	//how many other workplaces have we gone past?
	int offset = h_workplace_type_offset[row];

	//	printf("row: %d\ty: %f\tpdf[row]: %f\tfrac: %f\tret: %4d\toffset:%d\n",
	//			row, y, workplace_type_pdf[row], frac, ret, offset);


	return ret + offset;
}

int PandemicSim::setup_assignSchool()
{
	//fish out age group and resulting school type from CDF
	int row = 0;
	float y = (float) rand() / RAND_MAX;
	while(row < CHILD_DATA_ROWS - 1 && y > child_CDF[row])
		row++;


	int wp_type = child_wp_types[row];

	//of this school type, which one will this kid be assigned to?
	float frac;
	if(row == 0)
		frac = y / (child_CDF[row]);
	else
	{
		float pdf_here = child_CDF[row] - child_CDF[row - 1];
		float y_here = y - child_CDF[row - 1];
		//	printf("y here: %f\tpdf here: %f\n", (y - child_CDF[row - 1]), pdf);
		frac =  y_here / pdf_here;
	}

	int ret = frac * h_workplace_type_counts[wp_type];

	//how many other workplaces have we gone past?
	int offset = h_workplace_type_offset[wp_type];
	return ret + offset;
}


//Sets up the initial infection at the beginning of the simulation
//BEWARE: you must not generate dual infections with this code, all initial infected have one type to start
void PandemicSim::setup_initialInfected()
{
	//fill infected array with null info (not infected)
	thrust::fill(infected_days_pandemic.begin(), infected_days_pandemic.end(), DAY_NOT_INFECTED);
	thrust::fill(infected_days_seasonal.begin(), infected_days_seasonal.end(), DAY_NOT_INFECTED);
	thrust::fill(infected_generation_pandemic.begin(), infected_generation_pandemic.end(), GENERATION_NOT_INFECTED);
	thrust::fill(infected_generation_seasonal.begin(), infected_generation_seasonal.end(), GENERATION_NOT_INFECTED);

	int initial_infected = INITIAL_INFECTED_PANDEMIC + INITIAL_INFECTED_SEASONAL;

	//get N unique indexes - they should not be sorted
	h_vec h_init_indexes(initial_infected);
	n_unique_numbers(&h_init_indexes, initial_infected, number_people);
	thrust::copy(h_init_indexes.begin(), h_init_indexes.end(), infected_indexes.begin());

	///// INFECTED PANDEMIC:
	//infect first INITIAL_INFECTED_PANDEMIC people with pandemic
	//set status to infected
	thrust::fill(
		thrust::make_permutation_iterator(people_status_pandemic.begin(), infected_indexes.begin()),	//begin at infected 0
		thrust::make_permutation_iterator(people_status_pandemic.begin(), infected_indexes.begin() + INITIAL_INFECTED_PANDEMIC),	//end at index INITIAL_INFECTED_PANDEMIC
		STATUS_INFECTED);

	//set day/generation pandemic to 0 (initial)
	thrust::fill(
		infected_days_pandemic.begin(), 		//begin
		infected_days_pandemic.begin() + INITIAL_INFECTED_PANDEMIC, //end
		INITIAL_DAY);//val
	thrust::fill(
		infected_generation_pandemic.begin(),
		infected_generation_pandemic.begin() + INITIAL_INFECTED_PANDEMIC,
		0);	//fill infected with gen 0

	///// INFECTED SEASONAL:
	//set status to infected
	thrust::fill(
		thrust::make_permutation_iterator(people_status_seasonal.begin(), infected_indexes.begin()+ INITIAL_INFECTED_PANDEMIC), //begin at index INITIAL_INFECTED_PANDEMIC
		thrust::make_permutation_iterator(people_status_seasonal.begin(), infected_indexes.begin() + INITIAL_INFECTED_PANDEMIC + INITIAL_INFECTED_SEASONAL),	//end INITIAL_INFECTED_PANDEMIC + INITIAL_INFECTED_SEASONAL
		STATUS_INFECTED);

	//set day/generation seasonal to 0
	thrust::fill(
		infected_generation_seasonal.begin() + INITIAL_INFECTED_PANDEMIC,
		infected_generation_seasonal.begin() + INITIAL_INFECTED_PANDEMIC + INITIAL_INFECTED_SEASONAL,
		0);	//first generation
	thrust::fill(
		infected_days_seasonal.begin() + INITIAL_INFECTED_PANDEMIC,
		infected_days_seasonal.begin() + INITIAL_INFECTED_PANDEMIC + INITIAL_INFECTED_SEASONAL,
		INITIAL_DAY);		//day: 0

	//sort array after infection complete
	thrust::sort(
		thrust::make_zip_iterator(thrust::make_tuple(			//first
		infected_indexes.begin(),
		infected_days_pandemic.begin(),infected_days_seasonal.begin(),
		infected_generation_pandemic.begin(),infected_generation_seasonal.begin())),
		thrust::make_zip_iterator(thrust::make_tuple(			//first
		infected_indexes.begin() + initial_infected,
		infected_days_pandemic.begin() + initial_infected,infected_days_seasonal.begin() + initial_infected,
		infected_generation_pandemic.begin() + initial_infected,infected_generation_seasonal.begin() + initial_infected)),
		FiveTuple_SortByFirst_Struct());

	infected_count = initial_infected;
}

//sets up the locations which are the same every day and do not change
//i.e. workplace and household
void PandemicSim::setup_buildFixedLocations()
{
	///////////////////////////////////////
	//work/////////////////////////////////
	workplace_offsets.resize(number_workplaces + 1);	//size arrays
	workplace_people.resize(number_people);

	thrust::sequence(workplace_people.begin(), workplace_people.begin() + number_people);	//fill array with IDs to sort

	calcLocationOffsets(
		&workplace_people,
		people_workplaces,
		&workplace_offsets,
		number_people, number_workplaces);

	//TODO:  max contacts are currently 3 for all workplaces
	workplace_max_contacts.resize(number_workplaces);
	thrust::fill(workplace_max_contacts.begin(), workplace_max_contacts.begin() + number_workplaces, 3);	//fill max contacts


	///////////////////////////////////////
	//home/////////////////////////////////
	household_offsets.resize(number_households + 1);
	household_people.resize(number_people);

	thrust::sequence(household_people.begin(), household_people.begin() + number_people);	//fill array with IDs to sort
	calcLocationOffsets(
		&household_people,
		people_households,
		&household_offsets,
		number_people, number_households);

	household_max_contacts.resize(number_households);
	thrust::fill(household_max_contacts.begin(), household_max_contacts.begin() + number_households, 2);

}


//given an array of people's ID numbers and locations
//sort them by location, and then build the location offset/count tables
//ids_to_sort will be sorted by workplace
void PandemicSim::calcLocationOffsets(
	vec_t * ids_to_sort,
	vec_t lookup_table_copy,
	vec_t * location_offsets,
	int num_people, int num_locs)
{
	//sort people by workplace
	thrust::sort_by_key(
		lookup_table_copy.begin(),
		lookup_table_copy.end(),
		(*ids_to_sort).begin());

	//build count/offset table
	thrust::counting_iterator<int> count_iterator(0);
	thrust::lower_bound(		//find lower bound of each location
		lookup_table_copy.begin(),
		lookup_table_copy.end(),
		count_iterator,
		count_iterator + num_locs,
		(*location_offsets).begin());

	//originally, we calculated the count by using an upper bound and then subtracting the lower bound
	//instead, we can calculate the count by the following formula:
	//loc_count = loc_offset[i+1] - loc_offset[i]
	//i.e. people = {1, 1, 2, 2, 3}
	//location_numbers = {1, 2, 3}
	//loc_offsets = {0, 2, 4}
	//We need to add one extra offset so the last location doesn't go out of bounds - this is equal to
	//the number of people in the array
	//so loc_offsets = {0, 2, 4, 5}

	if(SANITY_CHECK)
	{
		debug_assert("Loc_offset not sized properly",num_locs + 1,location_offsets->size());
	}

	(*location_offsets)[num_locs] = num_people;
}

void PandemicSim::dump_people_info()
{
	h_vec h_wp = people_workplaces;
	h_vec h_hh = people_households;

	FILE * fPeopleInfo = fopen("../debug_people_info.csv.gz", "w");
	fprintf(fPeopleInfo, "i,workplace,household\n");
	for(int i = 0; i < number_people; i++)
	{
		fprintf(fPeopleInfo, "%d,%d,%d\n", i, h_wp[i], h_hh[i]);
	}
	fclose(fPeopleInfo);
}

void PandemicSim::logging_closeOutputStreams()
{
	if(log_infected_info)
	{
		fclose(fInfected);
	}

	if(log_location_info)
	{
		fclose(fLocationInfo);
	}

	if(log_contacts)
	{
		fclose(fContacts);
	}

	if(log_actions)
	{
		fclose(fActions);
	}

	if(log_actions_filtered)
	{
		fclose(fActionsFiltered);
	}

	fclose(fDebug);
	profiler.done();
} 



void PandemicSim::runToCompletion()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(-1, "runToCompletion");

	for(current_day = 0; current_day < MAX_DAYS; current_day++)
	{
		if(debug_log_function_calls)
		{
			fprintf(fDebug, "\n\n---------------------\nday %d\ninfected: %d\n---------------------\n\n", current_day, infected_count);
			fflush(fDebug);
		}

		if(CONSOLE_OUTPUT)
		{
			printf("Day %d:\tinfected: %5d\n", current_day + 1, infected_count);
		}
		daily_contacts = 0;	//start counting contacts/actions from 0 each day
		daily_actions = 0;

		/*
		//resize contacts array to fit expected number of contacts
		int contacts_expected;
		if(is_weekend())
			contacts_expected = 5; //3 home, 2 errand
		else
			contacts_expected = 8; //3 home, 2 work, 3 afterschool

		contacts_expected *= infected_count;
		daily_contact_infectors.resize(contacts_expected);
		daily_contact_victims.resize(contacts_expected);
		//daily_contact_kvals.resize(contacts_expected);*/
		

		//debug: dump infected info?
		if(log_infected_info || print_infected_info)
		{
//			debug_validate_infected();
			dump_infected_info();
		}

		//MAKE CONTACTS DEPENDING ON TYPE OF DAY
		if(is_weekend())
		{
			doWeekend();
		}
		else
		{
			doWeekday();
		}

		//PROCESS CONTACTS AND UPDATE INFECTED
		dailyUpdate();

		//if we're using the profiler, flush each day in case of crash
		if(PROFILE_SIMULATION)
		{
			profiler.dailyFlush();
		}
	}
	calculateFinalReproduction();

	if(PROFILE_SIMULATION)
		profiler.endFunction(-1, number_people);


	//moved to destructor for batching
	//close_output_streams();
}


//called at the end of the simulation, figures out the reproduction numbers for each generation
void PandemicSim::calculateFinalReproduction()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "calculateFinalReproduction");

	//copy to host
	thrust::host_vector<int> r_pandemic = generation_pandemic;
	thrust::host_vector<int> r_seasonal = generation_seasonal;

	FILE * out = fopen("../output_rn.csv", "w");
	fprintf(out, "gen, size_p, rn_p, size_s, rn_s\n");

	//loop and calculate reproduction
	int gen_size_p = INITIAL_INFECTED_PANDEMIC;
	int gen_size_s = INITIAL_INFECTED_SEASONAL;
	for(int i = 0; i < MAX_DAYS; i++)
	{
		float rn_pandemic = (float) r_pandemic[i] / gen_size_p;
		float rn_seasonal = (float) r_seasonal[i] / gen_size_s;

		fprintf(out, "%d, %d, %f, %d, %f\n",
			i, gen_size_p, rn_pandemic, gen_size_s, rn_seasonal);

		gen_size_p = r_pandemic[i];
		gen_size_s = r_seasonal[i];
	}
	fclose(out);

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, MAX_DAYS);
} 

void PandemicSim::debug_validate_infected()
{
	//ASSERT:  ALL INFECTED ARRAYS ARE THE CORRECT/SAME SIZE
	debug_assert("infected_indexes.size() != infected_count", infected_indexes.size(), infected_count);
	debug_assert("infected_days_pandemic.size() != infected_count", infected_days_pandemic.size(), infected_count);
	debug_assert("infected_days_seasonal.size() != infected_count", infected_days_seasonal.size(), infected_count);
	debug_assert("infected_generation_pandemic.size() != infected_count", infected_generation_pandemic.size(), infected_count);
	debug_assert("infected_generation_seasonal.size() != infected_count", infected_generation_seasonal.size(), infected_count);

	//ASSERT:  INFECTED INDEXES ARE SORTED
	bool sorted = thrust::is_sorted(infected_indexes.begin(), infected_indexes.begin() + infected_count);
	debug_assert(sorted, "infected indexes are not sorted");

	//ASSERT:  INFECTED INDEXES ARE UNIQUE
	d_vec unique_indexes(infected_count);
	IntIterator end = thrust::unique_copy(infected_indexes.begin(), infected_indexes.begin() + infected_count, unique_indexes.begin());
	int unique_count  = end - unique_indexes.begin();
	debug_assert("infected_indexes are not unique", infected_count, unique_count);


	//copy infected data to PC
	h_vec h_ii = infected_indexes;
	h_vec h_day_p = infected_days_pandemic;
	h_vec h_day_s = infected_days_seasonal;
	h_vec h_gen_p = infected_generation_pandemic;
	h_vec h_gen_s = infected_generation_seasonal;

	h_vec h_p_status_p = people_status_pandemic;
	h_vec h_p_status_s = people_status_seasonal;

	//begin intensive check of infected
	for(int i = 0; i < infected_count; i++)
	{
		int idx = h_ii[i];
		int day_p = h_day_p[i];
		int gen_p = h_gen_p[i];
		int day_s = h_day_s[i];
		int gen_s = h_gen_s[i];

		int status_p = h_p_status_p[idx];
		int status_s = h_p_status_s[idx];

		//ASSERT: person on infected list is infected with pandemic or seasonal
		bool has_infection = status_p || status_s;
		debug_assert(has_infection, "infected_index has no infection", idx);

		if(status_p == STATUS_INFECTED)
		{
			//check that day of pandemic infection is within bounds
			debug_assert(day_p > DAY_NOT_INFECTED, "status_p infected but day not set", idx);
			debug_assert(day_p <= current_day, "day_p is after today", idx);
			int day_of_infection = current_day - day_p;
			debug_assert(day_of_infection < CULMINATION_PERIOD, "pandemic infection should have been recovered", idx);

			//check that generation is within bounds
			debug_assert(gen_p > GENERATION_NOT_INFECTED, "status_p infected but generation not set", idx);
			debug_assert(gen_p <= current_day, "generation_p too high", idx);
		}
		else
		{
			//NOT INFECTED - these should not be set to valid data!
			debug_assert(day_p == DAY_NOT_INFECTED, "status_p not infected but day is set", idx);
			debug_assert(gen_p == GENERATION_NOT_INFECTED, "status_p not infected but gen is set", idx);
		}

		if(status_s == STATUS_INFECTED)
		{
			//check that day of seasonal infection is within bounds
			debug_assert(day_s > DAY_NOT_INFECTED, "status_s infected but day not set", idx);
			debug_assert(day_s <= current_day, "day_s is after today", idx);
			int day_of_infection = current_day - day_s;
			debug_assert(day_of_infection < CULMINATION_PERIOD, "seasonal infection should have been recovered", idx);

			//check that generation is within bounds
			debug_assert(gen_s > GENERATION_NOT_INFECTED, "status_s infected but generation not set", idx);
			debug_assert(gen_s <= current_day, "generation_s too high", idx);
		}
		else
		{
			//NOT INFECTED - these should not be set to valid data!
			debug_assert(day_s == DAY_NOT_INFECTED, "status_s not infected but day is set", idx);
			debug_assert(gen_s == GENERATION_NOT_INFECTED, "status_s not infected but gen is set", idx);
		}
	}

	fflush(fDebug);
}

//dumps all infected info to disk
//this is a big consumer of disk space, so it uses zlib to compress
//use zcat to dump the file
void PandemicSim::dump_infected_info()
{
	//copy to host PC
	h_vec h_ii(infected_count);
	thrust::copy_n(infected_indexes.begin(), infected_count, h_ii.begin());
	h_vec h_day_p(infected_count);
	thrust::copy_n(infected_days_pandemic.begin(), infected_count, h_day_p.begin());
	h_vec h_day_s(infected_count);
	thrust::copy_n(infected_days_seasonal.begin(), infected_count, h_day_s.begin());
	h_vec h_gen_p(infected_count);
	thrust::copy_n(infected_generation_pandemic.begin(), infected_count, infected_generation_seasonal.begin());
	h_vec h_gen_s(infected_count);
	thrust::copy_n(infected_generation_seasonal.begin(), infected_count, infected_generation_seasonal.begin());

	h_vec h_p_status_p = people_status_pandemic;
	h_vec h_p_status_s = people_status_seasonal;

	//iterate and dump data
	for(int i = 0; i < infected_count; i++)
	{
		int idx = h_ii[i];
		int status_p = h_p_status_p[idx];
		int status_s = h_p_status_s[idx];
		int d_p = h_day_p[i];
		int g_p = h_gen_p[i];
		int d_s = h_day_s[i];
		int g_s = h_gen_s[i];

		//day, i, idx, status_p, day_p, gen_p, status_s, day_s, gen_s
		fprintf(fInfected, "%d, %d, %d, %c, %d, %d, %c, %d, %d\n",
			current_day, i, idx, 
			status_int_to_char(status_p), d_p, g_p,
			status_int_to_char(status_s), d_s, g_s);
	}
//	gzflush(fInfected, Z_SYNC_FLUSH);
	fflush(fInfected);
	fflush(fDebug);
}

//generate one weekday's worth of contacts
void PandemicSim::doWeekday()
{
	//tests household and workplaces for each infected person
	if(0)
		test_locs();

	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "doWeekday");

	//make workplace contacts
	makeContacts_byLocationMax(
		"workplace",
		&infected_indexes, infected_count,
		&workplace_people, &workplace_max_contacts,
		&workplace_offsets, number_workplaces, &people_workplaces);

	if(debug_log_function_calls)
		debug_print("workplace contacts complete"); 

	//do afterschool for children, and errands for adults
	doWeekdayErrands();

	if(debug_log_function_calls)
		debug_print("errand contacts complete");

	//make household contacts
	makeContacts_byLocationMax(
		"household",
		&infected_indexes, infected_count,
		&household_people, &household_max_contacts,
		&household_offsets, number_households,
		&people_households);

	if(debug_log_function_calls)
		debug_print("household contacts complete");

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, infected_count);
}

//Makes one day of contacts according to weekend schedule
void PandemicSim::doWeekend()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "doWeekend");

	//everyone makes household contacts
	makeContacts_byLocationMax(
		"household",
		&infected_indexes, infected_count,
		&household_people, &household_max_contacts,
		&household_offsets, number_households,
		&people_households); //hh

	//each person will make errand contacts on 3 of 10 possible errand hours
	doWeekendErrands();

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, infected_count);
}

//generates contacts for the 6 errand hours on a weekend
void PandemicSim::doWeekendErrands()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "doWeekendErrands");

	//each person gets 3 errands
	int num_weekend_errands_total = NUM_WEEKEND_ERRANDS * number_people;

	//allocate arrays to store the errand locations

	//copy people's IDs and their 3 unique hours
	weekend_copyPeopleIndexes(&weekend_errand_people);
	weekend_generateThreeUniqueHours(&weekend_errand_hours);
	weekend_generateErrandDestinations(&weekend_errand_destinations);
	hipDeviceSynchronize();

	//extract a list of infected hours and destinations
	vec_t infected_hour_offsets(NUM_WEEKEND_ERRAND_HOURS + 1);

	//set up the infected for the errands
	//copies the errand hours and destinations for infected into a separate array,
	//finds the number of infected making an errand each hour, and assigns the number of contacts desired
	weekendErrand_doInfectedSetup(
		&weekend_errand_hours, &weekend_errand_destinations,
		&weekend_infectedPresentIndexes, &weekend_infectedLocations, &weekend_infectedContactsDesired,
		&infected_hour_offsets);

	//now sort the errand_people array into a large multi-hour location table
	thrust::sort_by_key(
		thrust::make_zip_iterator(thrust::make_tuple(weekend_errand_hours.begin(), weekend_errand_destinations.begin())),	//key.begin
		thrust::make_zip_iterator(thrust::make_tuple(weekend_errand_hours.end(), weekend_errand_destinations.end())),		//key.end
		weekend_errand_people.begin(),
		Pair_SortByFirstThenSecond_struct());									//data

	//count the number of people running errands on each hour
	//compute as count and offset for each hour
	vec_t errand_people_hour_offsets(NUM_WEEKEND_ERRAND_HOURS + 1);
	thrust::lower_bound(
		weekend_errand_hours.begin(),		//data.first
		weekend_errand_hours.end(),			//data.last
		thrust::counting_iterator<int>(0),		//search_val.first
		thrust::counting_iterator<int>(NUM_WEEKEND_ERRAND_HOURS), //search_val.last
		errand_people_hour_offsets.begin());
	errand_people_hour_offsets[NUM_WEEKEND_ERRAND_HOURS] = num_weekend_errands_total;

	if(print_location_info || log_location_info){
//		printf("dumping weekend errand setup...\n");
//		hipDeviceSynchronize();
//		dump_weekend_errands(errand_people, errand_hours, errand_locations, 5, number_people);
	}

	//for each hour, set up location arrays and make contacts
	for(int hour = 0; hour < NUM_WEEKEND_ERRAND_HOURS; hour++)
	{
		if(debug_log_function_calls)
		{
			fprintf(fDebug, "---------------------\nbeginning errand hour %d\n---------------------\n", hour);
			fflush(fDebug);
		}

		//get fancy string for debug output
		std::ostringstream s;
		s << "weekend_errand_";
		s << hour;
		std::string str = s.str();

		int people_offset = errand_people_hour_offsets[hour];		//index of first person for this hour
		int people_count = errand_people_hour_offsets[hour+1] - errand_people_hour_offsets[hour];		//number of people out on an errand this hour

		int infected_offset = infected_hour_offsets[hour];		//offset into infected_present of first infected person making contacts this hour
		int infected_present_count = infected_hour_offsets[hour+1] - infected_offset;	//number of infected making contacts this hour

		//build location offset table
		vec_t location_offsets(number_workplaces + 1);
		thrust::counting_iterator<int> count_iterator(0);
		thrust::lower_bound(
			weekend_errand_destinations.begin() + people_offset,
			weekend_errand_destinations.begin() + people_offset + people_count,
			count_iterator,
			count_iterator + number_workplaces,
			location_offsets.begin());
		location_offsets[number_workplaces] = people_count;
		
		clipContactsDesired_byLocationCount(
			weekend_infectedLocations.data() + infected_offset,		//infected locations for this hour
			infected_count, &location_offsets,	//number of infected and the location offset table
			weekend_infectedContactsDesired.data() + infected_offset);
		
		launchContactsKernel(
			str.c_str(),
			weekend_infectedPresentIndexes.data() + infected_offset,		//iterator to first infected index
			weekend_infectedLocations.data() + infected_offset,		//iterator to first infected destination
			weekend_infectedContactsDesired.data() + infected_offset,	//iterator to first infected contacts_desired
			infected_present_count,									//number of infected present
			thrust::raw_pointer_cast(weekend_errand_people.data() + people_offset),	//pointer to first person in the location_people table
			&location_offsets,		//pointer to location offset table
			number_workplaces);		//number_locations

		//validate contacts
		if(print_contacts || log_contacts)
		{
			int contacts_this_hour = thrust::reduce(
				weekend_infectedContactsDesired.begin() + infected_offset, 
				weekend_infectedContactsDesired.begin() + infected_offset + infected_present_count);

			validate_weekend_errand_contacts(
				str.c_str(),
				weekend_errand_people.data() + people_offset, 
				weekend_errand_destinations.data() + people_offset,
				people_count,
				location_offsets.data(), number_workplaces,
				contacts_this_hour);
		}

		if(debug_log_function_calls)
			debug_print("errand hour complete");
	}

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, infected_count);
}


//copies ID numbers into array for weekend errands
//we want three copies of each name, spaced out in collation style
__global__ void copy_weekend_errand_indexes_kernel(int * id_array, int N)
{	
	for(int myPos = blockIdx.x * blockDim.x + threadIdx.x;  myPos < N; myPos += gridDim.x * blockDim.x)
	{
		id_array[myPos] = myPos;
		id_array[myPos + N] = myPos;
		id_array[myPos + N + N] = myPos;
	}
}

//copies indexes 3 times into array, i.e. for IDS 1-3 produces array:
// 1 2 3 1 2 3 1 2 3
void PandemicSim::weekend_copyPeopleIndexes(vec_t * index_arr)
{
	int * index_arr_ptr = thrust::raw_pointer_cast(index_arr->data());
	copy_weekend_errand_indexes_kernel<<<cuda_blocks, cuda_threads>>>(index_arr_ptr, number_people);
}

//gets three UNIQUE errand hours 
__global__ void weekend_errand_hours_kernel(int * hours_array, int N, int rand_offset)
{
	threefry2x32_key_t tf_k = {{SEED_DEVICE[0], SEED_DEVICE[1]}};
	union{
		threefry2x32_ctr_t c;
		unsigned int i[2];
	} u;
	const int RNG_COUNTS_CONSUMED = 2;

	//for each person in simulation
	for(int myPos = blockIdx.x * blockDim.x + threadIdx.x;  myPos < N; myPos += gridDim.x * blockDim.x)
	{
		threefry2x32_ctr_t tf_ctr = {{(myPos * RNG_COUNTS_CONSUMED) + rand_offset, (myPos * RNG_COUNTS_CONSUMED) + rand_offset }};
		u.c = threefry2x32(tf_ctr, tf_k);

		int first, second, third;

		//get first hour
		first = u.i[0] % NUM_WEEKEND_ERRAND_HOURS;

		//get second hour, if it matches then increment
		second = u.i[1] % NUM_WEEKEND_ERRAND_HOURS;
		if(second == first)
			second = (second + 1) % NUM_WEEKEND_ERRAND_HOURS;

		threefry2x32_ctr_t tf_ctr_2 = {{(myPos * RNG_COUNTS_CONSUMED) + rand_offset + 1, (myPos * RNG_COUNTS_CONSUMED) + rand_offset + 1}};
		u.c = threefry2x32(tf_ctr_2, tf_k);

		//get third hour, increment until it no longer matches
		third = u.i[0] % NUM_WEEKEND_ERRAND_HOURS;
		while(third == first || third == second)
		{
			third = (third + 1 ) % NUM_WEEKEND_ERRAND_HOURS;
		}

		//store in output array
		hours_array[myPos] = first;
		hours_array[myPos + N] = second;
		hours_array[myPos + N + N] = third;
	}
}

//gets 3 DIFFERENT errand hours for each person, collated order
//i.e. 1 2 3 1 2 3 1 2 3
void PandemicSim::weekend_generateThreeUniqueHours(vec_t * hours_array)
{
	int * loc_arr_ptr = thrust::raw_pointer_cast(hours_array->data());
	weekend_errand_hours_kernel<<<cuda_blocks, cuda_threads>>>(loc_arr_ptr, number_people, rand_offset);
	rand_offset += number_people * 2;
}

__device__ int device_fishWeekendErrandDestination(float y)
{
	int row = FIRST_WEEKEND_ERRAND_ROW;
	while(row < NUM_BUSINESS_TYPES - 1 && y > weekend_errand_pdf[row])
	{
		y -= weekend_errand_pdf[row];
		row++;
	}
	y = y / weekend_errand_pdf[row];
	int business_num = y * (float) business_type_count[row];
	business_num += business_type_count_offset[row];

	return business_num;
}

//gets three errand locations for each person in collation style
__global__ void weekend_errand_locations_kernel(int * location_array, int N, int rand_offset)
{
	threefry2x32_key_t tf_k = {{SEED_DEVICE[0], SEED_DEVICE[1]}};
	union{
		threefry2x32_ctr_t c;
		unsigned int i[2];
	} u;

	//the number of times we will call the RNG
	const int RNG_COUNTS_CONSUMED = 2;

	//for each person in simulation
	for(int myPos = blockIdx.x * blockDim.x + threadIdx.x;  myPos < N; myPos += gridDim.x * blockDim.x)
	{
		//set up random number generator
		threefry2x32_ctr_t tf_ctr = {{(myPos * RNG_COUNTS_CONSUMED) + rand_offset, (myPos * RNG_COUNTS_CONSUMED) + rand_offset}};
		u.c = threefry2x32(tf_ctr, tf_k);

		//get first location - fish the type from the PDF
		float y_a =  (float) u.i[0] / UNSIGNED_MAX;
		int first = device_fishWeekendErrandDestination(y_a);

		//second 
		float y_b = (float) u.i[1] / UNSIGNED_MAX;
		int second = device_fishWeekendErrandDestination(y_b);

		//set up a second RNG run - use one number higher than the last RNG run
		threefry2x32_ctr_t tf_ctr_2 = {{(myPos * RNG_COUNTS_CONSUMED) + (rand_offset + 1), (myPos * RNG_COUNTS_CONSUMED) + (rand_offset + 1)}};
		u.c = threefry2x32(tf_ctr_2, tf_k);

		//third
		float y_c = (float) u.i[0] / UNSIGNED_MAX;
		int third = device_fishWeekendErrandDestination(y_c);

		location_array[myPos] = first;
		location_array[myPos + N] = second;
		location_array[myPos + N + N] = third;
	}
}

//gets 3 errand locations for each person according to PDF with collated order
//i.e. 1 2 3 1 2 3 1 2 3
void PandemicSim::weekend_generateErrandDestinations(vec_t * location_array)
{
	const int RNG_COUNTS_CONSUMED = 2;
	int * loc_arr_ptr = thrust::raw_pointer_cast(location_array->data());
	weekend_errand_locations_kernel<<<cuda_blocks, cuda_threads>>>(loc_arr_ptr, number_people, rand_offset);
	rand_offset += number_people * RNG_COUNTS_CONSUMED;
}

//prints some of the weekend errands to console
void PandemicSim::dump_weekend_errands(d_vec people, d_vec hours, d_vec locations, int num_to_print, int N)
{
	h_vec h_people = people;
	h_vec h_hours = hours;
	h_vec h_locs = locations;

	for(int i = 0; i < num_to_print; i++)
	{
		printf("i: %d\tidx: %6d\thour: %d\tloc: %3d\n",
			i, h_people[i], h_hours[i], h_locs[i]);
		printf("i: %d\tidx: %6d\thour: %d\tloc: %3d\n",
			i, h_people[i + N], h_hours[i + N], h_locs[i + N]);
		printf("i: %d\tidx: %6d\thour: %d\tloc: %3d\n",
			i, h_people[i + (N + N)], h_hours[i + (N + N)], h_locs[i + (N + N)]);
	}
}

//helper function that will automatically generate a list of contacts_desired
//uses the max_contacts number for each location
void PandemicSim::makeContacts_byLocationMax(const char * hour_string,
										vec_t *infected_list, int infected_list_count,
										vec_t *loc_people, vec_t *loc_max_contacts,
										vec_t *loc_offsets, int num_locs,
										vec_t *people_lookup)
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "makeContacts_byLocationMax");

	//get the locations of infected people
	//ASSUMES: people_locations[i] contains the location of person index i (all people are present)
	vec_t infected_locations(infected_list_count);
	thrust::gather(
		(*infected_list).begin(),	//map.begin
		(*infected_list).begin() + infected_list_count,		//map.end
		(*people_lookup).begin(),
		infected_locations.begin());

	//get contacts desired for each infected person
	//return max_contacts, or count if count < max_contacts, or 0 if count == 1
	vec_t contacts_desired(infected_list_count);
	buildContactsDesired_byLocationMax(
		&infected_locations, infected_list_count,
		loc_offsets, loc_max_contacts,
		&contacts_desired);

	//get total number of contacts this hour 
	int num_new_contacts = thrust::reduce(contacts_desired.begin(), contacts_desired.end());

	//get raw pointer into the location table
	int * location_people_ptr = thrust::raw_pointer_cast((*loc_people).data());

	//make contacts
	launchContactsKernel(
		hour_string,
		infected_list, &infected_locations, 
		&contacts_desired, infected_list_count,
		location_people_ptr, loc_offsets, num_locs);

	//validate the contacts
	if(log_contacts || print_contacts)
	{
		validate_contacts(hour_string, loc_people, people_lookup, loc_offsets, num_new_contacts);
	}

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, infected_count);
}
//method to set up afterschool activities and errands and make contacts
//children go to one randomly selected afterschool activity for 2 hours
//adults will go to two randomly selected errands, and make 2 contacts split between them
void PandemicSim::doWeekdayErrands()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "doWeekdayErrands");

	//errand arrays, will hold adults and children together
	vec_t errand_people_lookup(number_people);

	//generate child afterschool activities and first set of adult errands
	weekday_scatterAfterschoolLocations(&errand_people_lookup);
	weekday_scatterErrandLocations(&errand_people_lookup); 
	hipDeviceSynchronize();

	//generate list of IDs to sort by location
	vec_t errand_location_people(number_people);
	thrust::sequence(errand_location_people.begin(), errand_location_people.end());

	//allocate and build the location offset array
	vec_t errand_location_offsets(number_workplaces + 1);
	calcLocationOffsets(&errand_location_people, errand_people_lookup, &errand_location_offsets, number_people, number_workplaces);

	if(debug_log_function_calls)
		debug_print("first errand locations built");

	//make children errands first
	vec_t infected_children(infected_count);
	filterInfectedByPopulationGroup("afterschool", &people_child_indexes, &infected_children);
	makeContacts_byLocationMax(
		"afterschool",
		&infected_children, infected_children.size(),
		&errand_location_people, &workplace_max_contacts,
		&errand_location_offsets, number_workplaces,
		&errand_people_lookup);


	/////////////now do adult errands, first hour

	//get a list of infected adults
	vec_t infected_adults(infected_count);
	filterInfectedByPopulationGroup("errand1", &people_adult_indexes, &infected_adults);
	int infected_adults_count = infected_adults.size();

	//assign 2 contacts randomly between the 2 errands
	vec_t errand_contacts_desired(infected_adults_count);
	assign_weekday_errand_contacts(&errand_contacts_desired, infected_adults_count);

	makeContacts_byContactsDesiredArray(
		"errand1",
		&infected_adults, infected_adults_count,
		&errand_location_people, &errand_contacts_desired,
		&errand_location_offsets, number_workplaces,
		&errand_people_lookup);

	if(debug_log_function_calls)
		debug_print("first errand complete and validated");


	//////////////generate second errand hour and make contacts

	//get new locations for adults
	weekday_scatterErrandLocations(&errand_people_lookup);
	hipDeviceSynchronize();
	
	//rebuild the location offset array
	thrust::sequence(errand_location_people.begin(), errand_location_people.end());
	calcLocationOffsets(&errand_location_people, errand_people_lookup, &errand_location_offsets,number_people, number_workplaces);

	if(debug_log_function_calls)
		debug_print("second errand location array built");

	//reassign contacts: 2 - [contacts on first errand]
	thrust::transform(
		thrust::constant_iterator<int>(2),			//first.begin
		thrust::constant_iterator<int>(2) + infected_adults_count,	//first.end
		errand_contacts_desired.begin(),		//second.begin
		errand_contacts_desired.begin(),		//output - in place
		thrust::minus<int>());		

	makeContacts_byContactsDesiredArray(
		"errand2",		
		&infected_adults, infected_adults_count,
		&errand_location_people, &errand_contacts_desired,
		&errand_location_offsets, number_workplaces,
		&errand_people_lookup);

	if(debug_log_function_calls)
		debug_print("second errand complete and validated");

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, infected_count);
}

//kernel gets a random afterschool location for each child
__global__
	void get_afterschool_locations_kernel(int * child_indexes_arr, int * output_array, int number_children, int rand_offset)
{
	threefry2x32_key_t tf_k = {{SEED_DEVICE[0], SEED_DEVICE[1]}};
	union{
		threefry2x32_ctr_t c;
		unsigned int i[2];
	} u;

	//get the number of afterschool locations and their offset in the business array
	int afterschool_count = business_type_count[BUSINESS_TYPE_AFTERSCHOOL];
	int afterschool_offset = business_type_count_offset[BUSINESS_TYPE_AFTERSCHOOL];

	//for each child
	for(int myPos = blockIdx.x * blockDim.x + threadIdx.x;  myPos < number_children; myPos += gridDim.x * blockDim.x)
	{
		threefry2x32_ctr_t tf_ctr = {{myPos + rand_offset, myPos + rand_offset }};
		u.c = threefry2x32(tf_ctr, tf_k);

		//get a random float
		float frac = (float) u.i[0] / UNSIGNED_MAX;
		int ret = frac * afterschool_count;		//find which afterschool location they're at, between 0 <= X < count
		ret = ret + afterschool_offset;		//add the offset to the first afterschool location

		//scatter the afterschool locations into the lookup table according to the children's indexes
		int output_offset = child_indexes_arr[myPos];
		output_array[output_offset] = ret;
	}
}

//starts the kernel that gets a random afterschool location for each child.
void PandemicSim::weekday_scatterAfterschoolLocations(vec_t * people_locs)
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "weekday_scatterAfterschoolLocations");

	int * children_idxes_ptr = thrust::raw_pointer_cast(people_child_indexes.data());
	int * output_arr_ptr = thrust::raw_pointer_cast(people_locs->data());

	get_afterschool_locations_kernel<<<cuda_blocks, cuda_threads>>>(children_idxes_ptr, output_arr_ptr, number_children, rand_offset);

	rand_offset += number_children;

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, number_children);
}


__device__ int device_fishWeekdayErrand(float yval)
{
	int row = FIRST_WEEKDAY_ERRAND_ROW; //which business type

	while(yval > weekday_errand_pdf[row] && row < (NUM_BUSINESS_TYPES - 1))
	{
		yval -= weekday_errand_pdf[row];
		row++;
	}

	//figure out which business of this type we're at
	float frac = yval / weekday_errand_pdf[row];
	int business_num = frac * business_type_count[row];

	//add the offset to the first business of this type 
	int offset = business_type_count_offset[row];
	business_num += offset;

	return business_num;
}

//gets a random weekday errand location for each adult, where N = number_adults
__global__ void get_weekday_errand_locations_kernel(int * adult_indexes_arr, int * output_arr, int number_adults, int rand_offset)
{

	threefry2x32_key_t tf_k = {{SEED_DEVICE[0], SEED_DEVICE[1]}};
	union{
		threefry2x32_ctr_t c;
		unsigned int i[2];
	} u;

	//for each adult
	for(int myPos = blockIdx.x * blockDim.x + threadIdx.x;  myPos < number_adults; myPos += gridDim.x * blockDim.x)
	{
		int myRandOffset = myPos + rand_offset;

		threefry2x32_ctr_t tf_ctr = {{myRandOffset,myRandOffset}};
		u.c = threefry2x32(tf_ctr, tf_k);

		//fish out a business type
		float yval = (float) u.i[0] / UNSIGNED_MAX;
		int errand_destination = device_fishWeekdayErrand(yval);

		int output_offset = adult_indexes_arr[myPos];
		output_arr[output_offset] = errand_destination;

		//TODO:  use other rand number
	}
}

//gets a random weekday errand location for each adult
void PandemicSim::weekday_scatterErrandLocations(d_vec * people_lookup)
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "weekday_scatterErrandLocations");

	int * adult_indexes_ptr = thrust::raw_pointer_cast(people_adult_indexes.data());
	int * output_arr_ptr = thrust::raw_pointer_cast((*people_lookup).data());

	//start kernel
	get_weekday_errand_locations_kernel<<<cuda_blocks, cuda_threads>>>(adult_indexes_ptr, output_arr_ptr, number_adults, rand_offset);
	rand_offset += number_adults;

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, number_adults);
}


//for each infected adult, get the number of contacts they will make at their first errand
//valid outputs are {0,1,2}
__global__ void errand_contacts_kernel(int * array, int N, int global_rand_offset)
{		
	threefry2x32_key_t tf_k = {{SEED_DEVICE[0], SEED_DEVICE[1]}};
	union{
		threefry2x32_ctr_t c;
		unsigned int i[2];
	} u;

	for(int myPos = blockIdx.x * blockDim.x + threadIdx.x;  myPos < N; myPos += gridDim.x * blockDim.x)
	{

		int myRandOffset = myPos + global_rand_offset;

		threefry2x32_ctr_t tf_ctr = {{myRandOffset,0xfacecafe}};
		u.c = threefry2x32(tf_ctr, tf_k);

		array[myPos] = u.i[0] % 3;


		//TODO:  use other rand number
	}
}

//for each infected adult, get the number of contacts they will make at their first errand
//valid outputs are {0,1,2}
void PandemicSim::assign_weekday_errand_contacts(d_vec * contacts_desired, int num_infected_adults)
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "assign_weekday_errand_contacts");

	int * arr_ptr = thrust::raw_pointer_cast(contacts_desired->data());

	//start kernel
	errand_contacts_kernel<<<cuda_blocks, cuda_threads>>>(arr_ptr, num_infected_adults, rand_offset);
	hipDeviceSynchronize();

	rand_offset += num_infected_adults;

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, num_infected_adults);
}

__global__ void build_contacts_desired_kernel(
			int *infected_location_arr, 
			int *loc_offset_arr, int *loc_max_contacts_arr, 
			int *contacts_desired_arr,
			int num_infected)
{
	for(int myPos = blockIdx.x * blockDim.x + threadIdx.x;  myPos < num_infected; myPos += gridDim.x * blockDim.x)
	{
		int myLoc = infected_location_arr[myPos];

		//try to make maximum number of contacts possible
		int desired = loc_max_contacts_arr[myLoc];
		int loc_count = loc_offset_arr[myLoc + 1] - loc_offset_arr[myLoc];	//get number of people at location

		if(loc_count == 1)
			contacts_desired_arr[myPos] = 0;
		else
			contacts_desired_arr[myPos] = desired;
	}
}

void PandemicSim::buildContactsDesired_byLocationMax(
	vec_t *infected_locations, int num_infected,
	vec_t *loc_offsets,
	vec_t *loc_max_contacts,
	vec_t *contacts_desired)
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "buildContactsDesired_byLocationMax");

	int * infected_loc_ptr = thrust::raw_pointer_cast((*infected_locations).data());
	int * loc_offsets_ptr = thrust::raw_pointer_cast((*loc_offsets).data());
	int * max_contacts_ptr = thrust::raw_pointer_cast((*loc_max_contacts).data());
	int * contacts_desired_ptr = thrust::raw_pointer_cast((*contacts_desired).data());

	build_contacts_desired_kernel<<<cuda_blocks,cuda_threads>>>(infected_loc_ptr,loc_offsets_ptr,max_contacts_ptr,contacts_desired_ptr,num_infected);
	hipDeviceSynchronize();

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, num_infected);
}

//prints some of the weekend errands to console
void dump_weekend_errands(d_vec people, d_vec hours, d_vec locations, int num_to_print, int N)
{
	h_vec h_people = people;
	h_vec h_hours = hours;
	h_vec h_locs = locations;

	for(int i = 0; i < num_to_print; i++)
	{
		printf("i: %d\tidx: %6d\thour: %d\tloc: %3d\n",
			i, h_people[i], h_hours[i], h_locs[i]);
		printf("i: %d\tidx: %6d\thour: %d\tloc: %3d\n",
			i, h_people[i + N], h_hours[i + N], h_locs[i + N]);
		printf("i: %d\tidx: %6d\thour: %d\tloc: %3d\n",
			i, h_people[i + (N + N)], h_hours[i + (N + N)], h_locs[i + (N + N)]);
	}
}

//Randomly select people at the same location as the infector as contacts
//NOTE: assumes that a location count of 1 means that contacts_desired = 0 
//		for anyone at that location otherwise, someone could select themselves
__global__ void victim_index_kernel(
	int * infector_indexes_arr, int * contacts_desired_arr, int * output_offset_arr, int * infector_loc_arr,
	int * location_offsets_arr, int * location_people_arr,
	int * contact_infector_arr, int * contact_idx_arr,
	int N, int rand_offset)
{
	threefry2x32_key_t tf_k = {{SEED_DEVICE[0], SEED_DEVICE[1]}};
	union{
		threefry2x32_ctr_t c;
		unsigned int i[2];
	} u;

	//for each infector
	for(int myPos = blockIdx.x * blockDim.x + threadIdx.x;  myPos < N; myPos += gridDim.x * blockDim.x)
	{
		//info about the infector 
		int infector_idx = infector_indexes_arr[myPos];
		int output_offset = output_offset_arr[myPos];
		int contacts_desired = contacts_desired_arr[myPos];	//stores number of contacts we still want

		//info about the location
		int myLoc = infector_loc_arr[myPos];
		int loc_offset = location_offsets_arr[myLoc];
		int loc_count = location_offsets_arr[myLoc+1] - loc_offset;	//

		//get N contacts
		while(contacts_desired > 0)
		{
			threefry2x32_ctr_t tf_ctr = {{output_offset + rand_offset, output_offset + rand_offset + 1}};
			u.c = threefry2x32(tf_ctr, tf_k);

			//randomly select person at location
			int victim_offset = u.i[0] % loc_count;
			int victim_idx = location_people_arr[loc_offset + victim_offset];

			//if we have selected the infector, we need to get a different person
			if(victim_idx == infector_idx)
			{
				//get the next person
				victim_offset = (victim_offset + 1);
				if(victim_offset == loc_count)		//wrap around if needed
					victim_offset = 0;
				victim_idx = location_people_arr[loc_offset + victim_offset];
			}

			//save contact into output array
			contact_infector_arr[output_offset] = infector_idx;
			contact_idx_arr[output_offset] = victim_idx;
			output_offset++;	//move to next output slot
			contacts_desired--;	//decrement contacts remaining

			//R123 returns 2x32 bit numbers, so if we need another one we can use that
			if(contacts_desired > 0)
			{
				//randomly select person at that location
				victim_offset = u.i[1] % loc_count;
				int victim_idx = location_people_arr[loc_offset + victim_offset];

				//if we have selected the infector, get the next person
				if(victim_idx == infector_idx)
				{
					victim_offset = (victim_offset + 1);
					if(victim_offset == loc_count)		//wrap around
						victim_offset = 0;
					victim_idx = location_people_arr[loc_offset + victim_offset];
				}

				//save contact into output array
				contact_infector_arr[output_offset] = infector_idx;
				contact_idx_arr[output_offset] = victim_idx;
				output_offset++;
				contacts_desired--;
			}

		}  //end while(contacts_desired)
	}	//end for each infector
}



//this function does final setup and then calls the kernel to make contacts
//contacts can be validated afterwards if desired
void PandemicSim::launchContactsKernel(
	const char * hour_string, 
	d_ptr infected_indexes_present_begin, d_ptr infected_locations_begin, 
	d_ptr infected_contacts_desired_begin, int infected_present_count,
	int * loc_people_ptr, vec_t *location_offsets, int num_locs)
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "launchContactsKernel");

	if(debug_log_function_calls)
		debug_print("inside make_contacts");

	//each infected needs to know where to put their contacts in the contacts array
	vec_t output_offsets(infected_present_count);
	thrust::exclusive_scan(
		infected_contacts_desired_begin,
		infected_contacts_desired_begin + infected_present_count,
		output_offsets.begin());

	//how many contacts we are about to make in total
	int new_contacts = thrust::reduce(infected_contacts_desired_begin, infected_contacts_desired_begin + infected_present_count);

	//if the contacts array is too small, resize it to fit
	if(daily_contacts + new_contacts > daily_contact_infectors.size())
	{
		printf("warning:  contacts too small, old size: %d new size: %d\n", daily_contact_infectors.size(), daily_contacts + new_contacts);
		daily_contact_infectors.resize(daily_contacts + new_contacts);
		daily_contact_victims.resize(daily_contacts + new_contacts);
		//daily_contact_kvals.resize(daily_contacts + new_contacts);
	}

	//get raw pointers to infector data
	int * infected_idx_ptr = thrust::raw_pointer_cast(infected_indexes_present_begin);
	int * infected_loc_ptr = thrust::raw_pointer_cast(infected_locations_begin);
	int * infected_contacts_ptr = thrust::raw_pointer_cast(infected_contacts_desired_begin);
	int * output_offsets_ptr = thrust::raw_pointer_cast(output_offsets.data());

	//get raw pointers to location data
	int * loc_offsets_ptr = thrust::raw_pointer_cast(location_offsets->data());
	
	//get raw pointers into output array, and advance past spots already filled
	int * contact_infector_ptr = thrust::raw_pointer_cast(daily_contact_infectors.data());
	contact_infector_ptr += daily_contacts;
	int * contact_victim_ptr = thrust::raw_pointer_cast(daily_contact_victims.data());
	contact_victim_ptr += daily_contacts;

	if(print_contact_kernel_setup|| log_contact_kernel_setup)
	{
		dump_contact_kernel_setup(
			hour_string,
			infected_indexes_present_begin, infected_locations_begin, 
			infected_contacts_desired_begin, &output_offsets,
			loc_people_ptr, location_offsets, 
			infected_present_count);
	}

	if(CONSOLE_OUTPUT)
	{
		printf("daily contacts: %d\t new_contacts: %d\t rand_offset: %d\n", daily_contacts, new_contacts, rand_offset);

		//	printf("infected_present size: %d\ninfected present: %d\ninfected_locations size: %d\ninfected_contacts_desired size: %d\nloc_offsets size: %d\nloc_counts size: %d\nnum_locs: %d\n",
		//			infected_indexes_present.size(), infected_present, infected_locations.size(), infected_contacts_desired.size(), location_offsets.size(), location_counts.size(), num_locs);

	}


	//call the kernel
	if(debug_log_function_calls)
		debug_print("calling contacts kernel");
	victim_index_kernel<<<cuda_blocks, cuda_threads>>>(
		infected_idx_ptr, infected_contacts_ptr, output_offsets_ptr, infected_loc_ptr,
		loc_offsets_ptr, loc_people_ptr, 
		contact_infector_ptr, contact_victim_ptr, 
		infected_present_count, rand_offset);
	hipDeviceSynchronize();

	if(debug_log_function_calls)
		debug_print("contacts kernel sync'd");

	//increment the contacts counter and the RNG counter
	rand_offset += new_contacts;
	daily_contacts += new_contacts;

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, infected_present_count); 
}

//dumps the setup that is passed to the make_contacts kernel
void PandemicSim::dump_contact_kernel_setup(
	const char * hour_string,
	d_ptr infected_indexes_present_begin, d_ptr infected_locations_begin,
	d_ptr infected_contacts_desired_begin, d_vec * output_offsets,
	int * location_people_ptr, d_vec *location_offsets,
	int num_infected)
{
	h_vec i_idx(num_infected);
	thrust::copy(infected_indexes_present_begin, infected_indexes_present_begin + num_infected, i_idx.begin());

	h_vec i_loc(num_infected);
	thrust::copy(infected_locations_begin, infected_locations_begin + num_infected, i_loc.begin());

	h_vec i_contacts_desired(num_infected);
	thrust::copy(infected_contacts_desired_begin, infected_contacts_desired_begin + num_infected, i_contacts_desired.begin());

	h_vec o_o = (*output_offsets);

	h_vec loc_o = (*location_offsets);

	for(int i = 0; i < num_infected; i++)
	{
		int idx = i_idx[i];
		int loc = i_loc[i];
		int c_d = i_contacts_desired[i];
		int offset = o_o[i] + daily_contacts;

		int loc_offset = loc_o[loc];
		int loc_count = loc_o[loc+1] - loc_offset;

		if(print_contact_kernel_setup)
			printf("inf %3d\tidx: %4d\tloc: %3d\tloc_offset: %5d\tloc_count: %5d\tdesired: %d\tout_offset: %d\n",
				i, idx, loc, loc_offset, loc_count, c_d, offset);
		if(log_contact_kernel_setup)
			fprintf(fContactsKernelSetup,"%d,%s,%d,%d,%d,%d,%d,%d,%d\n",
				current_day, hour_string, i, idx, loc, loc_offset, loc_count, c_d, offset);
	}

	if(log_contact_kernel_setup)
		fflush(fContactsKernelSetup);

}


//this is a helper method to copy contacts from the device back to the host
void PandemicSim::validate_contacts(const char * hour_string, d_vec *d_people, d_vec *d_lookup, d_vec *d_offsets, int N)
{	
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "validate_contacts");
	if(debug_log_function_calls)
		debug_print("validating contacts");

	//copy data to host
	h_vec h_lookup(number_people);
	thrust::copy(d_lookup->begin(), d_lookup->begin() + number_people, h_lookup.begin());

	//these are needed to look into the people array to verify that both people are actually present
	//however doublechecking this is slow, so these are only copied if needed
	h_vec h_people;
	h_vec h_offsets;
	
	if(DOUBLECHECK_CONTACTS)
	{
		h_people.resize(d_people->size());
		thrust::copy(d_people->begin(), d_people->end(), h_people.begin());

		h_offsets.resize(d_offsets->size());
		thrust::copy(d_offsets->begin(), d_offsets->end(), h_offsets.begin());
	}

	int start_idx = daily_contacts - N;
	h_vec contact_infectors(N);
	thrust::copy_n(daily_contact_infectors.begin() + start_idx, N, contact_infectors.begin());
	h_vec contact_victims(N);
	thrust::copy_n(daily_contact_victims.begin() + start_idx, N, contact_victims.begin());

	validate_contacts(hour_string,&h_people, &h_lookup, &h_offsets, &contact_infectors, &contact_victims, N);

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, N);
}

//This method checks that the N contacts most recently generated are valid
//It should be called immediately after make_contacts
//Specifically, it checks that the infector and victim have the same location
//if the DOUBLECHECK_CONTACTS define is set to true, it will actually look in the location table to be sure (very slow)
void PandemicSim::validate_contacts(const char * hour_string, h_vec * h_people, h_vec *h_lookup, h_vec *h_offsets, h_vec *contact_infectors, h_vec *contact_victims, int N)
{
	if(print_contacts)
		printf("validating %d contacts...\n", N);

	if(0)
	{
		printf("h_people size: %d\n", h_people->size());
		printf("h_lookup size: %d\n", h_lookup->size());
		printf("h_offsets size: %d\n", h_offsets->size());
		printf("c_i size: %d\n", contact_infectors->size());
		printf("c_v size: %d\n", contact_victims->size());
		printf("daily contacts: %d\n", daily_contacts);
	}


	for(int i = 0; i < N; i++)
	{
		int infector_idx = (*contact_infectors)[i];
		int i_loc = (*h_lookup)[infector_idx];

		int victim_idx = (*contact_victims)[i];
		int v_loc = (*h_lookup)[victim_idx];

		int i_found = 0;
		int v_found = 0;

		if(DOUBLECHECK_CONTACTS)
		{
			if(i_loc == v_loc)
			{
				int loc_o = (*h_offsets)[i_loc];
				int loc_c = (*h_offsets)[i_loc + 1] - loc_o;
				for(int k = loc_o; k < loc_o + loc_c; k++)
				{
					if((*h_people)[k] == infector_idx)
						i_found = 1;
					if((*h_people)[k] == victim_idx)
						v_found = 1;

					if(i_found && v_found)
						break;
				}
			}
		}
		else
		{
			i_found = i_loc == v_loc;
			v_found = i_loc == v_loc;
		}


		int contact_number = (daily_contacts - N) + i;
		//current_day, i, infector_idx, victim_idx, infector_loc, victim_loc, infector_found, victim_found
		if(log_contacts)
			fprintf(fContacts, "%d, %d, %s, %d, %d, %d, %d, %d, %d\n",
			current_day, contact_number, hour_string, infector_idx, victim_idx, i_loc, v_loc, i_found, v_found);
		if(print_contacts)
			printf("%d\tinf_idx: %5d\tvic_idx: %5d\ti_loc: %5d\tv_loc: %5d\ti_found: %d v_found: %d\n",
			contact_number, infector_idx, victim_idx, i_loc, v_loc, i_found, v_found);
	}

	if(debug_log_function_calls)
		debug_print("contact validation complete");

	fflush(fContacts);
	fflush(fDebug);
}


//This method consumes the accumulated contacts, and causes infections and recovery to occur
void PandemicSim::dailyUpdate()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "dailyUpdate");

	if(debug_log_function_calls)
		debug_print("beginning daily update");

	if(CONSOLE_OUTPUT)
	{
		printf("average: %.2f contacts per infected\n", (float) daily_contacts / infected_count);
	}


	//process contacts into actions
	daily_contactsToActions();

	//filter invalid actions - not susceptible, duplicate, etc
	daily_filterActions();

	if(log_actions_filtered || print_actions_filtered)
		dump_actions_filtered();

	//counts the number of each generation in today's new infections 
	daily_countReproductionNumbers(ACTION_INFECT_PANDEMIC);
	daily_countReproductionNumbers(ACTION_INFECT_SEASONAL);

	//heals infected who are reaching culmination, and adds newly infected people
	daily_rebuildInfectedArray();


	if(CONSOLE_OUTPUT)
	{
		printf("now %d infected\n",infected_count);
		printf("update complete\n");
	}

	if(debug_log_function_calls)
		debug_print("daily update complete");

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, infected_count);
}

//after the contacts have been converted to actions, build an array of generations for the victims
//DEPRECATED: rolled into the contacts kernel
void PandemicSim::deprected_daily_assignVictimGenerations()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "build_action_generations");

	if(debug_log_function_calls)
		debug_print("building generations for actions");
	/*
	//find the ID of the infector
	d_vec inf_offset(daily_actions);
	thrust::lower_bound(
		infected_indexes.begin(),
		infected_indexes.begin() + infected_count,
		daily_action_infectors.begin(),
		daily_action_infectors.begin() + daily_actions,
		inf_offset.begin());

	//get the generation of the infector, add one, store in array
	thrust::transform(
		daily_action_type.begin(),			//input1.begin:  the actions to do, specifies which generation gets set
		daily_action_type.begin() + daily_actions,	//input1.end
		thrust::make_zip_iterator(thrust::make_tuple(
			thrust::make_permutation_iterator(infected_generation_pandemic.begin(), inf_offset.begin()),	//input2.begin: tuple (parent_gen_p,parent_gen_s)
			thrust::make_permutation_iterator(infected_generation_seasonal.begin(), inf_offset.begin()))),
		thrust::make_zip_iterator(thrust::make_tuple(
			daily_action_victim_gen_p.begin(), daily_action_victim_gen_s.begin())),	//output.begin
		generationOp());		//functor

		*/
	if(debug_log_function_calls)
		debug_print("generation actions built");

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, daily_actions);

}

//once the contacts have been properly set up, this kernel determines whether each contact
//is successful, and then outputs actions
__global__ void contacts_to_actions_kernel(
		int * output_offset_arr,
		int * inf_gen_p_arr, int * inf_gen_s_arr,
		int * day_p_arr, int * day_s_arr,
		int * action_types_arr,  
		int * victim_gen_p_arr, int * victim_gen_s_arr,
		// float * rand_1, float * rand_2, float* rand_3, float * rand_4,
		int num_infected, int global_rand_offset, int current_day)
{
	threefry2x32_key_t tf_k = {{SEED_DEVICE[0], SEED_DEVICE[1]}};
	union{
		threefry2x32_ctr_t c;
		unsigned int i[2];
	} u;
	
	//for each infected person
	for(int myPos = blockIdx.x * blockDim.x + threadIdx.x;  myPos < num_infected; myPos += gridDim.x * blockDim.x)
	{
		//output_offset holds the index where this action should be stored
		int output_offset = output_offset_arr[myPos];

		//how many contacts we're trying to make
		int contacts_desired = output_offset_arr[myPos + 1] - output_offset_arr[myPos];

		//these are the day this person was infected (from 0 to MAX_DAYS) - NOT which day of infection they are on
		int day_p = day_p_arr[myPos];
		int day_s = day_s_arr[myPos];

		//if they make a successful contact, the victim will receive these generations
		int victim_gen_p = inf_gen_p_arr[myPos] + 1;
		int victim_gen_s = inf_gen_s_arr[myPos] + 1;
			
		//start with zero probability of infection - if the person is infected, increase it
		float inf_prob_p = 0.0;
		
		if(day_p >= 0)
		{
			inf_prob_p = (infectiousness_profile[current_day - day_p] * BASE_REPRODUCTION_DEVICE[0]) / (float) contacts_desired;
//				inf_prob_p = day_p;
		}
			
		//same for seasonal
		float inf_prob_s = 0.0;
	
		if(day_s >= 0)
		{
			inf_prob_s = (infectiousness_profile[current_day - day_s] * BASE_REPRODUCTION_DEVICE[1]) / (float) contacts_desired;
//				inf_prob_s = day_s;
		}
			
		while(contacts_desired > 0)
		{
			//we need one random number set for each contact
			int myRandOffset = output_offset + global_rand_offset;
			
			threefry2x32_ctr_t tf_ctr = {{myRandOffset,myRandOffset}};
			u.c = threefry2x32(tf_ctr, tf_k);
				
			//convert uniform to float
			float f_pandemic = (float) u.i[0] / UNSIGNED_MAX;
			float f_seasonal = (float) u.i[1] / UNSIGNED_MAX;
			
			//if the random float is less than the infection threshold, infection succeeds
			bool infects_pandemic = f_pandemic < inf_prob_p;
			bool infects_seasonal = f_seasonal < inf_prob_s;
				
			//parse bools to action type
			if(infects_pandemic && infects_seasonal)
			{
				action_types_arr[output_offset] = ACTION_INFECT_BOTH;
				victim_gen_p_arr[output_offset] = victim_gen_p;
				victim_gen_s_arr[output_offset] = victim_gen_s;
			}
			else if(infects_pandemic)
			{
				action_types_arr[output_offset] = ACTION_INFECT_PANDEMIC;
				victim_gen_p_arr[output_offset] = victim_gen_p;
			}
			else if(infects_seasonal)
			{
				action_types_arr[output_offset] = ACTION_INFECT_SEASONAL;
				victim_gen_s_arr[output_offset] = victim_gen_s;
			}
			else
			{
				action_types_arr[output_offset] = ACTION_INFECT_NONE;
			}
				
			//for debug: we can output some internal values
//			rand_1[output_offset] = inf_prob_p;
//			rand_2[output_offset] = inf_prob_s;
//			rand_3[output_offset] = f_pandemic;
//			rand_4[output_offset] = f_seasonal;
				
			output_offset++;
			contacts_desired--;
		}
	}
}

//starts the kernel to convert contacts to actions
void PandemicSim::daily_contactsToActions()
{
	
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "contacts_to_actions");
	
	if(debug_log_function_calls)
		debug_print("beginning contacts-to-action setup");
	
	//sort contacts by infector
	thrust::sort_by_key(
			daily_contact_infectors.begin(),
			daily_contact_infectors.begin() + daily_contacts,
			daily_contact_victims.begin());
	
	//size our actino output arrays
	//moved to setup function, only done once to prevent fragmentation

	//get lower bound of each infector, aka index
	d_vec infector_contacts_offset (infected_count + 1);
	thrust::lower_bound(
			daily_contact_infectors.begin(),		//data.begin
			daily_contact_infectors.begin() + daily_contacts,		//data.end
			infected_indexes.begin(),		//value to search for
			infected_indexes.begin() + infected_count,			//value to search
			infector_contacts_offset.begin());
	infector_contacts_offset[infected_count] = daily_contacts;

	
	//convert thrust vectors to raw pointers
	int * inf_offset_ptr = thrust::raw_pointer_cast(infector_contacts_offset.data());
	
	int * day_p_ptr = thrust::raw_pointer_cast(infected_days_pandemic.data());
	int * day_s_ptr = thrust::raw_pointer_cast(infected_days_seasonal.data());

	int * inf_gen_p_ptr = thrust::raw_pointer_cast(infected_generation_pandemic.data());
	int * inf_gen_s_ptr= thrust::raw_pointer_cast(infected_generation_seasonal.data());

	//stores what type of action resulted
	int * actions_type_ptr = thrust::raw_pointer_cast(daily_action_type.data());

	int * victim_gen_p_ptr = thrust::raw_pointer_cast(daily_action_victim_gen_p.data());
	int * victim_gen_s_ptr = thrust::raw_pointer_cast(daily_action_victim_gen_s.data());
	
	//for debug: we can dump some internal stuff	
	float* rand1ptr = thrust::raw_pointer_cast(debug_float1.data());
	float* rand2ptr = thrust::raw_pointer_cast(debug_float2.data());
	float* rand3ptr = thrust::raw_pointer_cast(debug_float3.data());
	float* rand4ptr = thrust::raw_pointer_cast(debug_float4.data());

	if(debug_log_function_calls)
		debug_print("calling contacts-to-action kernel");
	
	//determine whether each infection was successful
	contacts_to_actions_kernel<<<cuda_blocks, cuda_threads>>>(
			inf_offset_ptr,
			inf_gen_p_ptr, inf_gen_s_ptr,
			day_p_ptr, day_s_ptr,
			actions_type_ptr, 
			victim_gen_p_ptr, victim_gen_s_ptr,
	//		rand1ptr, rand2ptr, rand3ptr, rand4ptr,
			infected_count, rand_offset, current_day);

	//copy the IDs of the infector and victim to the action array
	thrust::copy(daily_contact_victims.begin(), daily_contact_victims.begin() + daily_contacts, daily_action_victim_index.begin());
	hipDeviceSynchronize();
	
	rand_offset += daily_contacts; //increment rand counter
	daily_actions = daily_contacts; //stores # of actions to allow dumping - set in filter after filtering

	if(log_actions || print_actions)
		dump_actions(
//				rand1, rand2, rand3, rand4	//disable debug outputs from contacts kernel
				);

	/*
	if(0){
		h_vec actions = daily_action_type;
		thrust::host_vector<float> h_rand1 = rand1;
		thrust::host_vector<float> h_rand2 = rand2;
		thrust::host_vector<float> h_rand3 = rand3;
		thrust::host_vector<float> h_rand4 = rand4;
		for(int i = 0; i < daily_contacts; i++)
		{
			float t_p = h_rand1[i];
			float y_p = h_rand3[i];
			float t_s = h_rand2[i];
			float y_s = h_rand4[i];
			printf("action: %s\tthresh_p:%f \ty_p: %f\tinfects_p: %d\tthresh_s: %f\ty_s: %f\tinfects_s: %d\n", action_type_to_char(actions[i]), t_p, y_p, y_p < t_p, t_s, y_s, y_s < t_s);
		}
	} */
	

	if(debug_log_function_calls)
		debug_print("contacts_to_actions complete");
	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, infected_count);
}

//after the contacts_to_actions kernel runs, we need to filter out actions that are invalid and then remove no-ops
void PandemicSim::daily_filterActions()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "filter_actions");

	if(debug_log_function_calls)
		debug_print("filtering contacts");


	//if victim is not susceptible to the virus, turn action to none
	thrust::transform(
		daily_action_type.begin(),
		daily_action_type.begin() + daily_contacts,
		thrust::make_zip_iterator(thrust::make_tuple(
			thrust::make_permutation_iterator(people_status_pandemic.begin(), daily_contact_victims.begin()),
			thrust::make_permutation_iterator(people_status_seasonal.begin(), daily_contact_victims.begin()))),
		daily_action_type.begin(),
		filterPriorInfectedOp());
	
	//remove no infection
	ZipIntQuadIterator filter_begin = thrust::make_zip_iterator(thrust::make_tuple(
		daily_action_type.begin(), daily_action_victim_index.begin(), 
		daily_action_victim_gen_p.begin(), daily_action_victim_gen_s.begin()));

	ZipIntQuadIterator filter_end = thrust::remove_if(
		filter_begin,
		thrust::make_zip_iterator(thrust::make_tuple(
			daily_action_type.begin() + daily_contacts, daily_action_victim_index.begin() + daily_contacts,
			daily_action_victim_gen_p.begin() + daily_contacts, daily_action_victim_gen_s.begin() + daily_contacts)),
		removeNoActionOp());

	int remaining = filter_end - filter_begin;

	if(CONSOLE_OUTPUT)
	{
		printf("%d unfiltered contacts\n", remaining);
	}

	//sort by victim, then by action
	thrust::sort(
		filter_begin,
		filter_end,
		actionSortOp());
	//remove duplicate actions
	filter_end = thrust::unique(	
		filter_begin,
		filter_end,
		uniqueActionOp());

	//after actions have been filtered, count how many remain
	daily_actions = filter_end - filter_begin;

	if(CONSOLE_OUTPUT)
	{
		printf("%d final contacts\n", daily_actions);
	}

	if(debug_log_function_calls)
		debug_print("contact filtering complete");
	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, daily_contacts);
}


__global__ void countReproduction_kernel(int * generation_offset_array, int MAX_GENERATION_NUM, int * output_generation_array)
{
	for(int myGen = blockIdx.x * blockDim.x + threadIdx.x;  myGen < MAX_GENERATION_NUM; myGen += gridDim.x * blockDim.x)
	{
		int generation_count = generation_offset_array[myGen + 1] - generation_offset_array[myGen];

		int stored_gen_count = output_generation_array[myGen];
		output_generation_array[myGen] = stored_gen_count + generation_count;
	}
}

//given an action type, look through the filtered actions and count generations
void PandemicSim::daily_countReproductionNumbers(int action)
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "count_reproduction");

	if(debug_log_function_calls)
		debug_print("counting reproduction");

	//tests action type
	infectionTypePred r_pred;
	r_pred.reference_val = action;


	//TODO: Fix
	//"pandemic" is a bad word for this - to do seasonal, pass it the seasonal action type
	d_vec pandemic_gens(daily_actions);
	IntIterator gens_end;
	if(action == ACTION_INFECT_PANDEMIC)
	{
		gens_end = thrust::copy_if(
			daily_action_victim_gen_p.begin(),			//copy all generations from pandemic actions
			daily_action_victim_gen_p.begin() + daily_actions,
			daily_action_type.begin(),
			pandemic_gens.begin(),
			r_pred);

	}
	else if(action == ACTION_INFECT_SEASONAL)
	{
		gens_end = thrust::copy_if(
			daily_action_victim_gen_s.begin(),		//copy all generations from seasonal actions 
			daily_action_victim_gen_s.begin() + daily_actions,
			daily_action_type.begin(),
			pandemic_gens.begin(),
			r_pred);
	}
	else
	{
		throw;
	}

	int num_matching_actions = gens_end - pandemic_gens.begin();

	//sort generations so we can count them
	thrust::sort(pandemic_gens.begin(), gens_end);


	if(action == ACTION_INFECT_PANDEMIC)
	{
		printf("Pandemic gens:\tcount: %d\n",num_matching_actions);
		debug_dump_array("pandemic_gen", &pandemic_gens, num_matching_actions);
	}
	else
	{
		printf("Seasonal gens:\tcount: %d\n", num_matching_actions);
		debug_dump_array("seasonal_gen", &pandemic_gens, num_matching_actions);
	}

	thrust::counting_iterator<int> count_iterator(0);

	d_vec lower_bound(MAX_DAYS + 1);
	thrust::lower_bound(
		pandemic_gens.begin(),
		gens_end,
		count_iterator,
		count_iterator + MAX_DAYS,
		lower_bound.begin());
	lower_bound[MAX_DAYS] = num_matching_actions;

	printf("lower bound: \n");
	debug_dump_array("offset",&lower_bound, 10);

	//note: this does not use the global threading settings, since we are searching a fixed block of generations
	int threadsPerBlock = 32;
	int numBlocks = (int) MAX_DAYS / threadsPerBlock;
	//increment the generation counts in the appropriate array
	if(action == ACTION_INFECT_PANDEMIC)
	{
		int * today_generations_ptr = thrust::raw_pointer_cast(pandemic_gens.data());
		int * pandemic_gens_ptr = thrust::raw_pointer_cast(generation_pandemic.data());
		countReproduction_kernel<<<numBlocks,threadsPerBlock>>>(today_generations_ptr, num_matching_actions, pandemic_gens_ptr);
		hipDeviceSynchronize();

		printf("after reproduction kernel, pandemic is:\n");
		debug_dump_array("val",&generation_pandemic, 10);
	}
	else if(action == ACTION_INFECT_SEASONAL)
	{
		int * today_generations_ptr = thrust::raw_pointer_cast(pandemic_gens.data());
		int * seasonal_gens_ptr = thrust::raw_pointer_cast(generation_seasonal.data());
		countReproduction_kernel<<<numBlocks,threadsPerBlock>>>(today_generations_ptr, num_matching_actions,seasonal_gens_ptr);
		hipDeviceSynchronize();

		printf("after reproduction kernel, seasonal is:\n");
		debug_dump_array("val",&generation_seasonal, 10);
	}
	else{
		throw; //action should be either seasonal or pandemic
	}

	if(debug_log_function_calls)
		debug_print("reproduction updated");
	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, daily_actions);
}

//this will dump all actions (unfiltered) to disk, which allows immediate debugging of the contacts_to_actions kernel
//If you need further detail, there are four float outputs which you can enable to debug kernels more easily
//for example, you can dump the y_vals that decide whether a contact is successful
//This uses a lot of disk space, so outputs are compressed with zlib.
//Dump the logs to disk using the 'zcat' utility
void PandemicSim::dump_actions(
//		thrust::device_vector<float> rand1, thrust::device_vector<float> rand2, thrust::device_vector<float> rand3, thrust::device_vector<float> rand4
		)
{
	//copy data to host - only as much data as we have contacts
	h_vec h_ac_type(daily_contacts);	//type of contact
	thrust::copy(daily_action_type.begin(), daily_action_type.begin() + daily_contacts, h_ac_type.begin());

	h_vec h_ac_inf(daily_contacts);		//infector
	thrust::copy(daily_contact_infectors.begin(), daily_contact_infectors.begin() + daily_contacts, h_ac_inf.begin());

	h_vec h_ac_vic(daily_contacts);		//victim
	thrust::copy(daily_action_victim_index.begin(), daily_action_victim_index.begin() + daily_contacts, h_ac_vic.begin());

	h_vec h_vic_gen_p(daily_contacts);	//victim gen_p
	thrust::copy(daily_action_victim_gen_p.begin(), daily_action_victim_gen_p.begin() + daily_contacts, h_vic_gen_p.begin());

	h_vec h_vic_gen_s = daily_action_victim_gen_s;	//victim gen_s
	thrust::copy(daily_action_victim_gen_s.begin(), daily_action_victim_gen_s.begin() + daily_contacts, h_vic_gen_s.begin());
	
	//copy whole status arrays
	h_vec h_status_p = people_status_pandemic;
	h_vec h_status_s = people_status_seasonal;
	
//	thrust::host_vector<float> h_r1 = rand1;
//	thrust::host_vector<float> h_r2 = rand2;
//	thrust::host_vector<float> h_r3 = rand3;
//	thrust::host_vector<float> h_r4 = rand4;
	
	for(int i = 0; i < daily_actions; i++)
	{
		int inf = h_ac_inf[i];
		int vic = h_ac_vic[i];
		int type = h_ac_type[i];
		
		int status_p = h_status_p[inf];
		int status_s = h_status_s[inf];

		int gen_p = h_vic_gen_p[i];
		int gen_s = h_vic_gen_s[i];
		
		/*
		float thresh_p = h_r1[i];
		float y_p = h_r3[i];
		bool infects_p = y_p < thresh_p;
		
		float thresh_s = h_r2[i];
		float y_s = h_r4[i];
		bool infects_s = y_s < thresh_s;*/
		
		//current_day, i, type, infector, infector_status_p, infector_status_s, victim, y_p, thresh_p, infects_p, y_s, thresh_s, infects_s
		if(log_actions)
			fprintf(fActions, "%d, %d, %s, %d, %c, %c, %d, %d, %d\n", // %f, %f, %d, %f, %f, %d\n",
					current_day, i, action_type_to_char(type),
					inf, status_int_to_char(status_p), status_int_to_char(status_s),
					vic, gen_p, gen_s);
//					y_p, thresh_p, infects_p, y_s, thresh_s, infects_s);
		
		if(print_actions)
			printf("%2d\tinf: %6d\tstatus_p: %c\tstatus_s: %c\tvic: %6d\ttype: %s\n",
				i,  inf, status_int_to_char(status_p), status_int_to_char(status_s), vic, action_type_to_char(type));
	}
	
	fflush(fActions);
	fflush(fDebug);
}


//this will dump the actions to disk after they have been filtered for dupes, invalid infections, etc
//in combination with dump_actions(), this allows you to debug the filtering code
//This uses a lot of disk space, so outputs are compressed with zlib.
//Dump the logs to disk using the 'zcat' utility
void PandemicSim::dump_actions_filtered()
{
	//copy data to host
	h_vec h_ac_type(daily_actions);
	thrust::copy_n(daily_action_type.begin(), daily_actions, h_ac_type.begin());
	h_vec h_ac_vic(daily_actions);
	thrust::copy_n(daily_action_victim_index.begin(), daily_actions, h_ac_vic.begin());
	h_vec h_vic_gen_p(daily_actions);
	thrust::copy_n(daily_action_victim_gen_p.begin(), daily_actions, h_vic_gen_p.begin());
	h_vec h_vic_gen_s(daily_actions);
	thrust::copy_n(daily_action_victim_gen_s.begin(), daily_actions, h_vic_gen_s.begin());

	h_vec h_status_p = people_status_pandemic;
	h_vec h_status_s = people_status_seasonal;

	for(int i = 0; i < daily_actions; i++)
	{
		int vic = h_ac_vic[i];
		int type = h_ac_type[i];
		int gen_p = h_vic_gen_p[i];
		int gen_s = h_vic_gen_s[i];


		int v_status_p = h_status_p[vic];
		int v_status_s = h_status_s[vic];

		//current_day, i, type, infector, infector_status_p, infector_status_s, victim, victim_status_p, victim_status_s, gen_p, gen_s
		if(log_actions_filtered)
			fprintf(fActionsFiltered, "%d, %d, %s, %d, %c, %d, %c, %d\n",
			current_day, i, action_type_to_char(type),
			vic, 
			status_int_to_char(v_status_p), gen_p, 
			status_int_to_char(v_status_s), gen_s);

		if(print_actions_filtered)
			printf("%2d\tvic: %6d\tstatus_p: %c\tstatus_s: %c\ttype: %s\n",
			i,  vic, status_int_to_char(v_status_p), status_int_to_char(v_status_s), action_type_to_char(type));
	}

	fflush(fActionsFiltered);
	fflush(fDebug);
}


//this function controls the update process - it recovers infected people 
//and rebuilds the array for the next day
void PandemicSim::daily_rebuildInfectedArray()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "rebuild_infected_arr");

	if(debug_log_function_calls)
		debug_print("rebuilding infected array");

	//recover infected who have reached culmination
	int remaining = daily_recoverInfected();

	//get set of non-infected IDs to add to the index
	//this is to avoid creating duplicate people each with one of the infections

	//get unique action victim IDS to ensure set operation is unique
	d_vec unique_ids(daily_actions);
	IntIterator unique_ids_end = thrust::unique_copy(
		daily_action_victim_index.begin(),
		daily_action_victim_index.begin() + daily_actions,
		unique_ids.begin());

	int num_unique_ids = unique_ids_end - unique_ids.begin();
	d_vec unique_new_ids(num_unique_ids);

	//find action victims where the victim is not already in the infected array
	IntIterator unique_new_ids_end = thrust::set_difference(
		unique_ids.begin(),
		unique_ids_end,
		infected_indexes.begin(),
		infected_indexes.begin() + remaining,
		unique_new_ids.begin());

	int new_infected = unique_new_ids_end - unique_new_ids.begin();
	int new_infected_total = remaining + new_infected;

	if(CONSOLE_OUTPUT)
	{
		printf("%d still infected, %d new, %d total\n", remaining, new_infected, new_infected_total);
	}


	//copy in new indexes and set default values - will be overwritten
	thrust::copy(unique_new_ids.begin(), unique_new_ids_end, infected_indexes.begin() + remaining);
	thrust::fill(infected_days_pandemic.begin() + remaining, infected_days_pandemic.begin() + new_infected_total, DAY_NOT_INFECTED);
	thrust::fill(infected_days_seasonal.begin() + remaining, infected_days_seasonal.begin() + new_infected_total, DAY_NOT_INFECTED);
	thrust::fill(infected_generation_pandemic.begin() + remaining, infected_generation_pandemic.begin() + new_infected_total, GENERATION_NOT_INFECTED);
	thrust::fill(infected_generation_seasonal.begin() + remaining, infected_generation_seasonal.begin() + new_infected_total, GENERATION_NOT_INFECTED);

	//sort the new infected into the array
	thrust::sort(
		thrust::make_zip_iterator(thrust::make_tuple(
		infected_indexes.begin(), 
		infected_days_pandemic.begin(), infected_days_seasonal.begin(), 
		infected_generation_pandemic.begin(), infected_generation_seasonal.begin())),
		thrust::make_zip_iterator(thrust::make_tuple(
		infected_indexes.begin() + new_infected_total,
		infected_days_pandemic.begin() + new_infected_total, infected_days_seasonal.begin() + new_infected_total,
		infected_generation_pandemic.begin() + new_infected_total, infected_generation_seasonal.begin() + new_infected_total)),
		FiveTuple_SortByFirst_Struct());

	//store new infected count
	infected_count = new_infected_total;

	//perform infection actions
	do_infection_actions(ACTION_INFECT_PANDEMIC);
	do_infection_actions(ACTION_INFECT_SEASONAL);

	if(debug_log_function_calls)
		debug_print("infected array rebuilt");

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, infected_count);
}


//This takes the filtered actions stored in the actions array, and executes them on the infected
//It assumes that all actions are valid
//Furthermore, it assumes that new infected have been copied into the array with null data
void PandemicSim::do_infection_actions(int action)
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "do_infection_actions");

	if(debug_log_function_calls)
		debug_print("processing infection actions");

	//binary search the victims we are applying the actions to
	vec_t victim_offsets(daily_actions);
	thrust::lower_bound(
		infected_indexes.begin(),
		infected_indexes.begin() + infected_count,
		daily_action_victim_index.begin(),
		daily_action_victim_index.begin() + daily_actions,
		victim_offsets.begin());

	//first, do the pandemic infections

	infectionTypePred typePred;
	typePred.reference_val = ACTION_INFECT_PANDEMIC;

	//convert victims status to infected
	thrust::replace_if(
		thrust::make_permutation_iterator(people_status_pandemic.begin(), daily_action_victim_index.begin()),
		thrust::make_permutation_iterator(people_status_pandemic.begin(), daily_action_victim_index.begin() + daily_actions),
		daily_action_type.begin(),
		typePred,
		STATUS_INFECTED);

	//copy their generation to the array
	thrust::scatter_if(
		daily_action_victim_gen_p.begin(),		//data
		daily_action_victim_gen_p.begin() + daily_actions,
		victim_offsets.begin(),			//map
		daily_action_type.begin(),		//stencil
		infected_generation_pandemic.begin(),		//output
		typePred);

	//mark tomorrow as their first day
	thrust::replace_if(
		thrust::make_permutation_iterator(infected_days_pandemic.begin(), victim_offsets.begin()),
		thrust::make_permutation_iterator(infected_days_pandemic.begin(), victim_offsets.end()),
		daily_action_type.begin(),
		typePred,
		current_day + 1);


	//do it again for seasonal
	typePred.reference_val = ACTION_INFECT_SEASONAL;
	thrust::replace_if(
		thrust::make_permutation_iterator(people_status_seasonal.begin(), daily_action_victim_index.begin()),
		thrust::make_permutation_iterator(people_status_seasonal.begin(), daily_action_victim_index.begin() + daily_actions),
		daily_action_type.begin(),
		typePred,
		STATUS_INFECTED);
	thrust::scatter_if(
		daily_action_victim_gen_s.begin(),
		daily_action_victim_gen_s.begin() + daily_actions,
		victim_offsets.begin(),
		daily_action_type.begin(),
		infected_generation_seasonal.begin(),
		typePred);
	thrust::replace_if(
		thrust::make_permutation_iterator(infected_days_seasonal.begin(), victim_offsets.begin()),
		thrust::make_permutation_iterator(infected_days_seasonal.begin(), victim_offsets.end()),
		daily_action_type.begin(),
		typePred,
		current_day + 1);

	if(debug_log_function_calls)
		debug_print("infection actions processed");

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, daily_actions);
}


//tihs function will recover infected people who have reached the culmination period
//should be called from rebuild_infected_arr()
int PandemicSim::daily_recoverInfected()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "recover_infected");

	if(debug_log_function_calls)
		debug_print("beginning recover_infected");

	recoverInfectedOp recoverOp;
	recoverOp.current_day = current_day;		//store current day in functor

	//recover pandemic strains
	thrust::transform(
		infected_days_pandemic.begin(),
		infected_days_pandemic.begin() + infected_count,
		thrust::make_zip_iterator(thrust::make_tuple(
			thrust::make_permutation_iterator(people_status_pandemic.begin(), infected_indexes.begin()),
			infected_days_pandemic.begin(), infected_generation_pandemic.begin())),			
		thrust::make_zip_iterator(thrust::make_tuple(
			thrust::make_permutation_iterator(people_status_pandemic.begin(), infected_indexes.begin()),
			infected_days_pandemic.begin(), infected_generation_pandemic.begin())),
		recoverOp);

	//recover seasonal strains
	thrust::transform(
		infected_days_seasonal.begin(),
		infected_days_seasonal.begin() + infected_count,
		thrust::make_zip_iterator(thrust::make_tuple(
			thrust::make_permutation_iterator(people_status_seasonal.begin(), infected_indexes.begin()),
			infected_days_seasonal.begin(), infected_generation_seasonal.begin())),			
		thrust::make_zip_iterator(thrust::make_tuple(
			thrust::make_permutation_iterator(people_status_seasonal.begin(), infected_indexes.begin()),
			infected_days_seasonal.begin(), infected_generation_seasonal.begin())),
		recoverOp);

	//remove people who are no longer infected
	d_vec infected_indexes_copy(infected_count);		//NOTE: not sure if this is necessary
	thrust::copy(infected_indexes.begin(), infected_indexes.begin() + infected_count, infected_indexes_copy.begin());	//TODO: check

	ZipIntFiveTupleIterator infected_begin = thrust::make_zip_iterator(thrust::make_tuple(
			infected_indexes.begin(), 
			infected_days_pandemic.begin(),	infected_days_seasonal.begin(), 
			infected_generation_pandemic.begin(), infected_generation_seasonal.begin()));
	ZipIntFiveTupleIterator infected_end = thrust::remove_if(
		infected_begin,
		thrust::make_zip_iterator(thrust::make_tuple(
			infected_indexes.begin() + infected_count, 
			infected_days_pandemic.begin() + infected_count, infected_days_seasonal.begin() + infected_count,
			infected_generation_pandemic.begin() + infected_count, infected_generation_seasonal.begin() + infected_count)),
		thrust::make_zip_iterator(thrust::make_tuple(
			thrust::make_permutation_iterator(people_status_pandemic.begin(), infected_indexes_copy.begin()),
			thrust::make_permutation_iterator(people_status_seasonal.begin(), infected_indexes_copy.begin()))),
		notInfectedPredicate());

	int infected_remaining = infected_end - infected_begin;

	if(debug_log_function_calls)
		debug_print("infected recovery complete");

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, infected_count);

	return infected_remaining;
}

//tests the household/workplace locations for the first 10 infected people
void PandemicSim::test_locs()
{
	h_vec h_o = household_offsets;
	h_vec h_c = household_counts;
	h_vec h_p = household_people;

	h_vec w_o = workplace_offsets;
	h_vec w_c = workplace_counts;
	h_vec w_p = workplace_people;

	h_vec i_i = infected_indexes;
	h_vec p_hh = people_households;
	h_vec p_wp = people_workplaces; 

	for(int i = 0; i < 10; i++)
	{
		int idx = i_i[i];
		int hh = p_hh[idx];		//lookup household for 
		int wp = p_wp[idx];
		int wp_contains = 0;
		int hh_contains = 0;

		for(int j = 0; j < w_c[wp]; j++)
		{
			int offset = w_o[wp] + j;
			if(w_p[offset] == idx)
			{
				wp_contains = 1;
				break;
			}
		}

		for(int j = 0; j < h_c[hh]; j++)
		{
			int offset = h_o[hh] + j;
			if(h_p[offset] == idx)
			{
				hh_contains = 1;
				break;
			}
		}
		printf("%4d:\tID: %d\tHH: %4d\tWP: %4d\tHH_contains: %4d\tWP_contains: %4d\n",i,idx,hh,wp, hh_contains, wp_contains);
	}
}

void PandemicSim::setup_configureLogging()
{

}

void PandemicSim::filterInfectedByPopulationGroup(const char * hour_string, vec_t * population_group, vec_t * infected_present)
{
	if(PROFILE_SIMULATION)
	{
		profiler.beginFunction(current_day,"filterInfectedByPopulationGroup");
	}

	IntIterator infected_present_end = thrust::set_intersection(
		(*population_group).begin(),
		(*population_group).end(),
		infected_indexes.begin(),
		infected_indexes.begin() + infected_count,
		(*infected_present).begin()
		);

	int infected_present_count  = infected_present_end - infected_present->begin();
	infected_present->resize(infected_present_count);

	if(SANITY_CHECK)
	{
		debug_assert(infected_present_count >= 0,"negative infected present count");
		debug_assert(infected_present_count < infected_count, "too many infected present");
	}

	if(CONSOLE_OUTPUT)
	{
		printf("%s\tinfected present: %d\n", hour_string, infected_present_count);
	}

	if(PROFILE_SIMULATION)
	{
		profiler.endFunction(current_day, infected_count);
	}
}

__global__ void clipContactsDesired_byLocationCount_kernel(int * infected_loc_arr, int * loc_offset_arr, int * contacts_desired_arr, int num_infected)
{
	for(int myPos = blockIdx.x * blockDim.x + threadIdx.x;  myPos < num_infected; myPos += gridDim.x * blockDim.x)
	{
		int myLoc = infected_loc_arr[myPos];		//location of infected

		int loc_count = loc_offset_arr[myLoc + 1] - loc_offset_arr[myLoc];

		if(loc_count == 1)
			contacts_desired_arr[myPos] = 0;
	}

}

void PandemicSim::clipContactsDesired_byLocationCount(d_ptr infected_locations_devPtr_begin, int num_infected, vec_t *loc_offsets, d_ptr contacts_desired_devPtr_begin)
{
	if(PROFILE_SIMULATION)
	{
		profiler.beginFunction(current_day,"buildContactsDesired_fromArray");
	}

	int * infected_loc_arr_ptr = thrust::raw_pointer_cast(infected_locations_devPtr_begin);
	int * loc_offset_arr_ptr = thrust::raw_pointer_cast((*loc_offsets).data());
	int * contacts_desired_arr_ptr = thrust::raw_pointer_cast(contacts_desired_devPtr_begin);

	clipContactsDesired_byLocationCount_kernel<<<cuda_blocks,cuda_threads>>>(infected_loc_arr_ptr, loc_offset_arr_ptr, contacts_desired_arr_ptr,num_infected);
	hipDeviceSynchronize();


	if(PROFILE_SIMULATION)
	{
		profiler.endFunction(current_day, num_infected);
	}
}

void PandemicSim::makeContacts_byContactsDesiredArray(
	const char * hour_string, 
	vec_t *infected_list, int infected_list_count, 
	vec_t *loc_people, vec_t *contacts_desired,
	vec_t *loc_offsets, int num_locs, 
	vec_t *people_lookup)
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "makeContacts_byContactsDesiredArray");

	//get the locations of infected people
	//ASSUMES: people_locations[i] contains the location of person index i (all people are present)
	vec_t infected_locations(infected_list_count);
	thrust::gather(
		(*infected_list).begin(),	//map.begin
		(*infected_list).begin() + infected_list_count,		//map.end
		(*people_lookup).begin(),
		infected_locations.begin());

	//get contacts desired for each infected person
	//this will simply clip contacts_desired to zero if they are alone at a location
	clipContactsDesired_byLocationCount(
		infected_locations.data(), infected_list_count,
		loc_offsets, contacts_desired->data());

	//get total number of contacts this hour 
	int num_new_contacts = thrust::reduce((*contacts_desired).begin(), (*contacts_desired).begin() + infected_list_count);

	//get raw pointer into the location table
	int * location_people_ptr = thrust::raw_pointer_cast((*loc_people).data());

	//make contacts
	launchContactsKernel(
		hour_string,
		infected_list, &infected_locations, 
		contacts_desired, infected_list_count,
		location_people_ptr, loc_offsets, num_locs);

	//validate the contacts
	if(log_contacts || print_contacts)
	{
		validate_contacts(hour_string, loc_people, people_lookup, loc_offsets, num_new_contacts);
	}

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, infected_count);
}



__global__ void weekendErrand_getInfectedHoursAndDestinations_kernel(int * input_infected_indexes_ptr, int * input_errand_hours_ptr, int * input_errand_destinations_ptr,
																	 int * output_infected_present_ptr, int * output_infected_hour_ptr, 
																	 int * output_infected_dest_ptr, int * output_contacts_desired_ptr,
																	 int num_infected, int num_people, int rand_offset)
{
	threefry2x32_key_t tf_k = {{SEED_DEVICE[0], SEED_DEVICE[1]}};
	union{
		threefry2x32_ctr_t c;
		unsigned int i[2];
	} u;

	//for each infected index i, their first errand destination and hour will be at i, their second at i+N, the third at i+N+N
	//where N = num_people
	for(int myPos = blockIdx.x * blockDim.x + threadIdx.x;  myPos < num_infected; myPos += gridDim.x * blockDim.x)
	{
		//assign 2 contacts between 3 hours randomly
		threefry2x32_ctr_t tf_ctr = {{myPos + rand_offset, myPos + rand_offset }};
		u.c = threefry2x32(tf_ctr, tf_k);
		int contacts_assignment = u.i[0] % 6;		//parse uint to a number between 0 <= X < 6

		int myIdx = input_infected_indexes_ptr[myPos];

		int myInputOffset = myIdx;
		int output_offset = myPos * NUM_WEEKEND_ERRANDS;

		output_infected_present_ptr[output_offset] = myIdx;									//copy infected index
		output_infected_hour_ptr[output_offset] = input_errand_hours_ptr[myInputOffset];	//copy the hour of the errand
		output_infected_dest_ptr[output_offset] = input_errand_destinations_ptr[myInputOffset];			//copy the errand destination
		output_contacts_desired_ptr[output_offset] = weekend_errand_contact_assignments[contacts_assignment][0];	//assign contacts to this errand

		myInputOffset += num_people;	//move input slot by num_people
		output_offset++;				//move output slot over by one
		output_infected_present_ptr[output_offset] = myIdx;
		output_infected_hour_ptr[output_offset] = input_errand_hours_ptr[myInputOffset];
		output_infected_dest_ptr[output_offset] = input_errand_destinations_ptr[myInputOffset];
		output_contacts_desired_ptr[output_offset] = weekend_errand_contact_assignments[contacts_assignment][1];

		myInputOffset += num_people;
		output_offset++;
		output_infected_present_ptr[output_offset] = myIdx;
		output_infected_hour_ptr[output_offset] = input_errand_hours_ptr[myInputOffset];
		output_infected_dest_ptr[output_offset] = input_errand_destinations_ptr[myInputOffset];
		output_contacts_desired_ptr[output_offset] = weekend_errand_contact_assignments[contacts_assignment][2];
	}
}

void PandemicSim::weekendErrand_doInfectedSetup(vec_t * errand_hours, vec_t * errand_destinations, vec_t * infected_present, vec_t * infected_locations, vec_t * infected_contacts_desired, vec_t * infected_hour_offsets)
{
	if(SANITY_CHECK)
	{
		int all_errands_count = number_people * NUM_WEEKEND_ERRANDS;
		debug_assert("input errand_hours.size is wrong", all_errands_count, errand_hours->size());
		debug_assert("input errand_destinations.size is wrong", all_errands_count, errand_destinations->size());

		int infected_errands_count = infected_count * NUM_WEEKEND_ERRANDS;
		debug_assert("output infected_present.size is wrong", infected_errands_count,infected_present->size());
		debug_assert("output infected_locations.size is wrong", infected_errands_count,infected_locations->size());
		debug_assert("output contacts_desired.size is wrong", infected_errands_count, infected_contacts_desired->size());

		debug_assert("output infected_hour_offsets.size is wrong", NUM_WEEKEND_ERRAND_HOURS + 1, infected_hour_offsets->size());
	}

	//first input: a list of all infected
	int * global_infected_indexes_ptr = thrust::raw_pointer_cast(infected_indexes.data());

	//second inputs: collated lookup tables for hours and destinations
	int * errand_hour_ptr = thrust::raw_pointer_cast((*errand_hours).data());
	int * errand_dest_ptr = thrust::raw_pointer_cast((*errand_destinations).data());

	//outputs: a list of infected indexes, the hour of the errands, the destinations
	int * infected_present_ptr = thrust::raw_pointer_cast(infected_present->data());
	int * infected_destinations_ptr = thrust::raw_pointer_cast(infected_locations->data());
	int * infected_contacts_desired_ptr = thrust::raw_pointer_cast(infected_contacts_desired->data());

	//use temporary array to store the infected hours - we don't need to pass this back, as the data is conveyed in the offset array
	vec_t infected_hour(infected_count * NUM_WEEKEND_ERRANDS);
	int * infected_hour_ptr = thrust::raw_pointer_cast(infected_hour.data());

	weekendErrand_getInfectedHoursAndDestinations_kernel<<<cuda_blocks,cuda_threads>>>(
		global_infected_indexes_ptr,errand_hour_ptr,errand_dest_ptr,
		infected_present_ptr, infected_hour_ptr, 
		infected_destinations_ptr, infected_contacts_desired_ptr,
		infected_count, number_people, rand_offset);
	hipDeviceSynchronize();
	rand_offset += infected_count;

	//sort the list of infected indexes present and the destinations by hour
	//since the array is built pre-sorted by infected index, this will result in
	//an array sorted by hour, with each hour additionally sorted by infected index
	int infected_errand_count = infected_count * NUM_WEEKEND_ERRANDS;
	thrust::stable_sort_by_key(
		infected_hour.begin(),
		infected_hour.begin() + infected_errand_count,
		thrust::make_zip_iterator(thrust::make_tuple(
			infected_present->begin(), infected_locations->begin(), infected_contacts_desired->begin())));

	//count the number of infected in each hour	
	thrust::lower_bound(
		infected_hour.begin(),		//data.first
		infected_hour.begin() + infected_errand_count,			//data.last
		thrust::counting_iterator<int>(0),		//search val first
		thrust::counting_iterator<int>(NUM_WEEKEND_ERRAND_HOURS), //search val last
		infected_hour_offsets->begin());
	(*infected_hour_offsets)[NUM_WEEKEND_ERRAND_HOURS] = infected_count * NUM_WEEKEND_ERRANDS;

	if(SANITY_CHECK)
	{
		bool hours_sorted = thrust::is_sorted(infected_hour.begin(), infected_hour.begin() + infected_errand_count);
		debug_assert(hours_sorted, "infected hours did not sort properly!");

		for(int hour = 0; hour < NUM_WEEKEND_ERRAND_HOURS; hour++)
		{
			int offset = (*infected_hour_offsets)[hour];
			int count = (*infected_hour_offsets)[hour+1] - offset;

			debug_assert(count > 0, "warning: zero infected found in weekend errand hour ", hour);
			debug_assert(count < infected_count, "more than infected_count infected found in hour ", hour);

			//check that the infected indexes are still sorted for each hour
			bool infected_are_sorted =thrust::is_sorted(infected_present->begin() + offset, infected_present->begin() + offset + count);
			debug_assert(infected_are_sorted, "infected not sorted for weekend errand hour ", hour);
		}
	}
}

//helper method that will adapt older code to the IntIterator pointers
void PandemicSim::launchContactsKernel(const char * hour_string, vec_t *infected_indexes_present, vec_t *infected_locations, vec_t *infected_contacts_desired, int infected_present, int * loc_people_ptr, vec_t *location_offsets, int num_locs)
{
	if(SANITY_CHECK)
	{
		if((*location_offsets).size() != num_locs + 1)
		{
			printf("WARNING: Need to convert old-style loc_offset in function %c\n",profiler.getCurrentFuncName());
			exit(1);
		}

		//theoretically, the size of the list of infected people present should equal the parameter
		//If infected_preent < size(), this will be OK - it will just select the first N people
		//however, it really implies that something else went bad along the way
	//	debug_assert("infected present size mismatch", infected_indexes_present->size(), infected_present);
	}

	launchContactsKernel(
		hour_string,
		infected_indexes_present->data(), infected_locations->data(), 
		infected_contacts_desired->data(), infected_present, 
		loc_people_ptr, location_offsets, num_locs);
}

//will resize the infected, contact, and action arrays to fit the entire population
void PandemicSim::setup_sizeGlobalArrays()
{
	//assume that at peak, about 3/4 of people will be affected
//	float peak_infection_ratio = 0.75f;
//	int expected_max_infection = (int) peak_infection_ratio * number_people;

	//setup people status:
	people_status_pandemic.resize(number_people);
	people_status_seasonal.resize(number_people);
	thrust::fill(people_status_pandemic.begin(), people_status_pandemic.end(), STATUS_SUSCEPTIBLE);
	thrust::fill(people_status_seasonal.begin(), people_status_seasonal.end(), STATUS_SUSCEPTIBLE);

	//setup output reproduction number counters
	generation_pandemic.resize(MAX_DAYS);
	generation_seasonal.resize(MAX_DAYS);
	thrust::fill(generation_pandemic.begin(), generation_pandemic.end(), 0);
	thrust::fill(generation_seasonal.begin(), generation_seasonal.end(), 0);

	//assume that worst-case everyone gets infected
	infected_indexes.resize(number_people);
	infected_days_pandemic.resize(number_people);
	infected_days_seasonal.resize(number_people);
	infected_generation_pandemic.resize(number_people);
	infected_generation_seasonal.resize(number_people);

	int expected_max_contacts = number_people * MAX_CONTACTS_PER_DAY;

	//resize contact arrays
	daily_contact_infectors.resize(expected_max_contacts);
	daily_contact_victims.resize(expected_max_contacts);
	daily_contact_kvals.resize(expected_max_contacts);

	//resize action arrays
	daily_action_type.resize(expected_max_contacts);
	daily_action_victim_index.resize(expected_max_contacts);
	daily_action_victim_gen_p.resize(expected_max_contacts);
	daily_action_victim_gen_s.resize(expected_max_contacts);

	//weekend errands arrays tend to be very large, so pre-allocate them
	int num_weekend_errands = number_people * NUM_WEEKEND_ERRANDS;
	weekend_errand_people.resize(num_weekend_errands);
	weekend_errand_hours.resize(num_weekend_errands);
	weekend_errand_destinations.resize(num_weekend_errands);

	weekend_infectedPresentIndexes.resize(num_weekend_errands);
	weekend_infectedLocations.resize(num_weekend_errands);
	weekend_infectedHours.resize(num_weekend_errands);
	weekend_infectedContactsDesired.resize(num_weekend_errands);

	if(dump_contact_kernel_random_data)
	{
		debug_float1.resize(expected_max_contacts);
		debug_float2.resize(expected_max_contacts);
		debug_float3.resize(expected_max_contacts);
		debug_float4.resize(expected_max_contacts);
	}
}

//helper method to set up validation for sparse location arrays where not all people are present, eg weekend errands
void PandemicSim::validate_weekend_errand_contacts(const char * hour_string, d_ptr people_begin, d_ptr destinations_begin, int people_present, d_ptr loc_offsets_ptr, int number_locations, int num_contacts_to_dump)
{
	h_vec h_people;
	h_vec h_offsets;

	//we only need these if we're actually verifying the people are inside the table (slow)
	if(DOUBLECHECK_CONTACTS)
	{
		h_people.resize(people_present);
		thrust::copy_n(people_begin, people_present, h_people.begin());

		h_offsets.resize(number_locations + 1);
		thrust::copy_n(loc_offsets_ptr, number_locations + 1, h_offsets.begin());
	}

	//sort this segment of the table by people ID
	thrust::sort_by_key(
		people_begin,
		people_begin + people_present,
		destinations_begin);

	//copy the people and destinations to the host
	h_vec hour_people(people_present);
	thrust::copy_n(people_begin, people_present, hour_people.begin());
	h_vec hour_destinations(people_present);
	thrust::copy_n(destinations_begin, people_present, hour_destinations.begin());

	//scatter into a fake lookup table
	h_vec big_lookup(number_people);
	thrust::fill(big_lookup.begin(), big_lookup.end(), -1);
	thrust::scatter(
		hour_destinations.begin(),
		hour_destinations.end(),
		hour_people.begin(),
		big_lookup.begin());

	int start_offset = daily_contacts - num_contacts_to_dump;
	h_vec contact_infectors(num_contacts_to_dump);
	thrust::copy_n(daily_contact_infectors.begin() + start_offset, num_contacts_to_dump, contact_infectors.begin());
	h_vec contact_victims(num_contacts_to_dump);
	thrust::copy_n(daily_contact_victims.begin() + start_offset, num_contacts_to_dump, contact_victims.begin());

	//call validation method
	validate_contacts(hour_string, &h_people, &big_lookup, &h_offsets, &contact_infectors, &contact_victims,num_contacts_to_dump);
}

void PandemicSim::debug_dump_array(const char * description, d_vec * gens_array, int array_count)
{
	h_vec host_array(array_count);
	thrust::copy_n(gens_array->begin(), array_count, host_array.begin());

	for(int i = 0; i < array_count; i++)
	{
		printf("%3d\t%s: %d\n",i,description, host_array[i]);
	}
}


inline void debug_print(char * message)
{
	fprintf(fDebug, "%s\n", message);
	fflush(fDebug);
} 



inline void debug_assert(bool condition, char * message)
{
	if(!condition)
	{
		fprintf(fDebug, "ERROR: ");
		debug_print(message);
	}
}

inline void debug_assert(char *message, int expected, int actual)
{
	if(expected != actual)
	{
		fprintf(fDebug, "ERROR: %s expected: %d actual: %d\n", message, expected, actual);
		fflush(fDebug);
	}
}

inline void debug_assert(bool condition, char * message, int idx)
{
	if(!condition)
	{
		fprintf(fDebug, "ERROR: %s index: %d\n", message, idx);
	}
}

inline char status_int_to_char(int s)
{
	switch(s)
	{
	case STATUS_SUSCEPTIBLE:
		return 'S';
	case STATUS_INFECTED:
		return 'I';
	case STATUS_RECOVERED:
		return 'R';
	default:
		return '?';
	}
}

inline char * action_type_to_char(int action)
{
	switch(action)
	{
	case ACTION_INFECT_NONE:
		return "NONE";
	case ACTION_INFECT_PANDEMIC:
		return "PAND";
	case ACTION_INFECT_SEASONAL:
		return "SEAS";
	case ACTION_INFECT_BOTH:
		return "BOTH";
	default:
		return "????";
	}
}

struct weekend_getter : public thrust::unary_function<int, float>
{
	__device__ float operator() (const int& i)
	{
		return weekend_errand_pdf[i];
	}
};

//generates N unique numbers between 0 and max, exclusive
//assumes array is big enough that this won't be pathological
void n_unique_numbers(h_vec *array, int n, int max)
{
	for(int i = 0; i < n; i++)
	{
		do
		{
			(*array)[i] = rand() % max;
			for(int j =0; j < i; j++)
			{
				if((*array)[j] == (*array)[i])
				{
					(*array)[i] = -1;
					break;
				}
			}
		}while((*array)[i] == -1);
	}
}

