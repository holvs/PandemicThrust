#include "hip/hip_runtime.h"
#include "stdafx.h"

#include "simParameters.h"
#include "profiler.h"

#include "PandemicSim.h"
#include "thrust_functors.h"

#include <thrust/iterator/transform_iterator.h>
#include <thrust/scan.h>

//output status messages to console?  Slows things down

//Simulation profiling master control - low performance overhead
const int PROFILE_SIMULATION = 1;


int cuda_blocks = 32;
int cuda_threads = 256;


FILE * f_outputInfectedStats;

FILE * fDebug;

__device__ __constant__ int SEED_DEVICE[SEED_LENGTH];
int SEED_HOST[SEED_LENGTH];

__device__ __constant__ float WORKPLACE_TYPE_WEEKDAY_ERRAND_PDF_DEVICE[NUM_BUSINESS_TYPES];				//stores PDF for weekday errand destinations
float WORKPLACE_TYPE_WEEKDAY_ERRAND_PDF_HOST[NUM_BUSINESS_TYPES];
__device__ __constant__ float WORKPLACE_TYPE_WEEKEND_ERRAND_PDF_DEVICE[NUM_BUSINESS_TYPES];				//stores PDF for weekend errand destinations
float WORKPLACE_TYPE_WEEKEND_ERRAND_PDF_HOST[NUM_BUSINESS_TYPES];


__device__ __constant__ int WEEKEND_ERRAND_CONTACT_ASSIGNMENTS_DEVICE[6][2];
int WEEKEND_ERRAND_CONTACT_ASSIGNMENTS_HOST[6][2];
__device__ __constant__ int WEEKDAY_ERRAND_CONTACT_ASSIGNMENTS_DEVICE[3][2];
int WEEKDAY_ERRAND_CONTACT_ASSIGNMENT_HOST[3][2];

#define STRAIN_COUNT 2
#define STRAIN_PANDEMIC 0
#define STRAIN_SEASONAL 1
//__device__ __constant__ float BASE_REPRODUCTION_DEVICE[STRAIN_COUNT];
float BASE_REPRODUCTION_HOST[STRAIN_COUNT];

#define BASE_R_PANDEMIC_HOST BASE_REPRODUCTION_HOST[0]
#define BASE_R_SEASONAL_HOST BASE_REPRODUCTION_HOST[1]


__device__ __constant__ float INFECTIOUSNESS_FACTOR_DEVICE[STRAIN_COUNT];
float INFECTIOUSNESS_FACTOR_HOST[STRAIN_COUNT];

__device__ __constant__ float PERCENT_SYMPTOMATIC_DEVICE;
float PERCENT_SYMPTOMATIC_HOST;

__device__ __constant__ kval_t KVAL_LOOKUP_DEVICE[NUM_CONTACT_TYPES];
kval_t KVAL_LOOKUP_HOST[NUM_CONTACT_TYPES];

#define UNSIGNED_MAX (unsigned int) -1

float WORKPLACE_TYPE_ASSIGNMENT_PDF_HOST[NUM_BUSINESS_TYPES];
__device__ float WORKPLACE_TYPE_ASSIGNMENT_PDF_DEVICE[NUM_BUSINESS_TYPES];

int WORKPLACE_TYPE_OFFSET_HOST[NUM_BUSINESS_TYPES];
__device__ __constant__ int WORKPLACE_TYPE_OFFSET_DEVICE[NUM_BUSINESS_TYPES];			//stores location number of first business of this type
int WORKPLACE_TYPE_COUNT_HOST[NUM_BUSINESS_TYPES];
__device__ __constant__ int WORKPLACE_TYPE_COUNT_DEVICE[NUM_BUSINESS_TYPES];				//stores number of each type of business
int WORKPLACE_TYPE_MAX_CONTACTS_HOST[NUM_BUSINESS_TYPES];
__device__ __constant__ int WORKPLACE_TYPE_MAX_CONTACTS_DEVICE[NUM_BUSINESS_TYPES];


__device__ __constant__ float VIRAL_SHEDDING_PROFILES_DEVICE[NUM_PROFILES][CULMINATION_PERIOD];
float VIRAL_SHEDDING_PROFILES_HOST[NUM_PROFILES][CULMINATION_PERIOD];


float CHILD_AGE_CDF_HOST[CHILD_DATA_ROWS];
__device__ float CHILD_AGE_CDF_DEVICE[CHILD_DATA_ROWS];
int CHILD_AGE_SCHOOLTYPE_LOOKUP_HOST[CHILD_DATA_ROWS];
__device__ int CHILD_AGE_SCHOOLTYPE_LOOKUP_DEVICE[CHILD_DATA_ROWS];

float HOUSEHOLD_TYPE_CDF_HOST[HH_TABLE_ROWS];
__device__ __constant__ float HOUSEHOLD_TYPE_CDF_DEVICE[HH_TABLE_ROWS];
int HOUSEHOLD_TYPE_ADULT_COUNT_HOST[HH_TABLE_ROWS];
__device__ __constant__ int HOUSEHOLD_TYPE_ADULT_COUNT_DEVICE[HH_TABLE_ROWS];
int HOUSEHOLD_TYPE_CHILD_COUNT_HOST[HH_TABLE_ROWS];
__device__ __constant__ int HOUSEHOLD_TYPE_CHILD_COUNT_DEVICE[HH_TABLE_ROWS];



//the first row of the PDF with a value > 0
const int FIRST_WEEKDAY_ERRAND_ROW = 9;
const int FIRST_WEEKEND_ERRAND_ROW = 9;


PandemicSim::PandemicSim() 
{
	logging_openOutputStreams();

	if(PROFILE_SIMULATION)
		profiler.initStack();

	hipStreamCreate(&stream_secondary);

	setup_loadParameters();
	setup_scaleSimulation();
	setup_calculateInfectionData();

	//copy everything down to the GPU
	setup_pushDeviceData();

	if(TIMING_BATCH_MODE == 0)
	{
		setup_setCudaTopology();
	}

	if(debug_log_function_calls)
		debug_print("parameters loaded");

}


PandemicSim::~PandemicSim(void)
{
	hipStreamDestroy(stream_secondary);

	if(PROFILE_SIMULATION)
		profiler.done();
	logging_closeOutputStreams();
}

void PandemicSim::setupSim()
{
	if(PROFILE_SIMULATION)
	{
		profiler.beginFunction(-1,"setupSim");
	}

	//moved to constructor for batching
	//	open_debug_streams();
	//	setupLoadParameters();

	rand_offset = 0;				//set global rand counter to 0

	current_day = -1;
	
	if(debug_log_function_calls)
		debug_print("setting up households");
	
	//setup households
	setup_generateHouseholds();	//generates according to PDFs

	if(CONSOLE_OUTPUT)
		printf("%d people, %d households, %d workplaces\n",number_people, number_households, number_workplaces);

	setup_buildFixedLocations();	//household and workplace
	setup_initialInfected();

	if(SIM_VALIDATION)
	{
		hipDeviceSynchronize();

		debug_sizeHostArrays();
		debug_copyFixedData();
		debug_validatePeopleSetup();
	}

	if(POLL_MEMORY_USAGE)
		logging_pollMemoryUsage_takeSample(current_day);

	if(PROFILE_SIMULATION)
	{
		profiler.endFunction(-1, number_people);
	}

	if(debug_log_function_calls)
		debug_print("simulation setup complete");
}


void PandemicSim::logging_openOutputStreams()
{
	if(log_infected_info)
	{
		if(OUTPUT_FILES_IN_PARENTDIR)
			fInfected = fopen("../debug_infected.csv", "w");
		else
			fInfected = fopen("debug_infected.csv", "w");

		fprintf(fInfected, "current_day, i, idx, status_p, day_p, gen_p, status_s, day_s, gen_s\n");
	}

/*	if(log_location_info)
	{
		fLocationInfo = fopen("../debug_location_info.csv","w");
		fprintf(fLocationInfo, "current_day, hour_index, i, offset, count, max_contacts\n");
	}*/

	if(log_contacts)
	{
		if(OUTPUT_FILES_IN_PARENTDIR)
			fContacts = fopen("../debug_contacts.csv", "w");
		else
			fContacts = fopen("debug_contacts.csv", "w");
		
		fprintf(fContacts, "current_day, i, infector_idx, victim_idx, contact_type, infector_loc, victim_loc, locs_matched\n");
	}


	if(log_actions)
	{
		if(OUTPUT_FILES_IN_PARENTDIR)
			fActions = fopen("../debug_actions.csv", "w");
		else
			fActions = fopen("debug_actions.csv", "w");
		fprintf(fActions, "current_day, i, infector, victim, action_type, action_type_string\n");
	}

	if(log_actions_filtered)
	{
		if(OUTPUT_FILES_IN_PARENTDIR)
			fActionsFiltered = fopen("../debug_filtered_actions.csv", "w");
		else
			fActionsFiltered = fopen("debug_filtered_actions.csv", "w");
		fprintf(fActionsFiltered, "current_day, i, type, victim, victim_status_p, victim_gen_p, victim_status_s, victim_gen_s\n");
	}
	

	if(SIM_VALIDATION || debug_log_function_calls)
	{
		if(OUTPUT_FILES_IN_PARENTDIR)
			fDebug = fopen("../debug.txt", "w");
		else
			fDebug = fopen("debug.txt", "w");
	}

	if(OUTPUT_FILES_IN_PARENTDIR)
		f_outputInfectedStats=fopen("../output_infected_stats.csv","w");
	else
		f_outputInfectedStats=fopen("output_infected_stats.csv","w");
	fprintf(f_outputInfectedStats, "day,pandemic_susceptible,pandemic_infectious,pandemic_symptomatic,pandemic_asymptomatic,pandemic_recovered,seasonal_susceptible,seasonal_infectious,seasonal_symptomatic,seasonal_asymptomatic,seasonal_recovered\n");

}

void PandemicSim::setup_loadParameters()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(-1,"setup_loadParameters");

	setup_loadSeed();

	//if printing seeds is desired for debug, etc
	if(1)
	{
		printf("seeds:\t");
		for(int i = 0; i < SEED_LENGTH; i++)
			if(i < SEED_LENGTH - 1)
				printf("%d\t",SEED_HOST[i]);
			else
				printf("%d\n",SEED_HOST[i]);
	}

	//read constants file 
	FILE * fConstants = fopen("constants.csv","r");	//open file
	if(fConstants == NULL)
	{
		debug_print("failed to open constants file");
		perror("Error opening constants file");
		exit(1);
	}

	//get a line buffer
#define LINEBUFF_SIZE 512
	char line[LINEBUFF_SIZE];	

	fgets(line, LINEBUFF_SIZE, fConstants);	//read the first line into the buffer to skip it
	fscanf(fConstants,"%*[^,]%*c");	//skip the first column of the table
	fscanf(fConstants, "%d%*c", &MAX_DAYS);
	fscanf(fConstants, "%f%*c", &BASE_R_PANDEMIC_HOST);
	fscanf(fConstants, "%f%*c", &BASE_R_SEASONAL_HOST);
	fscanf(fConstants, "%d%*c", &INITIAL_INFECTED_PANDEMIC);
	fscanf(fConstants, "%d%*c", &INITIAL_INFECTED_SEASONAL);
	fscanf(fConstants, "%f%*c", &sim_scaling_factor);
	fscanf(fConstants, "%f%*c", &PERCENT_SYMPTOMATIC_HOST);
	fscanf(fConstants, "%f", &asymp_factor);
	fclose(fConstants);

	number_households = 1000000;
	number_workplaces = 12800;

	if(CONSOLE_OUTPUT)
		printf("max days: %d\nr_p: %f\nr_s: %f\ninitial_pandemic: %d\ninitial_seasonal: %d\nnumber_households: %d\n",
		MAX_DAYS,
		BASE_R_PANDEMIC_HOST,
		BASE_R_SEASONAL_HOST,
		INITIAL_INFECTED_PANDEMIC,
		INITIAL_INFECTED_SEASONAL,
		number_households);

	//read other parameter sets
	//hard coded these for time currently since we have no other sets

	//cdf for child age
	CHILD_AGE_CDF_HOST[0] = 0.24f;
	CHILD_AGE_CDF_HOST[1] = 0.47f;
	CHILD_AGE_CDF_HOST[2] = 0.72f;
	CHILD_AGE_CDF_HOST[3] = 0.85f;
	CHILD_AGE_CDF_HOST[4] = 1.0f;

	//what workplace type children get for this age
	CHILD_AGE_SCHOOLTYPE_LOOKUP_HOST[0] = 3;
	CHILD_AGE_SCHOOLTYPE_LOOKUP_HOST[1] = 4;
	CHILD_AGE_SCHOOLTYPE_LOOKUP_HOST[2] = 5;
	CHILD_AGE_SCHOOLTYPE_LOOKUP_HOST[3] = 6;
	CHILD_AGE_SCHOOLTYPE_LOOKUP_HOST[4] = 7;

	//workplace PDF for adults
	WORKPLACE_TYPE_ASSIGNMENT_PDF_HOST[0] = 0.06586f;
	WORKPLACE_TYPE_ASSIGNMENT_PDF_HOST[1] = 0.05802f;
	WORKPLACE_TYPE_ASSIGNMENT_PDF_HOST[2] = 0.30227f;
	WORKPLACE_TYPE_ASSIGNMENT_PDF_HOST[3] = 0.0048f;
	WORKPLACE_TYPE_ASSIGNMENT_PDF_HOST[4] = 0.00997f;
	WORKPLACE_TYPE_ASSIGNMENT_PDF_HOST[5] = 0.203f;
	WORKPLACE_TYPE_ASSIGNMENT_PDF_HOST[6] = 0.09736f;
	WORKPLACE_TYPE_ASSIGNMENT_PDF_HOST[7] = 0.10598f;
	WORKPLACE_TYPE_ASSIGNMENT_PDF_HOST[8] = 0.00681f;
	WORKPLACE_TYPE_ASSIGNMENT_PDF_HOST[9] = 0.02599f;
	WORKPLACE_TYPE_ASSIGNMENT_PDF_HOST[10] = 0.f;
	WORKPLACE_TYPE_ASSIGNMENT_PDF_HOST[11] = 0.08749f;
	WORKPLACE_TYPE_ASSIGNMENT_PDF_HOST[12] = 0.03181f;
	WORKPLACE_TYPE_ASSIGNMENT_PDF_HOST[13] = 0.00064f;

	//number of each type of workplace
	WORKPLACE_TYPE_COUNT_HOST[0] = 1000;
	WORKPLACE_TYPE_COUNT_HOST[1] = 7000;
	WORKPLACE_TYPE_COUNT_HOST[2] = 2400;
	WORKPLACE_TYPE_COUNT_HOST[3] = 300;
	WORKPLACE_TYPE_COUNT_HOST[4] = 100;
	WORKPLACE_TYPE_COUNT_HOST[5] = 200;
	WORKPLACE_TYPE_COUNT_HOST[6] = 100;
	WORKPLACE_TYPE_COUNT_HOST[7] = 100;
	WORKPLACE_TYPE_COUNT_HOST[8] = 300;
	WORKPLACE_TYPE_COUNT_HOST[9] = 500;
	WORKPLACE_TYPE_COUNT_HOST[10] = 0;
	WORKPLACE_TYPE_COUNT_HOST[11] = 300;
	WORKPLACE_TYPE_COUNT_HOST[12] = 400;
	WORKPLACE_TYPE_COUNT_HOST[13] = 100;

	//maximum number of contacts made at each workplace type
	WORKPLACE_TYPE_MAX_CONTACTS_HOST[0] = 3;
	WORKPLACE_TYPE_MAX_CONTACTS_HOST[1] = 3;
	WORKPLACE_TYPE_MAX_CONTACTS_HOST[2] = 3;
	WORKPLACE_TYPE_MAX_CONTACTS_HOST[3] = 2;
	WORKPLACE_TYPE_MAX_CONTACTS_HOST[4] = 2;
	WORKPLACE_TYPE_MAX_CONTACTS_HOST[5] = 3;
	WORKPLACE_TYPE_MAX_CONTACTS_HOST[6] = 3;
	WORKPLACE_TYPE_MAX_CONTACTS_HOST[7] = 2;
	WORKPLACE_TYPE_MAX_CONTACTS_HOST[8] = 2;
	WORKPLACE_TYPE_MAX_CONTACTS_HOST[9] = 2;
	WORKPLACE_TYPE_MAX_CONTACTS_HOST[10] = 0;
	WORKPLACE_TYPE_MAX_CONTACTS_HOST[11] = 2;
	WORKPLACE_TYPE_MAX_CONTACTS_HOST[12] = 2;
	WORKPLACE_TYPE_MAX_CONTACTS_HOST[13] = 2;

	//pdf for weekday errand location generation
	//most entries are 0.0
	for(int type = 0; type < NUM_BUSINESS_TYPES; type++)
		WORKPLACE_TYPE_WEEKDAY_ERRAND_PDF_HOST[type] = 0.0f;
	WORKPLACE_TYPE_WEEKDAY_ERRAND_PDF_HOST[9] = 0.61919f;
	WORKPLACE_TYPE_WEEKDAY_ERRAND_PDF_HOST[11] = 0.27812f;
	WORKPLACE_TYPE_WEEKDAY_ERRAND_PDF_HOST[12] = 0.06601f;
	WORKPLACE_TYPE_WEEKDAY_ERRAND_PDF_HOST[13] = 0.03668f;

	//pdf for weekend errand location generation
	//most entries are 0.0
	for(int type = 0; type < NUM_BUSINESS_TYPES; type++)
		WORKPLACE_TYPE_WEEKEND_ERRAND_PDF_HOST[type] = 0.0f;
	WORKPLACE_TYPE_WEEKEND_ERRAND_PDF_HOST[9] = 0.51493f;
	WORKPLACE_TYPE_WEEKEND_ERRAND_PDF_HOST[11] = 0.25586f;
	WORKPLACE_TYPE_WEEKEND_ERRAND_PDF_HOST[12] = 0.1162f;
	WORKPLACE_TYPE_WEEKEND_ERRAND_PDF_HOST[13] = 0.113f;


	//how many adults in each household type
	HOUSEHOLD_TYPE_ADULT_COUNT_HOST[0] = 1;
	HOUSEHOLD_TYPE_ADULT_COUNT_HOST[1] = 1;
	HOUSEHOLD_TYPE_ADULT_COUNT_HOST[2] = 2;
	HOUSEHOLD_TYPE_ADULT_COUNT_HOST[3] = 1;
	HOUSEHOLD_TYPE_ADULT_COUNT_HOST[4] = 2;
	HOUSEHOLD_TYPE_ADULT_COUNT_HOST[5] = 1;
	HOUSEHOLD_TYPE_ADULT_COUNT_HOST[6] = 2;
	HOUSEHOLD_TYPE_ADULT_COUNT_HOST[7] = 1;
	HOUSEHOLD_TYPE_ADULT_COUNT_HOST[8] = 2;

	//how many children in each household type
	HOUSEHOLD_TYPE_CHILD_COUNT_HOST[0] = 0;
	HOUSEHOLD_TYPE_CHILD_COUNT_HOST[1] = 1;
	HOUSEHOLD_TYPE_CHILD_COUNT_HOST[2] = 0;
	HOUSEHOLD_TYPE_CHILD_COUNT_HOST[3] = 2;
	HOUSEHOLD_TYPE_CHILD_COUNT_HOST[4] = 1;
	HOUSEHOLD_TYPE_CHILD_COUNT_HOST[5] = 3;
	HOUSEHOLD_TYPE_CHILD_COUNT_HOST[6] = 2;
	HOUSEHOLD_TYPE_CHILD_COUNT_HOST[7] = 4;
	HOUSEHOLD_TYPE_CHILD_COUNT_HOST[8] = 3;

	//the PDF of each household type
	HOUSEHOLD_TYPE_CDF_HOST[0] = 0.279f;
	HOUSEHOLD_TYPE_CDF_HOST[1] = 0.319f;
	HOUSEHOLD_TYPE_CDF_HOST[2] = 0.628f;
	HOUSEHOLD_TYPE_CDF_HOST[3] = 0.671f;
	HOUSEHOLD_TYPE_CDF_HOST[4] = 0.8f;
	HOUSEHOLD_TYPE_CDF_HOST[5] = 0.812f;
	HOUSEHOLD_TYPE_CDF_HOST[6] = 0.939f;
	HOUSEHOLD_TYPE_CDF_HOST[7] = 0.944f;
	HOUSEHOLD_TYPE_CDF_HOST[8] = 1.0f;

	//store all permutations of contact assignments

	//number of contacts made in each hour
	WEEKDAY_ERRAND_CONTACT_ASSIGNMENT_HOST[0][0] = 2;
	WEEKDAY_ERRAND_CONTACT_ASSIGNMENT_HOST[0][1] = 0;

	WEEKDAY_ERRAND_CONTACT_ASSIGNMENT_HOST[1][0] = 0;
	WEEKDAY_ERRAND_CONTACT_ASSIGNMENT_HOST[1][1] = 2;

	WEEKDAY_ERRAND_CONTACT_ASSIGNMENT_HOST[2][0] = 1;
	WEEKDAY_ERRAND_CONTACT_ASSIGNMENT_HOST[2][1] = 1;

	//DIFFERENT FORMAT: hours each of the 2 contacts are made in
	//2 contacts in errand  0
	WEEKEND_ERRAND_CONTACT_ASSIGNMENTS_HOST[0][0] = 0;
	WEEKEND_ERRAND_CONTACT_ASSIGNMENTS_HOST[0][1] = 0;

	//2 contacts in errand 1
	WEEKEND_ERRAND_CONTACT_ASSIGNMENTS_HOST[1][0] = 1;
	WEEKEND_ERRAND_CONTACT_ASSIGNMENTS_HOST[1][1] = 1;

	//2 contacts in errand 2
	WEEKEND_ERRAND_CONTACT_ASSIGNMENTS_HOST[2][0] = 2;
	WEEKEND_ERRAND_CONTACT_ASSIGNMENTS_HOST[2][1] = 2;

	//contact in errand 0 and errand 1
	WEEKEND_ERRAND_CONTACT_ASSIGNMENTS_HOST[3][0] = 0;
	WEEKEND_ERRAND_CONTACT_ASSIGNMENTS_HOST[3][1] = 1;

	//contact in errand 0 and errand 2
	WEEKEND_ERRAND_CONTACT_ASSIGNMENTS_HOST[4][0] = 0;
	WEEKEND_ERRAND_CONTACT_ASSIGNMENTS_HOST[4][1] = 2;

	//contact in errand 1 and 2
	WEEKEND_ERRAND_CONTACT_ASSIGNMENTS_HOST[5][0] = 1;
	WEEKEND_ERRAND_CONTACT_ASSIGNMENTS_HOST[5][1] = 2;


#pragma region profiles
	//gamma1
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA1][0] = 0.007339835f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA1][1] = 0.332600216f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA1][2] = 0.501192066f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA1][3] = 0.142183447f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA1][4] = 0.015675154f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA1][5] = 0.000967407f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA1][6] = 4.055E-05f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA1][7] = 1.29105E-06f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA1][8] = 3.34836E-08f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA1][9] = 7.41011E-10f;

	//lognorm1
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM1][0] = 0.002533572f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM1][1] = 0.348252834f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM1][2] = 0.498210218f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM1][3] = 0.130145145f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM1][4] = 0.018421298f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM1][5] = 0.002158374f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM1][6] = 0.000245489f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM1][7] = 2.88922E-05f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM1][8] = 3.61113E-06f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM1][9] = 4.83901E-07f;

	//weib1
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB1][0] = 0.05927385f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB1][1] = 0.314171688f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB1][2] = 0.411588802f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB1][3] = 0.187010054f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB1][4] = 0.026934715f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB1][5] = 0.001013098f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB1][6] = 7.78449E-06f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB1][7] = 9.29441E-09f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB1][8] = 1.29796E-12f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB1][9] = 0;

	//gamma2
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA2][0] = 0.04687299f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA2][1] = 0.248505983f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA2][2] = 0.30307952f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA2][3] = 0.211008627f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA2][4] = 0.11087006f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA2][5] = 0.049241932f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA2][6] = 0.019562658f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA2][7] = 0.007179076f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA2][8] = 0.002482875f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_GAMMA2][9] = 0.000820094f;

	//lognorm2
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM2][0] = 0.028667712f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM2][1] = 0.283445338f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM2][2] = 0.319240133f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM2][3] = 0.190123057f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM2][4] = 0.093989959f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM2][5] = 0.044155659f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM2][6] = 0.020682822f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM2][7] = 0.009841839f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM2][8] = 0.00479234f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_LOGNORM2][9] = 0.002393665f;

	//weib2
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB2][0] = 0.087866042f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB2][1] = 0.223005225f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB2][2] = 0.258992749f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB2][3] = 0.208637267f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB2][4] = 0.127489076f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB2][5] = 0.061148649f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB2][6] = 0.023406737f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB2][7] = 0.007216643f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB2][8] = 0.001802145f;
	VIRAL_SHEDDING_PROFILES_HOST[PROFILE_WEIB2][9] = 0.00036581f;

#pragma endregion profiles

	//store kvals - all 1 except for no-contact
	KVAL_LOOKUP_HOST[CONTACT_TYPE_NONE] = 0;
	for(int i = CONTACT_TYPE_NONE + 1; i < NUM_CONTACT_TYPES;i++)
		KVAL_LOOKUP_HOST[i] = 1;

	if(PROFILE_SIMULATION)
		profiler.endFunction(-1,1);
}

//push various things to device constant memory
void PandemicSim::setup_pushDeviceData()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(-1,"setup_pushDeviceData");

	//data for generating households
	hipMemcpyToSymbolAsync(HIP_SYMBOL(
		HOUSEHOLD_TYPE_CDF_DEVICE),
		HOUSEHOLD_TYPE_CDF_HOST,
		sizeof(float) * HH_TABLE_ROWS,
		0,hipMemcpyHostToDevice);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(
		HOUSEHOLD_TYPE_ADULT_COUNT_DEVICE),
		HOUSEHOLD_TYPE_ADULT_COUNT_HOST,
		sizeof(int) * HH_TABLE_ROWS,
		0,hipMemcpyHostToDevice);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(
		HOUSEHOLD_TYPE_CHILD_COUNT_DEVICE),
		HOUSEHOLD_TYPE_CHILD_COUNT_HOST,
		sizeof(int) * HH_TABLE_ROWS,
		0,hipMemcpyHostToDevice);

	//data for assigning children age and school
	hipMemcpyToSymbolAsync(HIP_SYMBOL(
		CHILD_AGE_CDF_DEVICE),
		CHILD_AGE_CDF_HOST,
		sizeof(float) * CHILD_DATA_ROWS,
		0,hipMemcpyHostToDevice);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(
		CHILD_AGE_SCHOOLTYPE_LOOKUP_DEVICE),
		CHILD_AGE_SCHOOLTYPE_LOOKUP_HOST,
		sizeof(int) * CHILD_DATA_ROWS,
		0,hipMemcpyHostToDevice);

	//data for assigning workplaces
	hipMemcpyToSymbolAsync(HIP_SYMBOL(
		WORKPLACE_TYPE_ASSIGNMENT_PDF_DEVICE),
		WORKPLACE_TYPE_ASSIGNMENT_PDF_HOST,
		sizeof(float) * NUM_BUSINESS_TYPES,
		0,hipMemcpyHostToDevice);

	//workplace location data
	hipMemcpyToSymbolAsync(HIP_SYMBOL(
		WORKPLACE_TYPE_COUNT_DEVICE),
		WORKPLACE_TYPE_COUNT_HOST,
		sizeof(int) * NUM_BUSINESS_TYPES,
		0,hipMemcpyHostToDevice);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(
		WORKPLACE_TYPE_OFFSET_DEVICE),
		WORKPLACE_TYPE_OFFSET_HOST,
		sizeof(int) * NUM_BUSINESS_TYPES,
		0,hipMemcpyHostToDevice);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(
		WORKPLACE_TYPE_MAX_CONTACTS_DEVICE),
		WORKPLACE_TYPE_MAX_CONTACTS_HOST,
		sizeof(int) * NUM_BUSINESS_TYPES,
		0,hipMemcpyHostToDevice);

	//weekday+weekend errand PDFs
	hipMemcpyToSymbolAsync(HIP_SYMBOL(
		WORKPLACE_TYPE_WEEKDAY_ERRAND_PDF_DEVICE),
		WORKPLACE_TYPE_WEEKDAY_ERRAND_PDF_HOST,
		sizeof(float) * NUM_BUSINESS_TYPES,
		0,hipMemcpyHostToDevice);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(
		WORKPLACE_TYPE_WEEKEND_ERRAND_PDF_DEVICE),
		WORKPLACE_TYPE_WEEKEND_ERRAND_PDF_HOST,
		sizeof(float) * NUM_BUSINESS_TYPES,
		0,hipMemcpyHostToDevice);


	//alternate weekend contacts_desired assignment mode
	hipMemcpyToSymbolAsync(HIP_SYMBOL(
		WEEKEND_ERRAND_CONTACT_ASSIGNMENTS_DEVICE),
		WEEKEND_ERRAND_CONTACT_ASSIGNMENTS_HOST,
		sizeof(int) * 6 * 2,
		0,hipMemcpyHostToDevice);

	hipMemcpyToSymbolAsync(HIP_SYMBOL(
		WEEKDAY_ERRAND_CONTACT_ASSIGNMENTS_DEVICE),
		WEEKDAY_ERRAND_CONTACT_ASSIGNMENT_HOST,
		sizeof(int) * 3 * 2,
		0,hipMemcpyHostToDevice);

	//seeds
	hipMemcpyToSymbolAsync(HIP_SYMBOL(
		SEED_DEVICE),
		SEED_HOST,
		sizeof(int) * SEED_LENGTH,
		0,hipMemcpyHostToDevice);

	//kvals
	hipMemcpyToSymbolAsync(HIP_SYMBOL(
		KVAL_LOOKUP_DEVICE),
		KVAL_LOOKUP_HOST,
		sizeof(kval_t) * NUM_CONTACT_TYPES);

	//copy adjusted reproduction numbers
	hipMemcpyToSymbolAsync(HIP_SYMBOL(
		INFECTIOUSNESS_FACTOR_DEVICE),
		INFECTIOUSNESS_FACTOR_HOST,
		sizeof(float) * STRAIN_COUNT,
		0,hipMemcpyHostToDevice);

	//copy viral shedding profiles
	hipMemcpyToSymbolAsync(HIP_SYMBOL(
		VIRAL_SHEDDING_PROFILES_DEVICE),
		VIRAL_SHEDDING_PROFILES_HOST,
		sizeof(float) * NUM_PROFILES * CULMINATION_PERIOD,
		0,hipMemcpyHostToDevice);

	hipMemcpyToSymbolAsync(HIP_SYMBOL(
		&PERCENT_SYMPTOMATIC_DEVICE),
		&PERCENT_SYMPTOMATIC_HOST,
		sizeof(float) * 1,
		0,hipMemcpyHostToDevice);

	//must synchronize later!

	if(PROFILE_SIMULATION)
		profiler.endFunction(-1,1);
}



//Sets up the initial infection at the beginning of the simulation
//BEWARE: you must not generate dual infections with this code, or you will end up with duplicate infected indexes
void PandemicSim::setup_initialInfected()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day,"setup_initialInfected");

	//fill infected array with null info (not infected)
	int initial_infected = INITIAL_INFECTED_PANDEMIC + INITIAL_INFECTED_SEASONAL;

	//get N unique indexes - they should not be sorted
	h_vec h_init_indexes(initial_infected);
	n_unique_numbers(&h_init_indexes, initial_infected, number_people);
	thrust::copy_n(h_init_indexes.begin(), initial_infected, infected_indexes.begin());

	///// INFECTED PANDEMIC:
	//infect first INITIAL_INFECTED_PANDEMIC people with pandemic
	//set status to infected
	thrust::fill(
		thrust::make_permutation_iterator(people_status_pandemic.begin(), infected_indexes.begin()),	//begin at infected 0
		thrust::make_permutation_iterator(people_status_pandemic.begin(), infected_indexes.begin() + INITIAL_INFECTED_PANDEMIC),	//end at index INITIAL_INFECTED_PANDEMIC
		STATUS_INFECTED);

	//set day/generation pandemic to 0 (initial)
	thrust::fill(
		thrust::make_permutation_iterator(people_days_pandemic.begin(), infected_indexes.begin()),	//begin at infected 0
		thrust::make_permutation_iterator(people_days_pandemic.begin(), infected_indexes.begin() + INITIAL_INFECTED_PANDEMIC),	//end at index INITIAL_INFECTED_PANDEMIC
		INITIAL_DAY);//val
	thrust::fill(
		thrust::make_permutation_iterator(people_gens_pandemic.begin(), infected_indexes.begin()),	//begin at infected 0
		thrust::make_permutation_iterator(people_gens_pandemic.begin(), infected_indexes.begin() + INITIAL_INFECTED_PANDEMIC),	//end at index INITIAL_INFECTED_PANDEMIC
		INITIAL_GEN);	//fill infected with gen 0

	///// INFECTED SEASONAL:
	//set status to infected
	thrust::fill(
		thrust::make_permutation_iterator(people_status_seasonal.begin(), infected_indexes.begin()+ INITIAL_INFECTED_PANDEMIC), //begin at index INITIAL_INFECTED_PANDEMIC
		thrust::make_permutation_iterator(people_status_seasonal.begin(), infected_indexes.begin() + INITIAL_INFECTED_PANDEMIC + INITIAL_INFECTED_SEASONAL),	//end INITIAL_INFECTED_PANDEMIC + INITIAL_INFECTED_SEASONAL
		STATUS_INFECTED);

	//set day/generation seasonal to 0
	thrust::fill(
		thrust::make_permutation_iterator(people_days_seasonal.begin(), infected_indexes.begin()+ INITIAL_INFECTED_PANDEMIC), //begin at index INITIAL_INFECTED_PANDEMIC
		thrust::make_permutation_iterator(people_days_seasonal.begin(), infected_indexes.begin() + INITIAL_INFECTED_PANDEMIC + INITIAL_INFECTED_SEASONAL),	//end INITIAL_INFECTED_PANDEMIC + INITIAL_INFECTED_SEASONAL
		INITIAL_DAY);		//day: 0
	thrust::fill(
		thrust::make_permutation_iterator(people_gens_seasonal.begin(), infected_indexes.begin()+ INITIAL_INFECTED_PANDEMIC), //begin at index INITIAL_INFECTED_PANDEMIC
		thrust::make_permutation_iterator(people_gens_seasonal.begin(), infected_indexes.begin() + INITIAL_INFECTED_PANDEMIC + INITIAL_INFECTED_SEASONAL),	//end INITIAL_INFECTED_PANDEMIC + INITIAL_INFECTED_SEASONAL
		INITIAL_GEN);	//first generation

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day,initial_infected);
}

//sets up the locations which are the same every day and do not change
//i.e. workplace and household
void PandemicSim::setup_buildFixedLocations()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(-1,"setup_buildFixedLocations");
	///////////////////////////////////////
	//home/////////////////////////////////

	//moved to size_global_arrays func
	//household_offsets.resize(number_households + 1);
	//household_people.resize(number_people);

	/*thrust::sequence(household_people.begin(), household_people.begin() + number_people);	//fill array with IDs to sort
	calcLocationOffsets(
		&household_people,
		people_households,
		&household_offsets,
		number_people, number_households);*/

	///////////////////////////////////////
	//work/////////////////////////////////
	//workplace_offsets.resize(number_workplaces + 1);	//size arrays
	//workplace_people.resize(number_people);

	thrust::sequence(workplace_people.begin(), workplace_people.begin() + number_people);	//fill array with IDs to sort

	setup_calcLocationOffsets(
		&workplace_people,
		people_workplaces,
		&workplace_offsets,
		number_people, number_workplaces);

	//set up workplace max contacts
	workplace_max_contacts.resize(number_workplaces);		//size the array

	//copy the number of contacts per location type to device
	vec_t workplace_type_max_contacts(NUM_BUSINESS_TYPES);		
	thrust::copy_n(WORKPLACE_TYPE_MAX_CONTACTS_HOST, NUM_BUSINESS_TYPES, workplace_type_max_contacts.begin());

	//TODO:  make this work right with device constant memory.  For now, just make a copy in global memory
	vec_t business_type_count_vec(NUM_BUSINESS_TYPES);
	thrust::copy_n(WORKPLACE_TYPE_COUNT_HOST,NUM_BUSINESS_TYPES,business_type_count_vec.begin());
	vec_t business_type_count_offset_vec(NUM_BUSINESS_TYPES);
	thrust::exclusive_scan(business_type_count_vec.begin(), business_type_count_vec.end(), business_type_count_offset_vec.begin());

	//scatter code is based on Thrust example: expand.cu
	//first, scatter the indexes of the type of business into the array mapped by the output offset
	thrust::counting_iterator<int> count_iterator(0);
	thrust::scatter_if(
		count_iterator,							//value to scatter - begin - index of the type to load
		count_iterator + NUM_BUSINESS_TYPES,		//value to scatter - end
		business_type_count_offset_vec.begin(),				//map of scatter destinations
		business_type_count_vec.begin(),			//stencil: no predicate given means scatter if the count for a type is >0
		workplace_max_contacts.begin());

	//next, use a max_scan to fill in the holes, so all entries in max_contacts hold the index of their business type
	thrust::inclusive_scan(
		workplace_max_contacts.begin(),
		workplace_max_contacts.end(),
		workplace_max_contacts.begin(),
		thrust::maximum<int>());

	//now use a gather to pull the max_contacts into position
	thrust::gather(
		workplace_max_contacts.begin(),
		workplace_max_contacts.end(),
		workplace_type_max_contacts.begin(),
		workplace_max_contacts.begin());

	if(PROFILE_SIMULATION)
		profiler.endFunction(-1,number_people);
}


//given an array of people's ID numbers and locations
//sort them by location, and then build the location offset/count tables
//ids_to_sort will be sorted by workplace
void PandemicSim::setup_calcLocationOffsets(
	vec_t * ids_to_sort,
	vec_t lookup_table_copy,
	vec_t * location_offsets,
	int num_people, int num_locs)
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(-1, "calcLocationOffsets");

	//sort people by workplace
	thrust::sort_by_key(
		lookup_table_copy.begin(),
		lookup_table_copy.begin() + num_people,
		ids_to_sort->begin());

	//build count/offset table
	thrust::counting_iterator<int> count_iterator(0);
	thrust::lower_bound(		//find lower bound of each location
		lookup_table_copy.begin(),
		lookup_table_copy.begin() + num_people,
		count_iterator,
		count_iterator + num_locs,
		location_offsets->begin());

	//originally, we calculated the count by using an upper bound and then subtracting the lower bound
	//instead, we can calculate the count by the following formula:
	//loc_count = loc_offset[i+1] - loc_offset[i]
	//i.e. people = {1, 1, 2, 2, 3}
	//location_numbers = {1, 2, 3}
	//loc_offsets = {0, 2, 4}
	//We need to add one extra offset so the last location doesn't go out of bounds - this is equal to
	//the number of people in the array
	//so loc_offsets = {0, 2, 4, 5}
	(*location_offsets)[num_locs] = num_people;

	if(PROFILE_SIMULATION)
		profiler.endFunction(-1,number_people);
}


void PandemicSim::logging_closeOutputStreams()
{
	if(log_infected_info)
	{
		fclose(fInfected);
	}

	/*if(log_location_info)
	{
		fclose(fLocationInfo);
	}*/

	if(log_contacts)
	{
		fclose(fContacts);
	}

	if(log_actions)
	{
		fclose(fActions);
	}

	if(log_actions_filtered)
	{
		fclose(fActionsFiltered);
	}

	if(SIM_VALIDATION || debug_log_function_calls)
		fclose(fDebug);

	fclose(f_outputInfectedStats);
} 



void PandemicSim::runToCompletion()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(-1, "runToCompletion");

	for(current_day = 0; current_day < MAX_DAYS; current_day++)
	{
		if(debug_log_function_calls)
			debug_print("beginning day...");

		if(debug_null_fill_daily_arrays)
			debug_nullFillDailyArrays();

		daily_actions = 0;

		//begin asynchronous count of the infected stats
		daily_countInfectedStats();			

		//build infected index array
		daily_buildInfectedArray_global();
		hipDeviceSynchronize();

		if(infected_count == 0)
			break;

		daily_clearActionsArray(); //must occur AFTER we have counted infected

		if(SIM_VALIDATION)
		{
			debug_validateInfectionStatus();

			fprintf(fDebug, "\n\n---------------------\nday %d\ninfected: %d\n---------------------\n\n", current_day, infected_count);
			fflush(fDebug);
		}

		if(CONSOLE_OUTPUT)
		{
			printf("Day %d:\tinfected: %5d\n", current_day + 1, infected_count);
		}

		if(POLL_MEMORY_USAGE)
			logging_pollMemoryUsage_takeSample(current_day);

		//MAKE CONTACTS DEPENDING ON TYPE OF DAY
		if(is_weekend())
		{
			doWeekend_wholeDay();
		}
		else
		{
			doWeekday_wholeDay();
		}

		//PROCESS CONTACTS AND UPDATE INFECTED
		dailyUpdate();

		if(1)
			fflush(f_outputInfectedStats);

		//if we're using the profiler, flush each day in case of crash
		if(PROFILE_SIMULATION)
		{
			profiler.dailyFlush();
		}
	}

	final_countReproduction();

	if(PROFILE_SIMULATION)
		profiler.endFunction(-1, number_people);


	//moved to destructor for batching
	//close_output_streams();
}


//copies indexes 3 times into array, i.e. for IDS 1-3 produces array:
// 1 2 3 1 2 3 1 2 3
__device__ void device_copyPeopleIndexes_weekend_wholeDay(int * id_dest_ptr, int myIdx)
{
	id_dest_ptr[0] = myIdx;
	id_dest_ptr[1] = myIdx;
	id_dest_ptr[2] = myIdx;
}

//gets three UNIQUE errand hours 
__device__ void device_assignErrandHours_weekend_wholeDay(int * hours_dest_ptr, randOffset_t myRandOffset)
{
	threefry2x64_key_t tf_k = {{SEED_DEVICE[0], SEED_DEVICE[1]}};
	union{
		threefry2x64_ctr_t c;
		unsigned int i[4];
	} u;
	
	threefry2x64_ctr_t tf_ctr = {{ myRandOffset,  myRandOffset}};
	u.c = threefry2x64(tf_ctr, tf_k);

	int first, second, third;

	//get first hour
	first = u.i[0] % NUM_WEEKEND_ERRAND_HOURS;

	//get second hour, if it matches then increment
	second = u.i[1] % NUM_WEEKEND_ERRAND_HOURS;
	if(second == first)
		second = (second + 1) % NUM_WEEKEND_ERRAND_HOURS;

	//get third hour, increment until it no longer matches
	third = u.i[2] % NUM_WEEKEND_ERRAND_HOURS;
	while(third == first || third == second)
		third = (third + 1 ) % NUM_WEEKEND_ERRAND_HOURS;

	//store in output array
	hours_dest_ptr[0] = first;
	hours_dest_ptr[1] = second;
	hours_dest_ptr[2] = third;
}

__device__ void device_fishWeekendErrandDestination(unsigned int * rand_val, int * output_ptr)
{
	float y = (float) *rand_val / UNSIGNED_MAX;

	int row = FIRST_WEEKEND_ERRAND_ROW;
	while(y > WORKPLACE_TYPE_WEEKEND_ERRAND_PDF_DEVICE[row] && row < (NUM_BUSINESS_TYPES - 1))
	{
		y -= WORKPLACE_TYPE_WEEKEND_ERRAND_PDF_DEVICE[row];
		row++;
	}
	float frac = y / WORKPLACE_TYPE_WEEKEND_ERRAND_PDF_DEVICE[row];
	int type_count = WORKPLACE_TYPE_COUNT_DEVICE[row];
	int business_num = frac * type_count;

	if(business_num >= type_count)
		business_num = type_count - 1;

	int type_offset = WORKPLACE_TYPE_OFFSET_DEVICE[row];

	*output_ptr = business_num + type_offset;
}

//This method consumes the accumulated contacts, and causes infections and recovery to occur
void PandemicSim::dailyUpdate()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "dailyUpdate");

	if(debug_log_function_calls)
		debug_print("beginning daily update");

	//synchronize the secondary stream - this ensures that the countInfected kernel has finished
	//and sent its data, and that the actions array has been nulled
	hipStreamSynchronize(stream_secondary);
	daily_writeInfectedStats();

	//process contacts into actions
	daily_contactsToActions_new();

	//filter invalid actions - not susceptible, duplicate, etc
	daily_filterActions_new();

	//do infection actions
	daily_doInfectionActions();

	//recover infected who have reached culmination
	daily_recoverInfected_new();

	if(debug_log_function_calls)
		debug_print("daily update complete");

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, infected_count);
}



//will resize the infected, contact, and action arrays to fit the entire population
void PandemicSim::setup_sizeGlobalArrays()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(-1,"setup_sizeGlobalArrays");
	//setup people status:
	people_status_pandemic.resize(number_people);
	people_status_seasonal.resize(number_people);
	thrust::fill(people_status_pandemic.begin(), people_status_pandemic.end(), STATUS_SUSCEPTIBLE);
	thrust::fill(people_status_seasonal.begin(), people_status_seasonal.end(), STATUS_SUSCEPTIBLE);

	people_days_pandemic.resize(number_people);
	people_days_seasonal.resize(number_people);
	thrust::fill(people_days_pandemic.begin(), people_days_pandemic.end(), DAY_NOT_INFECTED);
	thrust::fill(people_days_seasonal.begin(), people_days_seasonal.end(), DAY_NOT_INFECTED);

	people_gens_pandemic.resize(number_people);
	people_gens_seasonal.resize(number_people);
	thrust::fill(people_gens_pandemic.begin(), people_gens_pandemic.end(), GENERATION_NOT_INFECTED);
	thrust::fill(people_gens_seasonal.begin(), people_gens_seasonal.end(), GENERATION_NOT_INFECTED);

	people_ages.resize(number_people);
	people_households.resize(number_people);
	people_workplaces.resize(number_people);
	people_child_indexes.resize(number_children);
	people_adult_indexes.resize(number_adults);

	household_offsets.resize(number_households + 1);
	household_people.resize(number_people);

	workplace_offsets.resize(number_workplaces + 1);
	workplace_people.resize(number_people);
	workplace_max_contacts.resize(number_workplaces);

	//assume that worst-case everyone gets infected
	infected_indexes.resize(number_people);
	infected_daily_kval_sum.resize(number_people);

	int expected_max_contacts = number_people * MAX_CONTACTS_PER_DAY;

	//resize contact arrays
	daily_contact_infectors.resize(expected_max_contacts);
	daily_contact_victims.resize(expected_max_contacts);
	daily_contact_kval_types.resize(expected_max_contacts);
	daily_action_type.resize(expected_max_contacts);

	//weekend errands arrays tend to be very large, so pre-allocate them
	int num_weekend_errands = number_people * NUM_WEEKEND_ERRANDS;
	errand_people_table.resize(num_weekend_errands);
	errand_people_weekendHours.resize(num_weekend_errands);
	errand_people_destinations.resize(num_weekend_errands);

	errand_infected_locations.resize(num_weekend_errands);
	errand_infected_weekendHours.resize(num_weekend_errands);
	errand_infected_ContactsDesired.resize(number_people);

	errand_locationOffsets_multiHour.resize((number_workplaces * NUM_WEEKEND_ERRAND_HOURS) + 1);
	errand_hourOffsets_weekend.resize(NUM_WEEKEND_ERRAND_HOURS + 1);
	errand_hourOffsets_weekend[NUM_WEEKEND_ERRAND_HOURS] = NUM_WEEKEND_ERRANDS * number_people;

	status_counts.resize(16);

	if(SIM_VALIDATION)
	{
		debug_contactsToActions_float1.resize(expected_max_contacts);
		debug_contactsToActions_float2.resize(expected_max_contacts);
		debug_contactsToActions_float3.resize(expected_max_contacts);
		debug_contactsToActions_float4.resize(expected_max_contacts);
	}

	setup_fetchVectorPtrs(); //get the raw int * pointers

	if(PROFILE_SIMULATION)
	{
		profiler.endFunction(-1,number_people);
	}
}



void PandemicSim::debug_nullFillDailyArrays()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day,"debug_nullFillDailyArrays");

	thrust::fill(daily_contact_infectors.begin(), daily_contact_infectors.end(), -1);
	thrust::fill(daily_contact_victims.begin(), daily_contact_victims.end(), -1);
	thrust::fill(daily_contact_kval_types.begin(), daily_contact_kval_types.end(), CONTACT_TYPE_NONE);
	thrust::fill(infected_daily_kval_sum.begin(), infected_daily_kval_sum.end(), 0);

	thrust::fill(daily_action_type.begin(), daily_action_type.end(), ACTION_INFECT_NONE);

	thrust::fill(errand_infected_locations.begin(), errand_infected_locations.end(), -1);
	thrust::fill(errand_infected_weekendHours.begin(), errand_infected_weekendHours.end(), -1);
	thrust::fill(errand_infected_ContactsDesired.begin(), errand_infected_ContactsDesired.end(), -1);

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, number_people);
}

void PandemicSim::setup_scaleSimulation()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(-1,"setup_scaleSimulation");

	number_households = roundHalfUp_toInt(sim_scaling_factor * (double) number_households);

	int sum = 0;
	for(int business_type = 0; business_type < NUM_BUSINESS_TYPES; business_type++)
	{
		//for each type of business, scale by overall simulation scalar
		int original_type_count = roundHalfUp_toInt(WORKPLACE_TYPE_COUNT_HOST[business_type]);
		int new_type_count = roundHalfUp_toInt(sim_scaling_factor * original_type_count);

		//if at least one business of this type existed in the original data, make sure at least one exists in the new data
		if(new_type_count == 0 && original_type_count > 0)
			new_type_count = 1;

		WORKPLACE_TYPE_COUNT_HOST[business_type] = new_type_count;
		sum += new_type_count;
	}

	number_workplaces = sum;

	//calculate the offset of each workplace type
	thrust::exclusive_scan(
		WORKPLACE_TYPE_COUNT_HOST,
		WORKPLACE_TYPE_COUNT_HOST + NUM_BUSINESS_TYPES,
		WORKPLACE_TYPE_OFFSET_HOST);

	if(PROFILE_SIMULATION)
		profiler.endFunction(-1,NUM_BUSINESS_TYPES);
}

void PandemicSim::debug_dump_array_toTempFile(const char * filename, const char * description, d_vec * target_array, int array_count)
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "debug_dumpArray_toTempFile");

	h_vec host_array(array_count);
	thrust::copy_n(target_array->begin(), array_count, host_array.begin());

	FILE * fTemp = fopen(filename,"w");
	fprintf(fTemp,"i,%s\n",description);
	for(int i = 0; i < array_count; i++)
	{
		fprintf(fTemp,"%d,%d\n",i,host_array[i]);
	}
	fclose(fTemp);

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day,array_count);
}


void PandemicSim::doWeekday_wholeDay()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "doWeekday_wholeDay");

	//generate errands and afterschool locations
	weekday_scatterAfterschoolLocations_wholeDay(&errand_people_destinations);
	weekday_scatterErrandDestinations_wholeDay(&errand_people_destinations);
	hipDeviceSynchronize();

//	debug_dump_array_toTempFile("../unsorted_dests.txt","errand dest", &errand_people_destinations, number_people * NUM_WEEKDAY_ERRAND_HOURS);

	//fish out the locations of the infected people
	weekday_doInfectedSetup_wholeDay(&errand_people_destinations, &errand_infected_locations, &errand_infected_ContactsDesired);
	if(SIM_VALIDATION)
		debug_copyErrandLookup();	//debug: copy the lookup tables to host memory before they are sorted
	hipDeviceSynchronize();


	//generate location arrays for each hour
	for(int hour = 0; hour < NUM_WEEKDAY_ERRAND_HOURS; hour++)
	{
		int people_offset_start = hour * number_people;
		int people_offset_end = (hour+1) * number_people;

		//write sequential blocks of indexes, i.e. 0 1 2 0 1 2
		thrust::sequence(
			errand_people_table.begin() + people_offset_start,
			errand_people_table.begin() + people_offset_end);

		//sort the indexes by destination
		thrust::sort_by_key(
			errand_people_destinations.begin() + people_offset_start,	//key.begin
			errand_people_destinations.begin() + people_offset_end,		//key.end
			errand_people_table.begin() + people_offset_start);			//vals.begin

		int location_offset_start = hour * number_workplaces;
//		int location_offset_end = location_offset_start + number_workplaces;
		thrust::counting_iterator<int> count_it(0);

		//binary search the location offsets
		thrust::lower_bound(
			errand_people_destinations.begin() + people_offset_start,	//vals.begin: search workplace 0 to N for this hour
			errand_people_destinations.begin() + people_offset_end,			//vals.end
			count_it,
			count_it + number_workplaces,
			errand_locationOffsets_multiHour.begin() + location_offset_start);		//output.begin
	}

//	debug_dump_array_toTempFile("../sorted_dests.txt", "errand_dest", &errand_people_destinations, number_people * NUM_WEEKDAY_ERRAND_HOURS);
//	debug_dump_array_toTempFile("../loc_offsets.txt", "loc_offset", &errand_locationOffsets_multiHour, NUM_WEEKDAY_ERRAND_HOURS * number_workplaces);
//	debug_dump_array_toTempFile("../inf_locs.txt", "loc", &errand_infected_locations, infected_count * NUM_WEEKDAY_ERRAND_HOURS);

//	debug_dumpInfectedErrandLocs();

	makeContactsKernel_weekday<<<cuda_makeWeekdayContactsKernel_blocks,cuda_makeWeekdayContactsKernel_threads>>>(
		infected_count, infected_indexes_ptr, people_ages_ptr,
		people_households_ptr, household_offsets_ptr, household_people_ptr,
		workplace_max_contacts_ptr, people_workplaces_ptr, 
		workplace_offsets_ptr, workplace_people_ptr,
		errand_infected_ContactsDesired_ptr, errand_infected_locations_ptr,
		errand_locationOffsets_multiHour_ptr,errand_people_table_ptr,
		number_workplaces,
		daily_contact_infectors_ptr, daily_contact_victims_ptr, daily_contact_kval_types_ptr,
		infected_daily_kval_sum_ptr, rand_offset, number_people);

	if(TIMING_BATCH_MODE == 0)
	{
		const int rand_counts_consumed = 2;
		rand_offset += (rand_counts_consumed * infected_count);
	}
	hipDeviceSynchronize();

	if(SIM_VALIDATION)
		validateContacts_wholeDay();

//	debug_dump_array_toTempFile("../infected_kvals.txt","kval",&infected_daily_kval_sum, infected_count);

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day,infected_count);
}

void PandemicSim::weekday_scatterAfterschoolLocations_wholeDay(d_vec * people_locs)
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "weekday_scatterAfterschoolLocations_wholeDay");

	int * output_arr_ptr = thrust::raw_pointer_cast(people_locs->data());

	kernel_assignAfterschoolLocations_wholeDay<<<cuda_blocks,cuda_threads>>>(people_child_indexes_ptr,output_arr_ptr, number_children,number_people,rand_offset);
	rand_offset += number_children / 4;

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day,number_children);
}

void PandemicSim::weekday_scatterErrandDestinations_wholeDay(d_vec * people_locs)
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "weekday_scatterAfterschoolLocations_wholeDay");

	int * output_arr_ptr = thrust::raw_pointer_cast(people_locs->data());

	kernel_assignErrandLocations_weekday_wholeDay<<<cuda_blocks,cuda_threads>>>(people_adults_indexes_ptr, number_adults, number_people ,output_arr_ptr, rand_offset);
	rand_offset += number_adults / 2;

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day,number_adults);
}

void PandemicSim::doWeekend_wholeDay()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "doWeekend_wholeDay");

	//assign all weekend errands
	weekend_assignErrands(&errand_people_table, &errand_people_weekendHours, &errand_people_destinations);
	hipDeviceSynchronize();

	//fish the infected errands out
	weekend_doInfectedSetup_wholeDay(&errand_people_weekendHours,&errand_people_destinations, &errand_infected_weekendHours, &errand_infected_locations, &errand_infected_ContactsDesired);
	if(SIM_VALIDATION)
		debug_copyErrandLookup();
	hipDeviceSynchronize();

	//each person gets 3 errands
	const int num_weekend_errands_total = NUM_WEEKEND_ERRANDS * number_people;

	//now sort the errand_people array into a large multi-hour location table
	thrust::sort_by_key(
		thrust::make_zip_iterator(thrust::make_tuple(
			errand_people_weekendHours.begin(), 
			errand_people_destinations.begin())),	//key.begin
		thrust::make_zip_iterator(thrust::make_tuple(
			errand_people_weekendHours.begin() + num_weekend_errands_total, 
			errand_people_destinations.begin() + num_weekend_errands_total)),		//key.end
		errand_people_table.begin(),
		Pair_SortByFirstThenSecond_struct());									//data

	//find how many people are going on errands during each hour
	thrust::counting_iterator<int> count_it(0);
	thrust::lower_bound(
		errand_people_weekendHours.begin(),
		errand_people_weekendHours.begin() + num_weekend_errands_total,
		count_it,
		count_it + NUM_WEEKEND_ERRAND_HOURS,
		errand_hourOffsets_weekend.begin());
	//people_hour_offsets[NUM_WEEKEND_ERRAND_HOURS] = num_weekend_errands_total;	//moved to size_global_array method


//	debug_dump_array_toTempFile("../weekend_hour_offsets.txt","hour offset",&errand_hourOffsets_weekend,NUM_WEEKEND_ERRAND_HOURS + 1);

	for(int hour = 0; hour < NUM_WEEKEND_ERRAND_HOURS; hour++)
	{
		int location_offset_start = hour * number_workplaces;

		//search for the locations within this errand hour
		thrust::lower_bound(
			errand_people_destinations.begin() + errand_hourOffsets_weekend[hour],
			errand_people_destinations.begin() + errand_hourOffsets_weekend[hour+1],
			count_it,
			count_it + number_workplaces,
			errand_locationOffsets_multiHour.begin() + location_offset_start);
	}

	debug_validateLocationArrays();
//	debug_dump_array_toTempFile("../weekend_loc_offsets.csv","loc offset",&errand_locationOffsets_multiHour, (NUM_WEEKEND_ERRAND_HOURS * number_workplaces));


	//launch kernel
	hipDeviceSynchronize();

	makeContactsKernel_weekend<<<cuda_makeWeekendContactsKernel_blocks,cuda_makeWeekendContactsKernel_threads>>>(
		infected_count, infected_indexes_ptr,
		people_households_ptr, household_offsets_ptr, household_people_ptr,
		errand_infected_weekendHours_ptr, errand_infected_locations_ptr, errand_infected_ContactsDesired_ptr,
		errand_locationOffsets_multiHour_ptr ,errand_people_table_ptr, errand_hourOffsets_weekend_ptr,
		number_workplaces,
		daily_contact_infectors_ptr, daily_contact_victims_ptr, daily_contact_kval_types_ptr,
		infected_daily_kval_sum_ptr, rand_offset);

	if(TIMING_BATCH_MODE == 0)
	{
		int rand_counts_used = 2 * infected_count;
		rand_offset += rand_counts_used;
	}
	hipDeviceSynchronize();

	if(log_contacts)
		validateContacts_wholeDay();

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day,infected_count);
}

void PandemicSim::weekday_doInfectedSetup_wholeDay(vec_t * lookup_array, vec_t * inf_locs, vec_t * inf_contacts_desired)
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "weekday_doInfectedSetup_wholeDay");

	int * loc_lookup_ptr = thrust::raw_pointer_cast(lookup_array->data());
	int * inf_locs_ptr = thrust::raw_pointer_cast(inf_locs->data());
	int * inf_contacts_desired_ptr = thrust::raw_pointer_cast(inf_contacts_desired->data());

	kernel_doInfectedSetup_weekday_wholeDay<<<cuda_blocks, cuda_threads>>>(
		infected_indexes_ptr,infected_count,
		loc_lookup_ptr,people_ages_ptr,number_people,
		inf_locs_ptr,inf_contacts_desired_ptr, rand_offset);

	const int rand_counts_used = infected_count / 4;
	rand_offset += rand_counts_used;

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day,infected_count);
}

void PandemicSim::weekend_doInfectedSetup_wholeDay(vec_t * errand_hours, vec_t * errand_destinations, vec_t * infected_hours, vec_t * infected_destinations, vec_t * infected_contacts_desired)
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "weekend_doInfectedSetup");

	//second input: collated lookup tables for hours and destinations
	int * errand_hour_ptr = thrust::raw_pointer_cast(errand_hours->data());
	int * errand_dest_ptr = thrust::raw_pointer_cast(errand_destinations->data());

	//outputs: the hour of the errands and the destinations
	int * infected_hour_ptr = thrust::raw_pointer_cast(infected_hours->data());
	int * infected_destinations_ptr = thrust::raw_pointer_cast(infected_destinations->data());
	int * infected_contacts_desired_ptr = thrust::raw_pointer_cast(infected_contacts_desired->data());

	kernel_doInfectedSetup_weekend<<<cuda_blocks,cuda_threads>>>(
		infected_indexes_ptr,errand_hour_ptr,errand_dest_ptr,
		infected_hour_ptr, infected_destinations_ptr, infected_contacts_desired_ptr,
		infected_count, rand_offset);

	int rand_counts_consumed = infected_count / 4;
	rand_offset += rand_counts_consumed;

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day,infected_count);
}

void PandemicSim::weekend_assignErrands(vec_t * errand_people, vec_t * errand_hours, vec_t * errand_destinations)
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "weekend_assignErrands");

	int * errand_people_ptr = thrust::raw_pointer_cast(errand_people->data());
	int * errand_hours_ptr = thrust::raw_pointer_cast(errand_hours->data());
	int * errand_dests_ptr=  thrust::raw_pointer_cast(errand_destinations->data());

	kernel_assignErrands_weekend<<<cuda_blocks,cuda_threads>>>(errand_people_ptr,errand_hours_ptr,errand_dests_ptr, number_people,rand_offset);

	int rand_counts_consumed = 2 * number_people;
	rand_offset += rand_counts_consumed;

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day,number_people);
}

__device__ void device_assignContactsDesired_weekday_wholeDay(unsigned int rand_val, int myAge, int * output_contacts_desired)
{
	int contacts_hour[2];
	if(myAge == AGE_ADULT)
	{
		//get a profile between 0 and 2
		int contacts_profile = rand_val % 3;

		contacts_hour[0] = WEEKDAY_ERRAND_CONTACT_ASSIGNMENTS_DEVICE[contacts_profile][0];
		contacts_hour[1] = WEEKDAY_ERRAND_CONTACT_ASSIGNMENTS_DEVICE[contacts_profile][1];
	}
	else
	{
		contacts_hour[0] = WORKPLACE_TYPE_MAX_CONTACTS_DEVICE[BUSINESS_TYPE_AFTERSCHOOL];
		contacts_hour[1] = 0;
	}

	*(output_contacts_desired) = contacts_hour[0];
	*(output_contacts_desired) = contacts_hour[1];
}

__global__ void kernel_assignContactsDesired_weekday_wholeDay(int * infected_indexes_arr, int num_infected, int * age_lookup_arr, int * contacts_desired_arr, randOffset_t rand_offset)
{
	threefry2x64_key_t tf_k = {{(long) SEED_DEVICE[0], (long) SEED_DEVICE[1]}};
	union{
		threefry2x64_ctr_t c;
		unsigned int i[4];
	} rand_union;

	for(int myGridPos = blockIdx.x * blockDim.x + threadIdx.x;  myGridPos < num_infected / 4; myGridPos += gridDim.x * blockDim.x)
	{
		randOffset_t myRandOffset = myGridPos + rand_offset;
		threefry2x64_ctr_t tf_ctr = {{myRandOffset,	myRandOffset}};
		rand_union.c = threefry2x64(tf_ctr,tf_k);

		int myPos = num_infected * 4;
		int myIdx[4];
		int myAge[4];

		if(myPos < num_infected)
		{
			myIdx[0] = infected_indexes_arr[myPos];
			myAge[0] = age_lookup_arr[myIdx[0]];
			device_assignContactsDesired_weekday_wholeDay(rand_union.i[0], myAge[0], contacts_desired_arr + (myPos * 2));
		}

		if(myPos + 1 < num_infected)
		{
			myIdx[1] = infected_indexes_arr[myPos+1];
			myAge[1] = age_lookup_arr[myIdx[1]];
			device_assignContactsDesired_weekday_wholeDay(rand_union.i[1], myAge[1], contacts_desired_arr + ((myPos+1) * 2));
		}

		if(myPos + 2 < num_infected)
		{
			myIdx[2] = infected_indexes_arr[myPos+2];
			myAge[2] = age_lookup_arr[myIdx[2]];
			device_assignContactsDesired_weekday_wholeDay(rand_union.i[2], myAge[2], contacts_desired_arr + ((myPos+2) * 2));
		}

		if(myPos + 3 < num_infected)
		{
			myIdx[3] = infected_indexes_arr[myPos+3];
			myAge[3] = age_lookup_arr[myIdx[3]];
			device_assignContactsDesired_weekday_wholeDay(rand_union.i[3], myAge[3], contacts_desired_arr + ((myPos+3) * 2));
		}
	}
}

__device__ void device_assignContactsDesired_weekday_wholeDay(unsigned int * rand_val, int age, int * contactsDesiredHour1, int * contactsDesiredHour2)
{
	int hour1, hour2;
	if(age == AGE_ADULT)
	{
		//assign 2 contacts between the two hours
		hour1 = (*rand_val) % 3;
		hour2 = 2 - hour1;
	}
	else
	{
		//look up max contacts for afterschool type
		hour1 = WORKPLACE_TYPE_MAX_CONTACTS_DEVICE[BUSINESS_TYPE_AFTERSCHOOL];
		hour2 = 0;
	}
	*contactsDesiredHour1 = hour1;
	*contactsDesiredHour2 = hour2;
}
__device__ void device_copyInfectedErrandLocs_weekday(int * loc_lookup_ptr, int * output_infected_locs_ptr, int num_people)
{
	*(output_infected_locs_ptr) = *loc_lookup_ptr;
	*(output_infected_locs_ptr+1) = *(loc_lookup_ptr + num_people);
}

__device__ void device_doAllWeekdayInfectedSetup(unsigned int * rand_val, int myPos, int * infected_indexes_arr, int * loc_lookup_arr, int * ages_lookup_arr, int num_people, int * output_infected_locs, int * output_infected_contacts_desired)
{
	int myIdx = infected_indexes_arr[myPos];
	int myAge = ages_lookup_arr[myIdx];
	int output_offset = 2 * myPos;
	device_copyInfectedErrandLocs_weekday(loc_lookup_arr + myIdx, output_infected_locs + output_offset, num_people);
	device_assignContactsDesired_weekday_wholeDay(rand_val, myAge,
		output_infected_contacts_desired + output_offset,
		output_infected_contacts_desired + output_offset + 1);
}
__global__ void kernel_doInfectedSetup_weekday_wholeDay(int * infected_index_arr, int num_infected, int * loc_lookup_arr, int * ages_lookup_arr, int num_people, int * output_infected_locs, int * output_infected_contacts_desired, randOffset_t rand_offset)
{
	threefry2x64_key_t tf_k = {{(long) SEED_DEVICE[0], (long) SEED_DEVICE[1]}};
	union{
		threefry2x64_ctr_t c;
		unsigned int i[4];
	} rand_union;

	for(int myGridPos = blockIdx.x * blockDim.x + threadIdx.x;  myGridPos <= num_infected / 4; myGridPos += gridDim.x * blockDim.x)
	{
		randOffset_t myRandOffset = myGridPos + rand_offset;
		//get 4 random numbers
		threefry2x64_ctr_t tf_ctr = {{myRandOffset, myRandOffset}};
		rand_union.c = threefry2x64(tf_ctr,tf_k);

		//select a block of up to 4 infected people
		int myPos = myGridPos * 4;

		if(myPos < num_infected)
		{
			device_doAllWeekdayInfectedSetup(&(rand_union.i[0]),myPos, 
				infected_index_arr, loc_lookup_arr, ages_lookup_arr,
				num_people, output_infected_locs, output_infected_contacts_desired);
		}

		if(myPos + 1 < num_infected)
		{
			device_doAllWeekdayInfectedSetup(&(rand_union.i[1]),myPos + 1, 
				infected_index_arr, loc_lookup_arr, ages_lookup_arr,
				num_people, output_infected_locs, output_infected_contacts_desired);
		}

		if(myPos + 2 < num_infected)
		{
			device_doAllWeekdayInfectedSetup(&(rand_union.i[2]),myPos + 2, 
				infected_index_arr, loc_lookup_arr, ages_lookup_arr,
				num_people, output_infected_locs, output_infected_contacts_desired);
		}

		if(myPos + 3 < num_infected)
		{
			device_doAllWeekdayInfectedSetup(&(rand_union.i[3]),myPos + 3, 
				infected_index_arr, loc_lookup_arr, ages_lookup_arr,
				num_people, output_infected_locs, output_infected_contacts_desired);
		}
	}
}

#pragma region debug_printing_funcs

inline void debug_print(char * message)
{
	fprintf(fDebug, "%s\n", message);
	fflush(fDebug);
} 



inline void debug_assert(bool condition, char * message)
{
	if(!condition)
	{
		fprintf(fDebug, "ERROR: ");
		debug_print(message);
	}
}

inline void debug_assert(char *message, int expected, int actual)
{
	if(expected != actual)
	{
		fprintf(fDebug, "ERROR: %s expected: %d actual: %d\n", message, expected, actual);
		fflush(fDebug);
	}
}

inline void debug_assert(bool condition, char * message, int idx)
{
	if(!condition)
	{
		fprintf(fDebug, "ERROR: %s index: %d\n", message, idx);
	}
}
#pragma endregion debug_printing_funcs

#pragma region debug_lookup_funcs

inline char status_int_to_char(int s)
{
	switch(s)
	{
	case STATUS_SUSCEPTIBLE:
		return 'S';
	case STATUS_INFECTED:
		return 'I';
	case STATUS_RECOVERED:
		return 'R';
	default:
		return '?';
	}
}

inline char * action_type_to_string(int action)
{
	switch(action)
	{
	case ACTION_INFECT_NONE:
		return "NONE";
	case ACTION_INFECT_PANDEMIC:
		return "PAND";
	case ACTION_INFECT_SEASONAL:
		return "SEAS";
	case ACTION_INFECT_BOTH:
		return "BOTH";
	default:
		return "????";
	}
}

inline int lookup_school_typecode_from_age_code(int age_code)
{
	switch(age_code)
	{
	case AGE_5:
		return BUSINESS_TYPE_PRESCHOOL;
	case AGE_9:
		return BUSINESS_TYPE_ELEMENTARYSCHOOL;
	case AGE_14:
		return BUSINESS_TYPE_MIDDLESCHOOL;
	case AGE_17:
		return BUSINESS_TYPE_HIGHSCHOOL;
	case AGE_22:
		return BUSINESS_TYPE_UNIVERSITY;
	default:
		throw std::runtime_error("invalid school typecode");
	}
}

char * profile_int_to_string(int p)
{
	switch(p)
	{
	case PROFILE_GAMMA1:
		return "GAMMA1";
	case PROFILE_GAMMA2:
		return "GAMMA2";
	case PROFILE_LOGNORM1:
		return "LOGNORM1";
	case PROFILE_LOGNORM2:
		return "LOGNORM2";
	case PROFILE_WEIB1:
		return "WEIB1";
	case PROFILE_WEIB2:
		return "WEIB2";
	default:
		return "ERR_BAD_PROFILE_NUM";
	}
}
#pragma endregion debug_lookup_funcs

//generates N unique numbers between 0 and max, exclusive
//assumes array is big enough that this won't be pathological
void n_unique_numbers(h_vec *array, int n, int max)
{
	for(int i = 0; i < n; i++)
	{
		do
		{
			(*array)[i] = rand() % max;
			for(int j =0; j < i; j++)
			{
				if((*array)[j] == (*array)[i])
				{
					(*array)[i] = -1;
					break;
				}
			}
		}while((*array)[i] == -1);
	}
}


int roundHalfUp_toInt(double d)
{
	return floor(d + 0.5);
}



__global__ void makeContactsKernel_weekday(int num_infected, int * infected_indexes, int * people_age,
										   int * household_lookup, int * household_offsets, int * household_people,
										   int * workplace_max_contacts, int * workplace_lookup, 
										   int * workplace_offsets, int * workplace_people,
										   int * errand_contacts_desired, int * errand_infected_locs,
										   int * errand_loc_offsets, int * errand_people,
										   int number_locations, 
										   int * output_infector_arr, int * output_victim_arr, int * output_kval_arr,
										   kval_t * output_kval_sum_arr, int rand_offset, int number_people)

{
	for(int myPos = blockIdx.x * blockDim.x + threadIdx.x;  myPos < num_infected; myPos += gridDim.x * blockDim.x)
	{
		int output_offset_base = MAX_CONTACTS_WEEKDAY * myPos;

		int myIdx = infected_indexes[myPos];
		int myAge = people_age[myIdx];

		threefry2x64_key_t tf_k = {{(long) SEED_DEVICE[0], (long) SEED_DEVICE[1]}};
		union{
			threefry2x64_ctr_t c[2];
			unsigned int i[8];
		} rand_union;
		//generate first set of random numbers

		threefry2x64_ctr_t tf_ctr_1 = {{(long) ((myPos * 2) + rand_offset), (long) ((myPos * 2) + rand_offset)}};
		rand_union.c[0] = threefry2x64(tf_ctr_1, tf_k);

		kval_t household_kval_sum = 0;
		{
			int loc_offset, loc_count;

			//household: make three contacts
			device_lookupLocationData_singleHour(myIdx, household_lookup, household_offsets, &loc_offset, &loc_count);  //lookup location data for household
			device_selectRandomPersonFromLocation(
				myIdx, loc_offset, loc_count,rand_union.i[0], CONTACT_TYPE_HOME,
				household_people,
				output_infector_arr + output_offset_base + 0,
				output_victim_arr + output_offset_base + 0,
				output_kval_arr + output_offset_base + 0,
				&household_kval_sum);
			device_selectRandomPersonFromLocation(
				myIdx, loc_offset, loc_count,rand_union.i[1], CONTACT_TYPE_HOME,
				household_people,
				output_infector_arr + output_offset_base + 1,
				output_victim_arr + output_offset_base + 1,
				output_kval_arr + output_offset_base + 1,
				&household_kval_sum);
			device_selectRandomPersonFromLocation(
				myIdx, loc_offset, loc_count,rand_union.i[2], CONTACT_TYPE_HOME,
				household_people,
				output_infector_arr + output_offset_base + 2,
				output_victim_arr + output_offset_base + 2,
				output_kval_arr + output_offset_base + 2,
				&household_kval_sum);			
		}

		//generate the second set of random numbers
		threefry2x64_ctr_t tf_ctr_2 = {{(long) ((myPos * 2) + rand_offset + 1), (long) ((myPos * 2) + rand_offset + 1)}};
		rand_union.c[1] = threefry2x64(tf_ctr_2, tf_k);

		//now the number of contacts made will diverge, so we need to count it
		int contacts_made = 3;
		kval_t workplace_kval_sum = 0;
		{
			int contacts_desired, loc_offset, loc_count, kval_type;
			int local_contacts_made = contacts_made;			//this will let both loops interleave

			//look up max_contacts into contacts_desired
			device_lookupLocationData_singleHour(
				myIdx, workplace_lookup,workplace_offsets, workplace_max_contacts,	//input
				&loc_offset, &loc_count, &contacts_desired);	
			contacts_made += contacts_desired;

			if(myAge == AGE_ADULT)
				kval_type = CONTACT_TYPE_WORKPLACE;
			else
				kval_type = CONTACT_TYPE_SCHOOL;

			while(contacts_desired > 0 && local_contacts_made < MAX_CONTACTS_WEEKDAY)
			{
				int output_offset = output_offset_base + local_contacts_made;
				device_selectRandomPersonFromLocation(
					myIdx,loc_offset, loc_count, rand_union.i[local_contacts_made], kval_type,
					workplace_people,
					output_infector_arr + output_offset,
					output_victim_arr + output_offset,
					output_kval_arr + output_offset,
					&workplace_kval_sum);

				contacts_desired--;
				local_contacts_made++;
			}
		}
		
		//do errands
		kval_t errand_kval_sum = 0;
		{
			
			int kval_type;

			//set kval for the errands
			if(myAge == AGE_ADULT)
				kval_type = CONTACT_TYPE_ERRAND;
			else
				kval_type = CONTACT_TYPE_AFTERSCHOOL;

			for(int hour = 0; hour < NUM_WEEKDAY_ERRAND_HOURS; hour++)
			{
				int contacts_desired, loc_offset, loc_count;

				//fish out location offset, count, and contacts desired
				device_lookupInfectedLocation_multiHour(
					myPos, hour, 
					errand_infected_locs, errand_loc_offsets, number_locations, number_people,
					errand_contacts_desired, NUM_WEEKDAY_ERRAND_HOURS,
					&loc_offset, &loc_count, &contacts_desired);
				
				//make contacts
				while(contacts_desired > 0 && contacts_made < MAX_CONTACTS_WEEKDAY)
				{
					int output_offset = output_offset_base + contacts_made;
					device_selectRandomPersonFromLocation(
						myIdx, loc_offset, loc_count, rand_union.i[contacts_made], kval_type,
						errand_people, 
						output_infector_arr + output_offset,
						output_victim_arr + output_offset,
						output_kval_arr + output_offset,
						&errand_kval_sum);

					contacts_desired--;
					contacts_made++;
				}
			}

			//if person has made less than max contacts, fill the end with null contacts
			while(contacts_made < MAX_CONTACTS_WEEKDAY)
			{
				int output_offset = output_offset_base + contacts_made;
				device_nullFillContact(myIdx,
					output_infector_arr + output_offset,
					output_victim_arr + output_offset,
					output_kval_arr + output_offset);
				contacts_made++;
			}

			output_kval_sum_arr[myPos] = household_kval_sum + workplace_kval_sum + errand_kval_sum;
		}
	}
}


__global__ void makeContactsKernel_weekend(int num_infected, int * infected_indexes,
										   int * household_lookup, int * household_offsets, int * household_people,
										   int * infected_errand_hours, int * infected_errand_destinations,
										   int * infected_errand_contacts_profile,
										   int * errand_loc_offsets, int * errand_people,
										   int * errand_populationCount_exclusiveScan,
										   int number_locations, 
										   int * output_infector_arr, int * output_victim_arr, int * output_kval_arr,
										   kval_t * output_kval_sum_arr, int rand_offset)
{
	for(int myPos = blockIdx.x * blockDim.x + threadIdx.x;  myPos < num_infected; myPos += gridDim.x * blockDim.x)
	{
		int output_offset_base = MAX_CONTACTS_WEEKEND * myPos;

		int myIdx = infected_indexes[myPos];


		threefry2x64_key_t tf_k = {{(long) SEED_DEVICE[0], (long) SEED_DEVICE[1]}};
		union{
			threefry2x64_ctr_t c;
			unsigned int i[4];
		} rand_union;
		//generate first set of random numbers

		threefry2x64_ctr_t tf_ctr_1 = {{(long) ((myPos * 2) + rand_offset), (long) ((myPos * 2) + rand_offset)}};
		rand_union.c = threefry2x64(tf_ctr_1, tf_k);

		//household: make three contacts
		kval_t household_kval_sum = 0;
		{
			int loc_offset, loc_count;
			device_lookupLocationData_singleHour(myIdx, household_lookup, household_offsets, &loc_offset, &loc_count);  //lookup location data for household
			device_selectRandomPersonFromLocation(
				myIdx, loc_offset, loc_count,rand_union.i[0], CONTACT_TYPE_HOME,
				household_people,
				output_infector_arr + output_offset_base + 0,
				output_victim_arr + output_offset_base + 0,
				output_kval_arr + output_offset_base + 0, 
				&household_kval_sum);
			device_selectRandomPersonFromLocation(
				myIdx, loc_offset, loc_count,rand_union.i[1], CONTACT_TYPE_HOME,
				household_people,
				output_infector_arr + output_offset_base + 1,
				output_victim_arr + output_offset_base + 1,
				output_kval_arr + output_offset_base + 1, 
				&household_kval_sum);
			device_selectRandomPersonFromLocation(
				myIdx, loc_offset, loc_count,rand_union.i[2], CONTACT_TYPE_HOME,
				household_people,
				output_infector_arr + output_offset_base + 2,
				output_victim_arr + output_offset_base + 2,
				output_kval_arr + output_offset_base + 2, 
				&household_kval_sum);
		}

		//we need two more random numbers for the errands
		threefry2x32_key_t tf_k_32 = {{ SEED_DEVICE[0], SEED_DEVICE[1]}};
		threefry2x32_ctr_t tf_ctr_32 = {{((myPos * 2) + rand_offset + 1),((myPos * 2) + rand_offset + 1)}};		
		union{
			threefry2x32_ctr_t c;
			unsigned int i[2];
		} rand_union_32;
		rand_union_32.c = threefry2x32(tf_ctr_32, tf_k_32);

		kval_t errand_kval_sum = 0;
		int contacts_profile = infected_errand_contacts_profile[myPos];

		{
			int loc_offset, loc_count;
			int errand_slot = WEEKEND_ERRAND_CONTACT_ASSIGNMENTS_DEVICE[contacts_profile][0]; //the errand number the contact will be made in

			device_lookupLocationData_weekendErrand(		//lookup the location data for this errand: we just need the offset and count
				myPos, errand_slot, 
				infected_errand_hours, infected_errand_destinations, 
				errand_loc_offsets, number_locations, 
				errand_populationCount_exclusiveScan, 
				&loc_offset, &loc_count);
			device_selectRandomPersonFromLocation(			//select a random person at the location
				myIdx, loc_offset, loc_count, rand_union_32.i[0], CONTACT_TYPE_ERRAND,
				errand_people,
				output_infector_arr + output_offset_base + 3,
				output_victim_arr + output_offset_base + 3,
				output_kval_arr + output_offset_base + 3,
				&errand_kval_sum);
		}
		{
			//do it again for the second errand contact
			int loc_offset, loc_count;
			int errand_slot = WEEKEND_ERRAND_CONTACT_ASSIGNMENTS_DEVICE[contacts_profile][1];		
			device_lookupLocationData_weekendErrand(			//lookup the location data for this errand
				myPos, errand_slot, 
				infected_errand_hours, infected_errand_destinations, 
				errand_loc_offsets, number_locations, 
				errand_populationCount_exclusiveScan, 
				&loc_offset, &loc_count);
			device_selectRandomPersonFromLocation(			//select a random person at the location
				myIdx, loc_offset, loc_count, rand_union_32.i[1], CONTACT_TYPE_ERRAND,
				errand_people,
				output_infector_arr + output_offset_base + 4,
				output_victim_arr + output_offset_base + 4,
				output_kval_arr + output_offset_base + 4,
				&errand_kval_sum);
		}

		output_kval_sum_arr[myPos] = household_kval_sum + errand_kval_sum;
	}
}

/// <summary> given an index, look up the location and fetch the offset/count data from the memory array </summary>
/// <param name="myIdx">Input: Index of the infector to look up</param>
/// <param name="lookup_arr">Input: Pointer to an array containing all infector locations</param>
/// <param name="loc_offset_arr">Input: Pointer to an array containing location offsets</param>
/// <param name="loc_offset">Output value: offset to first person in infector's location</param>
/// <param name="loc_count">Output value: number of people at infector's location</param>
__device__ void device_lookupLocationData_singleHour(int myIdx, int * lookup_arr, int * loc_offset_arr, int * loc_offset, int * loc_count)
{
	int myLoc = lookup_arr[myIdx];

	//NOTE: these arrays have the final number_locs+1 value set, so we do not need to do the trick for the last location
	(*loc_offset) = loc_offset_arr[myLoc];
	(*loc_count) = loc_offset_arr[myLoc + 1] - loc_offset_arr[myLoc];
}

/// <summary> given an index, look up the location and fetch the offset/count/max_contacts values from the memory array </summary>
/// <param name="myIdx">Input: Index of the infector to look up</param>
/// <param name="lookup_arr">Input: Pointer to an array containing all infector locations</param>
/// <param name="loc_offset_arr">Input: Pointer to an array containing a location offsets</param>
/// <param name="loc_max_contacts_arr">Input: pointer to an array containing max_contact values</param>
/// <param name="loc_offset">Output: offset to first person in infector's location</param>
/// <param name="loc_count">Output: number of people at infector's location</param>
/// <param name="loc_max_contacts">Output: max_contacts for infector's location</param>
__device__ void device_lookupLocationData_singleHour(int myIdx, int * lookup_arr, int * loc_offset_arr, int * loc_max_contacts_arr, int * loc_offset, int * loc_count, int * loc_max_contacts)
{
	int myLoc = lookup_arr[myIdx];

	//NOTE: these arrays have the final number_locs+1 value set, so we do not need to do the trick for the last location
	(*loc_offset) = loc_offset_arr[myLoc];
	(*loc_count) = loc_offset_arr[myLoc + 1] - loc_offset_arr[myLoc];
	(*loc_max_contacts) = loc_max_contacts_arr[myLoc];
}

/// <summary> Look up the location information for an infected person for weekend errands </summary>
/// <param name="myPos">Input: Which of the N infected individuals we are working with, 0 <= myPos <= infected_count</param>
/// <param name="errand_slot">Input: Infected go on three errands, this is which of the three the contact is for </param>
/// <param name="infected_hour_val_arr">Input:Array containing hour numbers that infected will go on errands in</param>
/// <param name="infected_hour_destination_arr">Input: Array containing the location number the errands are to</param>
__device__ void device_lookupLocationData_weekendErrand(int myPos, int errand_slot, int * infected_hour_val_arr, int * infected_hour_destination_arr, int * loc_offset_arr, int number_locations, int * hour_populationCount_exclusiveScan, int * output_location_offset, int * output_location_count)
{
	//this code is overall very similar to the multi-hour code for weekday, but modified to handle variable numbers
	//of people per hour (since errands are randomly generated between 10 hours)

	int hour_data_position = (myPos * NUM_WEEKEND_ERRANDS) + errand_slot;

	int hour = infected_hour_val_arr[hour_data_position];			//which hour the errand will be made on
	int myLoc = infected_hour_destination_arr[hour_data_position];	//destination of the errand

	//location offsets are stored in collated format, eg for 3 locations and 2 hours:
	// 1 2 3 1 2 3
	int location_offset_position = (hour * number_locations) + myLoc;

	int loc_offset = loc_offset_arr[location_offset_position];
	int next_loc_offset;
	
	//next_loc_offset is normally loc_offset_arr[loc_offset_pos + 1] but the last location is a special case
	if(myLoc == number_locations - 1)
	{
		//next_loc_offset = number of people present this hour
		int number_people_thisHour = hour_populationCount_exclusiveScan[hour + 1] - hour_populationCount_exclusiveScan[hour];
		next_loc_offset = number_people_thisHour;
	}
	else
		next_loc_offset = loc_offset_arr[location_offset_position + 1];

	(*output_location_count) = next_loc_offset - loc_offset;

	//the hourly binary searches are only the offset within the hour, so we need to add the offset to the first person for this hour
	loc_offset += hour_populationCount_exclusiveScan[hour];
	(*output_location_offset) = loc_offset;
}

/// <summary>Gets location data and number of contacts desired from a multi-hour errand array</summary>
/// <param name="myPos">Input: Which of the N infected individuals we are working with, 0 <= myPos <= infected_count</param>
/// <param name="hour">Input: Which hour we are looking up information for, 0 < hour <= <paramref name="number_hours" /></param>
/// <param name="infected_loc_arr">Input: Pointer to an array containing the errand destinations of infected in packed arrangement</param>
/// <param name="loc_offset_arr>Input: pointer to an array containing location offsets in collated arrangement</param>
/// <param name="number_locations>Input: the number of locations (excluding households) in the simulation</param>
/// <param name="number_people">Input: number of people present (must be same all hours)</param>
/// <param name="contacts_desired_lookup">Input: pointer to an array containing the number of contacts desired for each hour, in packed form</param>
/// <param name="number_hours">Input: The number of hours stored in the multi-hour array, probably NUM_WEEKEND_ERRAND_HOURS or NUM_WEEKDAY_ERRAND_HOURS</param>
/// <param name="output_location_offset">Output: the offset from the start of the array to the first person at this location for this hour</param>
/// <param name="output_location_count">Output: the number of people at this location for this hour</param>
/// <param name="output_contacts_desired">Output: the number of contacts we will make this hour</param>
__device__ void device_lookupInfectedLocation_multiHour(int myPos, int hour, int * infected_loc_arr, int * loc_offset_arr, int number_locations, int number_people, int * contacts_desired_lookup, int number_hours, int * output_loc_offset, int * output_loc_count, int * output_contacts_desired)
{
	//infected locations and contacts_desired are stored packed, eg for infected_idx 1,2
	// 1 1 2 2

	int infected_loc_offset = (number_hours * myPos) + hour;	//position of this person's location within the infected_location array
	int myLoc = infected_loc_arr[infected_loc_offset];		//which of the 1300 locations this person is at for this hour

	*output_contacts_desired = contacts_desired_lookup[infected_loc_offset];	//output the number of contacts this person will make this hour


	//location offsets are stored in collated format, eg for locations 1 2 3
	// 1 2 3 1 2 3

	int loc_offset_position = (hour * number_locations) + myLoc;	//position of the location's offset within the multi-hour offset array

	int loc_o = loc_offset_arr[loc_offset_position];
	int next_loc_o;		//stores loc_offset_arr[loc_offset_position + 1]

	//hack: next_loc_o normally gets loc_offset_arr[loc_offset_pos + 1], but this array is not set up with an extra slot for the last location
	//therefore, if we are at the last location, we need to fudge this value
	if(myLoc == number_locations - 1)
		next_loc_o = number_people;
	else
		next_loc_o = loc_offset_arr[loc_offset_position + 1];

	*output_loc_count = next_loc_o - loc_o;	//calculate the number of people at this location

	//hack: the binary search is done on a per-hour basis, so we need to offset to the first person of this hour
	loc_o += (hour * number_people);
	*output_loc_offset = loc_o;
}


__device__ void device_selectRandomPersonFromLocation(int infector_idx, int loc_offset, int loc_count, unsigned int rand_val, int desired_kval, int * location_people_arr, int * output_infector_idx_arr, int * output_victim_idx_arr, int * output_kval_arr, kval_t * output_kval_sum)
{
	//start with null data
	int victim_idx = NULL_PERSON_INDEX;
	int contact_type = CONTACT_TYPE_NONE;

	//if there is only one person, keep the null data, else select one other person who is not our infector
	if(loc_count > 1)
	{
		int victim_offset = rand_val % loc_count;	//select a random person between 0 and loc_count
		victim_idx = location_people_arr[loc_offset + victim_offset];	//get the index

		//if we have selected the infector, we need to get a different person
		if(victim_idx == infector_idx)
		{
			//get the next person
			victim_offset = victim_offset + 1;
			if(victim_offset == loc_count)		//wrap around if needed
				victim_offset = 0;
			victim_idx = location_people_arr[loc_offset + victim_offset];
		}

		contact_type = desired_kval;
	}

	//write data into output memory locations
	(*output_infector_idx_arr) = infector_idx;
	(*output_victim_idx_arr) = victim_idx;
	(*output_kval_arr) = contact_type;

	//increment the kval sum by the kval of this contact type
	if(contact_type != CONTACT_TYPE_NONE)
		*output_kval_sum += KVAL_LOOKUP_DEVICE[contact_type];
}

//write a null contact to the memory locations
__device__ void device_nullFillContact(int myIdx, int * output_infector_idx, int * output_victim_idx, int * output_kval)
{
	(*output_infector_idx) = myIdx;
	(*output_victim_idx) = NULL_PERSON_INDEX;
	(*output_kval) = CONTACT_TYPE_NONE;
}

__device__ void device_lookupInfectedErrand_weekend(int myPos, int hour_slot,
													int * inf_hour_arr, int * inf_location_arr, 
													int * output_hour, int * output_location)
{
	int offset = (myPos * NUM_WEEKEND_ERRANDS) + hour_slot;

	*output_hour = inf_hour_arr[offset];
	*output_location = inf_location_arr[offset];
}


__global__ void kernel_assignAfterschoolLocations_wholeDay(int * child_indexes_arr, int * output_array, int number_children, int number_people, randOffset_t rand_offset)
{
	threefry2x64_key_t tf_k = {{SEED_DEVICE[0], SEED_DEVICE[1]}};
	union{
		threefry2x64_ctr_t c;
		unsigned int i[4];
	} u;

	//get the number of afterschool locations and their offset in the business array
	int afterschool_count = WORKPLACE_TYPE_COUNT_DEVICE[BUSINESS_TYPE_AFTERSCHOOL];
	int afterschool_offset = WORKPLACE_TYPE_OFFSET_DEVICE[BUSINESS_TYPE_AFTERSCHOOL];

	//for each child
	for(int myGridPos = blockIdx.x * blockDim.x + threadIdx.x;  myGridPos <= number_children / 4; myGridPos += gridDim.x * blockDim.x)
	{
		randOffset_t myRandOffset = myGridPos + rand_offset;
		threefry2x64_ctr_t tf_ctr = {{myRandOffset, myRandOffset}};
		u.c = threefry2x64(tf_ctr, tf_k);

		int myPos = myGridPos * 4;
		if(myPos < number_children)
		{
			int myIdx = child_indexes_arr[myPos];
			device_fishAfterschoolLocation(&u.i[0], number_people, afterschool_count, afterschool_offset, output_array + myIdx);
		}

		if(myPos + 1 < number_children)
		{
			int myIdx_1 = child_indexes_arr[myPos + 1];
			device_fishAfterschoolLocation(&u.i[1],number_people, afterschool_count, afterschool_offset, output_array + myIdx_1);
		}
		if(myPos + 2 < number_children)
		{
			int myIdx_2 = child_indexes_arr[myPos + 2];
			device_fishAfterschoolLocation(&u.i[2],number_people, afterschool_count, afterschool_offset,output_array + myIdx_2);
		}
		if(myPos + 3 < number_children)
		{
			int myIdx_3 = child_indexes_arr[myPos + 3];
			device_fishAfterschoolLocation(&u.i[3], number_people, afterschool_count, afterschool_offset, output_array + myIdx_3);
		}
	}
}

__device__ void device_fishAfterschoolLocation(unsigned int * rand_val, int number_people, int afterschool_count, int afterschool_offset, int * output_schedule)
{
	//turn random number into fraction between 0 and 1
	float frac = (float) *rand_val / UNSIGNED_MAX;

	int business_num = frac * afterschool_count;		//find which afterschool location they're at, between 0 <= X < count
	
	if(business_num >= afterschool_count)
		business_num = afterschool_count - 1;

	business_num = business_num + afterschool_offset;		//add the offset to the first afterschool location

	*output_schedule = business_num;					//store in the indicated output location
	*(output_schedule + number_people) = business_num;	//children go to the same location for both hours, so put it in their second errand slot
}


__global__ void kernel_assignErrandLocations_weekday_wholeDay(int * adult_indexes_arr, int number_adults, int number_people, int * output_arr, randOffset_t rand_offset)
{
	threefry2x64_key_t tf_k = {{SEED_DEVICE[0], SEED_DEVICE[1]}};
	union{
		threefry2x64_ctr_t c;
		unsigned int i[4];
	} u;

	//for each adult
	for(int myGridPos = blockIdx.x * blockDim.x + threadIdx.x;  myGridPos <= number_adults / 2; myGridPos += gridDim.x * blockDim.x)
	{
		randOffset_t myRandOffset = myGridPos + rand_offset;
		threefry2x64_ctr_t tf_ctr = {{myRandOffset, myRandOffset}};
		u.c = threefry2x64(tf_ctr, tf_k);

		int myPos = myGridPos * 2;

		//fish out a destination
		if(myPos < number_adults)
		{
			int myAdultIdx_1 = adult_indexes_arr[myPos];
			device_fishWeekdayErrandDestination(&u.i[0], &output_arr[myAdultIdx_1]);	//for adult index i, output the destination to arr[i]
			device_fishWeekdayErrandDestination(&u.i[1], &output_arr[myAdultIdx_1 + number_people]);	//output a second destination to arr[i] for the second hour
		}
		//if still in bounds, assign another person
		if(myPos + 1 < number_adults)
		{
			int myAdultIdx_2 = adult_indexes_arr[myPos + 1];
			device_fishWeekdayErrandDestination(&u.i[2], &output_arr[myAdultIdx_2]);
			device_fishWeekdayErrandDestination(&u.i[3], &output_arr[myAdultIdx_2 + number_people]);
		}
	}
}


__device__ void device_fishWeekdayErrandDestination(unsigned int * rand_val, int * output_destination)
{
	float yval = (float) *rand_val / UNSIGNED_MAX;

	int row = FIRST_WEEKDAY_ERRAND_ROW; //which business type

	while(yval > WORKPLACE_TYPE_WEEKDAY_ERRAND_PDF_DEVICE[row] && row < (NUM_BUSINESS_TYPES - 1))
	{
		yval -= WORKPLACE_TYPE_WEEKDAY_ERRAND_PDF_DEVICE[row];
		row++;
	}

	//figure out which business of this type we're at
	float frac = yval / WORKPLACE_TYPE_WEEKDAY_ERRAND_PDF_DEVICE[row];
	int type_count = WORKPLACE_TYPE_COUNT_DEVICE[row];
	int business_num = frac * type_count;

	if(business_num >= type_count)
		business_num = type_count - 1;

	//add the offset to the first business of this type 
	int type_offset = WORKPLACE_TYPE_OFFSET_DEVICE[row];

	*output_destination = business_num + type_offset;
}


inline const char * lookup_contact_type(int contact_type)
{
	switch(contact_type)
	{
	case 0:
		return "CONTACT_TYPE_NONE";
	case 1:
		return "CONTACT_TYPE_WORKPLACE";
	case 2:
		return "CONTACT_TYPE_SCHOOL";
	case 3:
		return "CONTACT_TYPE_ERRAND";
	case 4:
		return "CONTACT_TYPE_AFTERSCHOOL";
	case 5:
		return "CONTACT_TYPE_HOME";
	default:
		return "BAD_CONTACT_TYPE_NUM";
	}
}

inline const char * lookup_workplace_type(int workplace_type)
{
	switch(workplace_type)
	{
	case 0:
		return "home";
	case 1:
		return "factory";
	case 2:
		return "office";
	case 3:
		return "preschool";
	case 4:
		return "elementary school";
	case 5:
		return "middle school";
	case 6:
		return "highschool";
	case 7:
		return "university";
	case 8:
		return "afterschool center";
	case 9:
		return "grocery store";
	case 10:
		return "other store";
	case 11:
		return "restaurant";
	case 12:
		return "entertainment";
	case 13:
		return "church";
	default:
		return "INVALID WORKPLACE TYPE";
	}
}

const char * lookup_age_type(int age_type)
{
	switch(age_type)
	{
	case 0:
		return "AGE_5";
	case 1:
		return "AGE_9";
	case 2:
		return "AGE_14";
	case 3:
		return "AGE_17";
	case 4:
		return "AGE_22";
	case 5:
		return "AGE_ADULT";
	default:
		return "INVALID AGE CODE";
	}
}

__global__ void kernel_assignErrands_weekend(int * people_indexes_arr, int * errand_hours_arr, int * errand_destination_arr, int num_people, randOffset_t rand_offset)
{
	const int RAND_COUNTS_CONSUMED = 2;	//one for hours, one for destinations

	for(int myPos = blockIdx.x * blockDim.x + threadIdx.x;  myPos < num_people; myPos += gridDim.x * blockDim.x)
	{
		int offset = myPos * NUM_WEEKEND_ERRANDS;
		randOffset_t myRandOffset = rand_offset + (myPos * RAND_COUNTS_CONSUMED);
		
		device_copyPeopleIndexes_weekend_wholeDay(people_indexes_arr + offset, myPos);
		device_assignErrandHours_weekend_wholeDay(errand_hours_arr + offset, myRandOffset);
		device_assignErrandDestinations_weekend_wholeDay(errand_destination_arr + offset, myRandOffset + 1);
	}
}

__device__ void device_assignErrandDestinations_weekend_wholeDay(int * errand_destination_ptr, int my_rand_offset)
{
	threefry2x64_key_t tf_k = {{(long) SEED_DEVICE[0], (long) SEED_DEVICE[1]}};
	union{
		threefry2x64_ctr_t c;
		unsigned int i[4];
	} rand_union;

	threefry2x64_ctr_t tf_ctr = {{((long)my_rand_offset), ((long) my_rand_offset)}};
	rand_union.c = threefry2x64(tf_ctr, tf_k);

	device_fishWeekendErrandDestination(&rand_union.i[0], errand_destination_ptr);
	device_fishWeekendErrandDestination(&rand_union.i[1], errand_destination_ptr+1);
	device_fishWeekendErrandDestination(&rand_union.i[2], errand_destination_ptr+2);
}

__global__ void kernel_doInfectedSetup_weekend(int * input_infected_indexes_ptr, int * input_errand_hours_ptr, int * input_errand_destinations_ptr,
											   int * output_infected_hour_ptr, int * output_infected_dest_ptr, int * output_contacts_desired_ptr,
											   int num_infected, randOffset_t rand_offset)
{
	threefry2x64_key_t tf_k = {{(long) SEED_DEVICE[0], (long) SEED_DEVICE[1]}};
	union{
		threefry2x64_ctr_t c;
		unsigned int i[4];
	} rand_union;

	for(int myGridPos = blockIdx.x * blockDim.x + threadIdx.x;  myGridPos <= num_infected / 4; myGridPos += gridDim.x * blockDim.x)
	{
		randOffset_t myRandOffset = rand_offset + myGridPos;
		threefry2x64_ctr_t tf_ctr = {{myRandOffset, myRandOffset}};
		rand_union.c = threefry2x64(tf_ctr, tf_k);

		int myPos = myGridPos * 4;
		if(myPos < num_infected)
		{
			device_doAllInfectedSetup_weekend(&rand_union.i[0], 
				myPos, input_infected_indexes_ptr, 
				input_errand_hours_ptr, input_errand_destinations_ptr,
				output_infected_hour_ptr, output_infected_dest_ptr, output_contacts_desired_ptr);
		}

		if(myPos + 1 < num_infected)
		{
			device_doAllInfectedSetup_weekend(&rand_union.i[1], 
				myPos+1, input_infected_indexes_ptr, 
				input_errand_hours_ptr, input_errand_destinations_ptr,
				output_infected_hour_ptr, output_infected_dest_ptr, output_contacts_desired_ptr);
		}

		if(myPos + 2  < num_infected)
		{
			device_doAllInfectedSetup_weekend(&rand_union.i[2], 
				myPos+2, input_infected_indexes_ptr, 
				input_errand_hours_ptr, input_errand_destinations_ptr,
				output_infected_hour_ptr, output_infected_dest_ptr, output_contacts_desired_ptr);
		}

		if(myPos + 3 < num_infected)
		{
			device_doAllInfectedSetup_weekend(&rand_union.i[3], 
				myPos+3, input_infected_indexes_ptr, 
				input_errand_hours_ptr, input_errand_destinations_ptr,
				output_infected_hour_ptr, output_infected_dest_ptr, output_contacts_desired_ptr);
		}
	}
}

__device__ void device_copyInfectedErrandLocs_weekend(int * input_hours_ptr, int * input_dests_ptr, int * output_hours_ptr, int * output_dests_ptr)
{
	output_hours_ptr[0] = input_hours_ptr[0];
	output_hours_ptr[1] = input_hours_ptr[1];
	output_hours_ptr[2] = input_hours_ptr[2];

	output_dests_ptr[0] = input_dests_ptr[0];
	output_dests_ptr[1] = input_dests_ptr[1];
	output_dests_ptr[2] = input_dests_ptr[2];
}

__device__ void device_doAllInfectedSetup_weekend(unsigned int * rand_val, int myPos, int * infected_indexes_arr, int * input_hours_arr, int * input_dests_arr, int * output_hours_arr, int * output_dests_arr, int * output_contacts_desired_arr)
{
	int myIdx = infected_indexes_arr[myPos];
	int input_offset = NUM_WEEKEND_ERRANDS * myIdx;
	int output_offset = NUM_WEEKEND_ERRANDS * myPos;

	device_copyInfectedErrandLocs_weekend(
		input_hours_arr + input_offset,
		input_dests_arr + input_offset,
		output_hours_arr + output_offset,
		output_dests_arr + output_offset);

	const int NUM_POSSIBLE_CONTACT_ASSIGNMENTS = 6;
	int profile = *rand_val % NUM_POSSIBLE_CONTACT_ASSIGNMENTS;
	output_contacts_desired_arr[myPos] = profile;
}


__global__ void kernel_countInfectedStatus(int * pandemic_status_array, int * seasonal_status_array, int num_people, int * output_pandemic_counts, int * output_seasonal_counts)
{
	int tid = threadIdx.x;
	__shared__ int pandemic_reduction_array[COUNTING_GRID_THREADS][8];
	__shared__ int seasonal_reduction_array[COUNTING_GRID_THREADS][8];

	//zero out the counters
	pandemic_reduction_array[tid][0] = 0;
	pandemic_reduction_array[tid][1] = 0;
	pandemic_reduction_array[tid][2] = 0;
	pandemic_reduction_array[tid][3] = 0;
	pandemic_reduction_array[tid][4] = 0;
	pandemic_reduction_array[tid][5] = 0;
	pandemic_reduction_array[tid][6] = 0;
	pandemic_reduction_array[tid][7] = 0;

	seasonal_reduction_array[tid][0] = 0;
	seasonal_reduction_array[tid][1] = 0;
	seasonal_reduction_array[tid][2] = 0;
	seasonal_reduction_array[tid][3] = 0;
	seasonal_reduction_array[tid][4] = 0;
	seasonal_reduction_array[tid][5] = 0;
	seasonal_reduction_array[tid][6] = 0;
	seasonal_reduction_array[tid][7] = 0;

	//valid status condition codes are between -2 and 5 inclusive, get a pointer to where status 0 should go
	int * pandemic_pointer = &pandemic_reduction_array[tid][2];
	int * seasonal_pointer = &seasonal_reduction_array[tid][2];

	//count all statuses
	for(int myPos = blockIdx.x * blockDim.x + threadIdx.x;  myPos < num_people; myPos += gridDim.x * blockDim.x)
	{
		int status_pandemic = pandemic_status_array[myPos];
		pandemic_pointer[status_pandemic]++;
		int status_seasonal = seasonal_status_array[myPos];
		seasonal_pointer[status_seasonal]++;
	}
	__syncthreads();   //wait for all threads to finish, or reduction will hit a race condition
	

	//do reduction
	for(int offset = blockDim.x / 2; offset > 0;  offset /= 2)
	{
		if(tid < offset)
		{
			pandemic_reduction_array[tid][0] += pandemic_reduction_array[tid+offset][0];
			pandemic_reduction_array[tid][1] += pandemic_reduction_array[tid+offset][1];
			pandemic_reduction_array[tid][2] += pandemic_reduction_array[tid+offset][2];
			pandemic_reduction_array[tid][3] += pandemic_reduction_array[tid+offset][3];
			pandemic_reduction_array[tid][4] += pandemic_reduction_array[tid+offset][4];
			pandemic_reduction_array[tid][5] += pandemic_reduction_array[tid+offset][5];
			pandemic_reduction_array[tid][6] += pandemic_reduction_array[tid+offset][6];
			pandemic_reduction_array[tid][7] += pandemic_reduction_array[tid+offset][7];

			seasonal_reduction_array[tid][0] += seasonal_reduction_array[tid+offset][0];
			seasonal_reduction_array[tid][1] += seasonal_reduction_array[tid+offset][1];
			seasonal_reduction_array[tid][2] += seasonal_reduction_array[tid+offset][2];
			seasonal_reduction_array[tid][3] += seasonal_reduction_array[tid+offset][3];
			seasonal_reduction_array[tid][4] += seasonal_reduction_array[tid+offset][4];
			seasonal_reduction_array[tid][5] += seasonal_reduction_array[tid+offset][5];
			seasonal_reduction_array[tid][6] += seasonal_reduction_array[tid+offset][6];
			seasonal_reduction_array[tid][7] += seasonal_reduction_array[tid+offset][7];
		}
		__syncthreads();
	}

	//thread 0 stores results
	if(tid == 0)
	{
		atomicAdd(output_pandemic_counts + 0, pandemic_reduction_array[0][0]);
		atomicAdd(output_pandemic_counts + 1, pandemic_reduction_array[0][1]);
		atomicAdd(output_pandemic_counts + 2, pandemic_reduction_array[0][2]);
		atomicAdd(output_pandemic_counts + 3, pandemic_reduction_array[0][3]);
		atomicAdd(output_pandemic_counts + 4, pandemic_reduction_array[0][4]);
		atomicAdd(output_pandemic_counts + 5, pandemic_reduction_array[0][5]);
		atomicAdd(output_pandemic_counts + 6, pandemic_reduction_array[0][6]);
		atomicAdd(output_pandemic_counts + 7, pandemic_reduction_array[0][7]);

		atomicAdd(output_seasonal_counts + 0, seasonal_reduction_array[0][0]);
		atomicAdd(output_seasonal_counts + 1, seasonal_reduction_array[0][1]);
		atomicAdd(output_seasonal_counts + 2, seasonal_reduction_array[0][2]);
		atomicAdd(output_seasonal_counts + 3, seasonal_reduction_array[0][3]);
		atomicAdd(output_seasonal_counts + 4, seasonal_reduction_array[0][4]);
		atomicAdd(output_seasonal_counts + 5, seasonal_reduction_array[0][5]);
		atomicAdd(output_seasonal_counts + 6, seasonal_reduction_array[0][6]);
		atomicAdd(output_seasonal_counts + 7, seasonal_reduction_array[0][7]);
	}
}

struct isInfectedPred
{
	__device__ bool operator() (thrust::tuple<int,int> status_tuple)
	{
		int status_seasonal = thrust::get<0>(status_tuple);
		int status_pandemic = thrust::get<1>(status_tuple);

		return status_pandemic >= 0 || status_seasonal >= 0;
	}
};

void PandemicSim::daily_buildInfectedArray_global()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "daily_buildInfectedArray_global");

	thrust::counting_iterator<int> count_it(0);
	IntIterator infected_indexes_end = thrust::copy_if(
		count_it, count_it + number_people,
		thrust::make_zip_iterator(thrust::make_tuple(
			people_status_pandemic.begin(), people_status_seasonal.begin())),
		infected_indexes.begin(),
		isInfectedPred());

	infected_count = infected_indexes_end - infected_indexes.begin();

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, infected_count);
}


struct recoverInfected_pred
{
	int recover_infections_from_day;
	__device__ bool operator() (thrust::tuple<int,int> status_obj)
	{
		int status_type = thrust::get<0>(status_obj);

		//if there is no active infection, do not try to set recovered status
		if(status_type < 0)
			return false;

		//get the day this infection began
		int day_infection_began = thrust::get<1>(status_obj);
			
		//return true if it matches the day we're looking for, otherwise false
		return recover_infections_from_day == day_infection_began;
	}
};

void PandemicSim::daily_recoverInfected_new()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "daily_recoverInfected");

	int recover_day = (current_day + 1) - CULMINATION_PERIOD;
//	if(recover_day >= 0)
	if(1)
	{
		recoverInfected_pred recover_obj;
		recover_obj.recover_infections_from_day = recover_day;

			thrust::replace_if(
			thrust::make_permutation_iterator(people_status_pandemic.begin(), infected_indexes.begin()),
			thrust::make_permutation_iterator(people_status_pandemic.begin(), infected_indexes.begin() + infected_count),
			thrust::make_zip_iterator(thrust::make_tuple(
				thrust::make_permutation_iterator(people_status_pandemic.begin(), infected_indexes.begin()),
				thrust::make_permutation_iterator(people_days_pandemic.begin(), infected_indexes.begin()))),
			recover_obj,
			STATUS_RECOVERED);

		thrust::replace_if(
			thrust::make_permutation_iterator(people_status_seasonal.begin(), infected_indexes.begin()),
			thrust::make_permutation_iterator(people_status_seasonal.begin(), infected_indexes.begin() + infected_count),
			thrust::make_zip_iterator(thrust::make_tuple(
				thrust::make_permutation_iterator(people_status_seasonal.begin(), infected_indexes.begin()),
				thrust::make_permutation_iterator(people_days_seasonal.begin(), infected_indexes.begin()))),
			recover_obj,
			STATUS_RECOVERED);
	}

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, infected_count);
}

void PandemicSim::final_countReproduction()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(-1,"final_countReproduction");

	thrust::sort(people_gens_pandemic.begin(), people_gens_pandemic.end());
	thrust::sort(people_gens_seasonal.begin(), people_gens_seasonal.end());

	thrust::counting_iterator<int> count_it(0);

	vec_t pandemic_gen_counts(MAX_DAYS + 1);
	pandemic_gen_counts[MAX_DAYS] = number_people;
	thrust::lower_bound(
		people_gens_pandemic.begin(), people_gens_pandemic.end(),
		count_it, count_it + MAX_DAYS,
		pandemic_gen_counts.begin());

	vec_t seasonal_gen_counts(MAX_DAYS + 1);
	seasonal_gen_counts[MAX_DAYS] = number_people;
	thrust::lower_bound(
		people_gens_seasonal.begin(), people_gens_seasonal.end(),
		count_it, count_it + MAX_DAYS,
		seasonal_gen_counts.begin());

	//copy to host
	h_vec h_pandemic_gens = pandemic_gen_counts;
	h_vec h_seasonal_gens = seasonal_gen_counts;

	FILE * fReproduction;
	if(OUTPUT_FILES_IN_PARENTDIR)
		fReproduction = fopen("../output_rn.csv","w");
	else
		fReproduction = fopen("output_rn.csv","w");

	fprintf(fReproduction, "gen,gen_size_p,rn_p,gen_size_s,rn_s\n");
	//calculate reproduction numbers
	for(int gen = 0; gen < MAX_DAYS-1; gen++)
	{
		int gen_size_p = h_pandemic_gens[gen+1] - h_pandemic_gens[gen];
		int next_gen_size_p = h_pandemic_gens[gen+2] - h_pandemic_gens[gen+1];
		float rn_p = (float) next_gen_size_p / gen_size_p;

		int gen_size_s = h_seasonal_gens[gen+1] - h_seasonal_gens[gen];
		int next_gen_size_s = h_seasonal_gens[gen+2] - h_seasonal_gens[gen+1];
		float rn_s = (float) next_gen_size_s / gen_size_s;

		fprintf(fReproduction, "%d,%d,%f,%d,%f\n",
			gen, gen_size_p, rn_p, gen_size_s, rn_s);
	}
	fclose(fReproduction);

	if(PROFILE_SIMULATION)
		profiler.endFunction(-1,number_people);
}

__device__ void device_checkActionAndWrite(bool infects_pandemic, bool infects_seasonal, int victim, int * pandemic_status_arr, int * seasonal_status_arr, int * dest_ptr)
{
	if(infects_pandemic)
	{
		int victim_status_p = pandemic_status_arr[victim];
		if(victim_status_p != STATUS_SUSCEPTIBLE)
			infects_pandemic = false;
	}
	if(infects_seasonal)
	{
		int victim_status_s = seasonal_status_arr[victim];
		if(victim_status_s != STATUS_SUSCEPTIBLE)
			infects_seasonal = false;
	}

	if(infects_pandemic && infects_seasonal)
		*dest_ptr = ACTION_INFECT_BOTH;
	else if(infects_pandemic)
		*dest_ptr = ACTION_INFECT_PANDEMIC;
	else if(infects_seasonal)
		*dest_ptr = ACTION_INFECT_SEASONAL;
}

__device__ float device_calculateInfectionProbability(int profile, int day_of_infection, int strain, kval_t kval_sum)
{
	if(kval_sum == 0)
		kval_sum = 1;

	//alpha: fraction of infectiousness that will occur on this day of infection for this profile
	float alpha = VIRAL_SHEDDING_PROFILES_DEVICE[profile][day_of_infection];

	//strain_adjustment_factor: equals rn_base / ((1.0-asymp) * pct_symptomatic)
	float strain_adjustment_factor = INFECTIOUSNESS_FACTOR_DEVICE[strain];

	//the average number of infections this person will generate today
	float average_infections_today = alpha * strain_adjustment_factor;

	//the average chance for infection for a contact with kappa = 1.0
	float normalized_infection_prob = (float) average_infections_today / kval_sum;

	return normalized_infection_prob;
}


__global__ void kernel_householdTypeAssignment(int * hh_type_array, int num_households, randOffset_t rand_offset)
{
	threefry2x64_key_t tf_k = {{(long) SEED_DEVICE[0], (long) SEED_DEVICE[1]}};
	union{
		threefry2x64_ctr_t c;
		unsigned int i[4];
	} rand_union;

	for(int myGridPos = blockIdx.x * blockDim.x + threadIdx.x;  myGridPos <= num_households / 4; myGridPos += gridDim.x * blockDim.x)
	{
		randOffset_t myRandOffset = rand_offset + myGridPos;
		threefry2x64_ctr_t tf_ctr = {{myRandOffset, myRandOffset}};
		rand_union.c = threefry2x64(tf_ctr,tf_k);

		int myPos = myGridPos * 4;

		if(myPos < num_households)
			hh_type_array[myPos+0] = device_setup_fishHouseholdType(rand_union.i[0]);

		if(myPos + 1 < num_households)
			hh_type_array[myPos+1] = device_setup_fishHouseholdType(rand_union.i[1]);

		if(myPos + 2 < num_households)
			hh_type_array[myPos+2] = device_setup_fishHouseholdType(rand_union.i[2]);

		if(myPos + 3 < num_households)
			hh_type_array[myPos+3] = device_setup_fishHouseholdType(rand_union.i[3]);
	}
}

__device__ int device_setup_fishHouseholdType(unsigned int rand_val)
{
	float y = (float) rand_val / UNSIGNED_MAX;

	int row = 0;
	while(y > HOUSEHOLD_TYPE_CDF_DEVICE[row] && row < HH_TABLE_ROWS - 1)
		row++;

	return row;
}



__device__ int device_setup_fishWorkplace(unsigned int rand_val)
{
	float y = (float) rand_val / UNSIGNED_MAX;

	int row = 0;
	while(WORKPLACE_TYPE_ASSIGNMENT_PDF_DEVICE[row] < y && row < NUM_BUSINESS_TYPES - 1)
	{
		y -= WORKPLACE_TYPE_ASSIGNMENT_PDF_DEVICE[row];
		row++;
	}

	//of this workplace type, which number is this?
	float frac = y / WORKPLACE_TYPE_ASSIGNMENT_PDF_DEVICE[row];
	int type_count = WORKPLACE_TYPE_COUNT_DEVICE[row];
	int business_num = frac * type_count;  //truncate to int

	if(business_num >= type_count)
		business_num = type_count - 1;

	//how many other workplaces have we gone past?
	int type_offset = WORKPLACE_TYPE_OFFSET_DEVICE[row];

	return business_num + type_offset;
}

__device__ void device_setup_fishSchoolAndAge(unsigned int rand_val, int * output_age_ptr, int * output_school_ptr)
{
	float y = (float) rand_val / RAND_MAX;

	//fish out age group and resulting school type from CDF
	int row = 0;
	while(row < CHILD_DATA_ROWS - 1 && y > CHILD_AGE_CDF_DEVICE[row])
		row++;

	int wp_type = CHILD_AGE_SCHOOLTYPE_LOOKUP_DEVICE[row];

	//of this school type, which one will this kid be assigned to?
	float frac;
	if(row == 0)
		frac = y / (CHILD_AGE_CDF_DEVICE[row]);
	else
	{
		//we need to back out a PDF from the CDF
		float pdf_here = CHILD_AGE_CDF_DEVICE[row] - CHILD_AGE_CDF_DEVICE[row - 1];
		float y_here = y - CHILD_AGE_CDF_DEVICE[row - 1];

		frac =  y_here / pdf_here;
	}

	int type_count = WORKPLACE_TYPE_COUNT_DEVICE[wp_type];
	int business_num = frac * type_count;

	if(business_num >= type_count)
		business_num = type_count - 1;

	//how many other workplaces have we gone past?
	int type_offset = WORKPLACE_TYPE_OFFSET_DEVICE[wp_type];
	*output_school_ptr = business_num + type_offset;
	*output_age_ptr = row;
}


__global__ void kernel_generateHouseholds(
	int * hh_type_array, int * adult_exscan_arr, 
	int * child_exscan_arr, int num_households, 
	int * adult_index_arr, int * child_index_arr, 
	int * household_offset_arr,
	int * people_age_arr, int * people_households_arr, int * people_workplaces_arr, randOffset_t rand_offset)
{
	threefry2x64_key_t tf_k = {{(long) SEED_DEVICE[0], (long) SEED_DEVICE[1]}};
	union{
		threefry2x64_ctr_t c;
		unsigned int i[4];
	} rand_union;

	const int rand_counts_consumed = 2;

	for(int hh = blockIdx.x * blockDim.x + threadIdx.x;  hh < num_households ; hh += gridDim.x * blockDim.x)
	{
		int adults_offset = adult_exscan_arr[hh];
		int children_offset = child_exscan_arr[hh];

		int hh_type = hh_type_array[hh];
		int adults_count = HOUSEHOLD_TYPE_ADULT_COUNT_DEVICE[hh_type];
		int children_count = HOUSEHOLD_TYPE_CHILD_COUNT_DEVICE[hh_type];

		int hh_offset = adults_offset + children_offset;
		household_offset_arr[hh] = hh_offset;

		//get random numbers
		randOffset_t myRandOffset = rand_offset + (hh * rand_counts_consumed);
		threefry2x64_ctr_t tf_ctr_1 = {{myRandOffset, myRandOffset}};
		rand_union.c = threefry2x64(tf_ctr_1, tf_k);

		for(int people_generated = 0; people_generated < adults_count; people_generated++)
		{
			int person_id = hh_offset + people_generated;
			people_households_arr[person_id] = hh;				//store the household number

			people_age_arr[person_id] = AGE_ADULT;					//mark as an adult
			people_workplaces_arr[person_id] = device_setup_fishWorkplace(rand_union.i[people_generated]);

			adult_index_arr[adults_offset + people_generated] = person_id; //store this ID in the adults index array
		}

		//get more random numbers
		threefry2x64_ctr_t tf_ctr_2 = {{myRandOffset + 1, myRandOffset + 1}};
		rand_union.c = threefry2x64(tf_ctr_2, tf_k);

		//increment the base ID number by the adults we just added
		hh_offset += adults_count;

		for(int people_generated = 0; people_generated < children_count; people_generated++)
		{
			int person_id = hh_offset + people_generated;
			people_households_arr[person_id] = hh;		//store the household number

			device_setup_fishSchoolAndAge(
				rand_union.i[people_generated],	
				people_age_arr + person_id,			//ptr into age_array
				people_workplaces_arr + person_id);		//ptr into workplace array

			child_index_arr[children_offset + people_generated] = person_id;	//store as a child
		}
	}
}


struct hh_adult_count_functor : public thrust::unary_function<int,int>
{
	__device__ int operator () (int hh_type) const
	{
		return HOUSEHOLD_TYPE_ADULT_COUNT_DEVICE[hh_type];
	}
};

struct hh_child_count_functor : public thrust::unary_function<int,int>
{
	__device__ int operator () (int hh_type) const
	{
		return HOUSEHOLD_TYPE_CHILD_COUNT_DEVICE[hh_type];
	}
};


//Sets up people's households and workplaces according to the probability functions
void PandemicSim::setup_generateHouseholds()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(-1,"setup_generateHouseholds");

	d_vec hh_types_array(number_households+1);
	int * hh_types_array_ptr = thrust::raw_pointer_cast(hh_types_array.data());

	//finish copydown of __constant__ sim data
	hipDeviceSynchronize();

	//assign household types
	kernel_householdTypeAssignment<<<cuda_householdTypeAssignmentKernel_blocks,cuda_householdTypeAssignmentKernel_threads>>>(hh_types_array_ptr, number_households,rand_offset);
	hipDeviceSynchronize();


	if(TIMING_BATCH_MODE == 0)
	{
		int rand_counts_consumed_1 = number_households / 4;
		rand_offset += rand_counts_consumed_1;
	}

	d_vec adult_count_exclScan(number_households+1);
	d_vec child_count_exclScan(number_households+1);

	//these count_functors convert household types into the number of children/adults in that type
	//use a transform-functor to convert the HH types and take an exclusive_scan of each
	//this will let us build the adult_index and child_index arrays
	thrust::exclusive_scan(
		thrust::make_transform_iterator(hh_types_array.begin(), hh_adult_count_functor()),
		thrust::make_transform_iterator(hh_types_array.end(), hh_adult_count_functor()),
		adult_count_exclScan.begin());
	thrust::exclusive_scan(
		thrust::make_transform_iterator(hh_types_array.begin(), hh_child_count_functor()),
		thrust::make_transform_iterator(hh_types_array.end(), hh_child_count_functor()),
		child_count_exclScan.begin());
	hipDeviceSynchronize();
	
	/*
	h_vec h_hh_types = hh_types_array;
	h_vec h_child_count_exscan = child_count_exclScan;
	h_vec h_adult_count_exscan = adult_count_exclScan;
	FILE * ftemp = fopen("../households.txt","w");
	fprintf(ftemp,"i,hh_type,adult_exscan,child_exscan\n");
	for(int i = 0; i < number_households + 1; i++)
		fprintf(ftemp, "%d,%d,%d,%d\n", i, h_hh_types[i], h_adult_count_exscan[i],h_child_count_exscan[i]);
	fclose(ftemp);*/

	//the exclusive_scan of number_households+1 holds the total number of adult and children in the sim
	//(go one past the end to find the totals)
	number_adults = adult_count_exclScan[number_households];
	number_children = child_count_exclScan[number_households];
	number_people = number_adults + number_children;
	
	//now we can allocate the rest of our memory
	setup_sizeGlobalArrays();

	if(SIM_VALIDATION)
	{
		thrust::fill_n(people_ages.begin(), number_people, -1);
		thrust::fill_n(people_households.begin(), number_people, -1);
		thrust::fill_n(people_workplaces.begin(), number_people, -1);

		thrust::fill_n(household_offsets.begin(), number_people, -1);
	}

	int * adult_exscan_ptr = thrust::raw_pointer_cast(adult_count_exclScan.data());
	int * child_exscan_ptr = thrust::raw_pointer_cast(child_count_exclScan.data());

	//and then do the rest of the setup
	kernel_generateHouseholds<<<cuda_peopleGenerationKernel_blocks,cuda_peopleGenerationKernel_threads>>>(
		hh_types_array_ptr, adult_exscan_ptr, child_exscan_ptr, number_households,
		people_adults_indexes_ptr, people_child_indexes_ptr,
		household_offsets_ptr,
		people_ages_ptr, people_households_ptr, people_workplaces_ptr,
		rand_offset);
	if(TIMING_BATCH_MODE == 0)
	{
		const int rand_counts_consumed_2 = 2 * number_households;
		rand_offset += rand_counts_consumed_2;
	}

	thrust::sequence(household_people.begin(), household_people.begin() + number_people); //copy the ID numbers into the household_people table
	household_offsets[number_households] = number_people;  //put the last household_offset in position

	hipDeviceSynchronize();

	if(PROFILE_SIMULATION)
	{
		profiler.endFunction(-1,number_people);
	}
}


struct filterContacts_pred
{
	__device__ bool operator() (thrust::tuple<int,int,int> action_tuple)
	{
		int action_type = thrust::get<0>(action_tuple);

		if(action_type == ACTION_INFECT_NONE)
			return true;

		return false;
	}
};

struct actionSortOp_new
{
	__device__
		bool operator () (thrust::tuple<int,int,int> a, thrust::tuple<int,int,int> b)
	{

		int victim_a = thrust::get<2>(a);
		int victim_b = thrust::get<2>(b);

		if(victim_a != victim_b)
		{
			return victim_a < victim_b;
		}

		int action_a = thrust::get<0>(a);
		int action_b = thrust::get<0>(b);

		return action_a > action_b;
	}
};


void PandemicSim::daily_filterActions_new()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day,"daily_filterActions");

	int num_possible_contacts = is_weekend() ? MAX_CONTACTS_WEEKEND * infected_count : MAX_CONTACTS_WEEKDAY * infected_count;

	ZipIntTripleIterator actions_begin = 
		thrust::make_zip_iterator(thrust::make_tuple(
			daily_action_type.begin(), 
			daily_contact_infectors.begin(), 
			daily_contact_victims.begin()));

	//compact - filter out null contacts
	filterContacts_pred contact_filter_obj;
	ZipIntTripleIterator actions_end = thrust::remove_if(
		actions_begin,
		thrust::make_zip_iterator(thrust::make_tuple(
			daily_action_type.begin() + num_possible_contacts, 
			daily_contact_infectors.begin() + num_possible_contacts, 
			daily_contact_victims.begin() + num_possible_contacts)),
		contact_filter_obj);

//	int size_a = actions_end - actions_begin;

	//sort - by victim_id ascending, then by action code descending
	thrust::sort(actions_begin, actions_end,actionSortOp_new());
	
	//unique - remove duplicate infection actions
	actions_end = thrust::unique(actions_begin,actions_end,uniqueActionOp());
	daily_actions = actions_end - actions_begin;

	if(CONSOLE_OUTPUT)
		printf("after filtering: %d actions remaining\n", daily_actions);

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, infected_count);
}

__global__ void kernel_contactsToActions(int * infected_idx_arr, kval_t * infected_kval_sum_arr, int infected_count,
										 int * contact_victims_arr, int *contact_type_arr, int contacts_per_infector,
										 int * people_day_pandemic_arr, int * people_day_seasonal_arr,
										 int * people_status_p_arr, int * people_status_s_arr,
										 int * output_action_arr,
										 float * rand_arr_1, float * rand_arr_2, float * rand_arr_3, float * rand_arr_4,
										 int current_day, randOffset_t rand_offset)
{
	threefry2x64_key_t tf_k = {{(long) SEED_DEVICE[0], (long) SEED_DEVICE[1]}};
	union{
		threefry2x64_ctr_t c[4];
		unsigned int i[16];
	} rand_union;

	const int rand_counts_consumed = 4;

	for(int myPos = blockIdx.x * blockDim.x + threadIdx.x;  myPos < infected_count ; myPos += gridDim.x * blockDim.x)
	{
		int myIdx = infected_idx_arr[myPos];
		kval_t kval_sum = infected_kval_sum_arr[myPos];

//		if(kval_sum == 0)
//			continue;

		int status_p = people_status_p_arr[myIdx];
		int status_s = people_status_s_arr[myIdx];

		float inf_prob_p = -1.f;
		float inf_prob_s = -1.f;

		//int profile_day_p = -1;
		if(status_p >= 0)
		{
	//		int day_of_pandemic_infection = people_day_pandemic_arr[myIdx];
			int profile_day_p = current_day - people_day_pandemic_arr[myIdx];
			inf_prob_p = device_calculateInfectionProbability(status_p,profile_day_p, STRAIN_PANDEMIC,kval_sum);
		}
		//int profile_day_s = -1;
		if(status_s >= 0)
		{
			//int day_of_seasonal_infection = people_day_seasonal_arr[myIdx];
			int profile_day_s = current_day - people_day_seasonal_arr[myIdx];
			inf_prob_s = device_calculateInfectionProbability(status_s,profile_day_s, STRAIN_SEASONAL,kval_sum);
		}

		randOffset_t myRandOffset = rand_offset + (myPos * rand_counts_consumed);
		threefry2x64_ctr_t tf_ctr_1 = {{myRandOffset, myRandOffset}};
		rand_union.c[0] = threefry2x64(tf_ctr_1, tf_k);
		threefry2x64_ctr_t tf_ctr_2 = {{myRandOffset + 1, myRandOffset + 1}};
		rand_union.c[1] = threefry2x64(tf_ctr_2, tf_k);
		threefry2x64_ctr_t tf_ctr_3 = {{myRandOffset + 2, myRandOffset + 2}};
		rand_union.c[2] = threefry2x64(tf_ctr_3, tf_k);
		threefry2x64_ctr_t tf_ctr_4 = {{myRandOffset + 3, myRandOffset + 3}};
		rand_union.c[3] = threefry2x64(tf_ctr_4, tf_k);

		int contact_offset_base = contacts_per_infector * myPos;
		int rand_vals_used = 0;
		for(int contacts_processed = 0; contacts_processed < contacts_per_infector; contacts_processed++)
		{
			int contact_victim = contact_victims_arr[contact_offset_base + contacts_processed];
			int contact_type = contact_type_arr[contact_offset_base + contacts_processed];

			kval_t contact_kval = KVAL_LOOKUP_DEVICE[contact_type];

			float y_p = (float) rand_union.i[rand_vals_used++] / UNSIGNED_MAX;
			bool infects_p = y_p < (float) (inf_prob_p * contact_kval);

			float y_s = (float) rand_union.i[rand_vals_used++] / UNSIGNED_MAX;
			bool infects_s = y_s < (float) (inf_prob_s * contact_kval);

			//function handles parsing bools into an action and checking that victim is susceptible
			device_checkActionAndWrite(
				infects_p, infects_s, 
				contact_victim, 
				people_status_p_arr, people_status_s_arr,
				output_action_arr + contact_offset_base + contacts_processed);

			if(SIM_VALIDATION)
			{
				rand_arr_1[contact_offset_base + contacts_processed] = y_p;
				rand_arr_2[contact_offset_base + contacts_processed] = (float) (inf_prob_p * contact_kval);
				rand_arr_3[contact_offset_base + contacts_processed] = y_s;
				rand_arr_4[contact_offset_base + contacts_processed] = (float) (inf_prob_s * contact_kval);
			}
		}
	}
}

void PandemicSim::daily_contactsToActions_new()
{
	if(ACTION_INFECT_NONE != 0)
		throw new std::runtime_error(std::string("ACTION_INFECT_NONE must be zero for memset!"));

	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day,"daily_contactsToActions");

	int contacts_per_infector = is_weekend() ? MAX_CONTACTS_WEEKEND : MAX_CONTACTS_WEEKDAY;
	int total_contacts = contacts_per_infector * infected_count;

	kernel_contactsToActions<<<cuda_contactsToActionsKernel_blocks,cuda_contactsToActionsKernel_threads>>>(
		infected_indexes_ptr, infected_daily_kval_sum_ptr, infected_count,
		daily_contact_victims_ptr, daily_contact_kval_types_ptr, contacts_per_infector,
		people_days_pandemic_ptr, people_days_seasonal_ptr,
		people_status_pandemic_ptr, people_status_seasonal_ptr,
		daily_action_type_ptr,
		debug_contactsToActions_float1_ptr, debug_contactsToActions_float2_ptr,
		debug_contactsToActions_float3_ptr, debug_contactsToActions_float4_ptr,
		current_day, rand_offset);
	if(TIMING_BATCH_MODE == 0)
	{
		int rand_counts_consumed = 4 * infected_count;
		rand_offset += rand_counts_consumed;
	}
	hipDeviceSynchronize();

	if(SIM_VALIDATION)
	{
		debug_validateActions();
	}

	if(CONSOLE_OUTPUT)
	{
		int successful_actions = thrust::count_if(daily_action_type.begin(), daily_action_type.begin() + total_contacts, actionIsSuccessful_pred());
		printf("before filtering: %d successful infection attempts\n",successful_actions);
	}

	if(PROFILE_SIMULATION)
		profiler.endFunction(current_day, infected_count);
}

__device__ void device_assignProfile(unsigned int rand_val, int * output_status_ptr)
{
	/*
	//assign a profile between 0 and 2 inclusive
	int profile = rand_val % 3;

	//convert the rand to a float between 0 and 1
	float y = (float) rand_val / UNSIGNED_MAX;

	//if the symptomatic threshold is exceeded, make the profile asymptomatic
	if(y > PERCENT_SYMPTOMATIC_DEVICE)
		profile += 3;*/

	//*output_status_ptr = profile;
	*output_status_ptr = STATUS_INFECTED;
}

__device__ void device_doInfectionAction(
	unsigned int rand_val1, unsigned int rand_val2,
	int day_tomorrow,
	int action_type, int infector, int victim,
	int * people_status_p_arr, int * people_status_s_arr,
	int * people_gen_p_arr, int * people_gen_s_arr,
	int * people_day_p_arr, int * people_day_s_arr)
{
	if(action_type == ACTION_INFECT_BOTH || action_type == ACTION_INFECT_PANDEMIC)
	{
		//get infector's generation and increment for the victim
		int inf_gen_p = people_gen_p_arr[infector];
		people_gen_p_arr[victim] = inf_gen_p + 1;

		//mark tomorrow as their first day of infection
		people_day_p_arr[victim] = day_tomorrow;

		//assign them a profile
		device_assignProfile(rand_val1, people_status_p_arr + victim);
	}
	if(action_type == ACTION_INFECT_BOTH || action_type == ACTION_INFECT_SEASONAL)
	{
		//get infector's generation and increment for the victim
		int inf_gen_s = people_gen_s_arr[infector];
		people_gen_s_arr[victim] = inf_gen_s + 1;

		//mark tomorrow as their first day of infection
		people_day_s_arr[victim] = day_tomorrow;

		//assign them a profile
		device_assignProfile(rand_val2, people_status_s_arr + victim);
	}
}

__global__ void kernel_doInfectionActions(
	int * contact_action_arr, int * contact_victim_arr, int * contact_infector_arr,
	int action_count,
	int * people_status_p_arr, int * people_status_s_arr,
	int * people_gen_p_arr, int * people_gen_s_arr,
	int * people_day_p_arr, int * people_day_s_arr,
	int day_tomorrow, randOffset_t rand_offset)
{
	threefry2x64_key_t tf_k = {{(long) SEED_DEVICE[0], (long) SEED_DEVICE[1]}};
	union{
		threefry2x64_ctr_t c;
		unsigned int i[4];
	} rand_union;

	for(int myGridPos = blockIdx.x * blockDim.x + threadIdx.x;  myGridPos <= action_count ; myGridPos += gridDim.x * blockDim.x)
	{
		int myPos = myGridPos * 2;

		//get random numbers
		randOffset_t myRandOffset = rand_offset + myGridPos;
		threefry2x64_ctr_t tf_ctr_1 = {{myRandOffset, myRandOffset}};
		rand_union.c = threefry2x64(tf_ctr_1, tf_k);

		if(myPos < action_count)
		{
			int action_type = contact_action_arr[myPos];
			int victim = contact_victim_arr[myPos];
			int infector = contact_infector_arr[myPos];

			device_doInfectionAction(
				rand_union.i[0],rand_union.i[1], 
				day_tomorrow,
				action_type, infector, victim,
				people_status_p_arr, people_status_s_arr,
				people_gen_p_arr,people_gen_s_arr,
				people_day_p_arr, people_day_s_arr);
		}
		if(myPos + 1 < action_count)
		{
			int action_type = contact_action_arr[myPos+1];
			int victim = contact_victim_arr[myPos+1];
			int infector = contact_infector_arr[myPos+1];

			device_doInfectionAction(
				rand_union.i[2],rand_union.i[3], 
				day_tomorrow,
				action_type, infector, victim,
				people_status_p_arr, people_status_s_arr,
				people_gen_p_arr,people_gen_s_arr,
				people_day_p_arr, people_day_s_arr);
		}
	}

}


void PandemicSim::daily_doInfectionActions()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(current_day, "daily_doInfectionActions");

	kernel_doInfectionActions<<<cuda_doInfectionActionsKernel_blocks, cuda_doInfectionAtionsKernel_threads>>>(
		daily_action_type_ptr, daily_contact_victims_ptr, daily_contact_infectors_ptr,
		daily_actions,
		people_status_pandemic_ptr, people_status_seasonal_ptr,
		people_gens_pandemic_ptr, people_gens_seasonal_ptr,
		people_days_pandemic_ptr, people_days_seasonal_ptr,
		current_day + 1, rand_offset);

	if(TIMING_BATCH_MODE == 0)
	{
		int rand_counts_consumed = daily_actions / 2;
		rand_offset += rand_counts_consumed;
	}

	hipDeviceSynchronize();

	if(PROFILE_SIMULATION)
	{
		profiler.endFunction(current_day, daily_actions);
	}
}


void PandemicSim::setup_fetchVectorPtrs()
{
	if(PROFILE_SIMULATION)
		profiler.beginFunction(-1,"setup_fetchVectorPtrs");

	people_status_pandemic_ptr = thrust::raw_pointer_cast(people_status_pandemic.data());
	people_status_seasonal_ptr = thrust::raw_pointer_cast(people_status_seasonal.data());
	people_households_ptr = thrust::raw_pointer_cast(people_households.data());
	people_workplaces_ptr = thrust::raw_pointer_cast(people_workplaces.data());
	people_ages_ptr = thrust::raw_pointer_cast(people_ages.data());

	people_days_pandemic_ptr = thrust::raw_pointer_cast(people_days_pandemic.data());
	people_days_seasonal_ptr = thrust::raw_pointer_cast(people_days_seasonal.data());
	people_gens_pandemic_ptr = thrust::raw_pointer_cast(people_gens_pandemic.data());
	people_gens_seasonal_ptr = thrust::raw_pointer_cast(people_gens_seasonal.data());

	people_adults_indexes_ptr = thrust::raw_pointer_cast(people_adult_indexes.data());
	people_child_indexes_ptr = thrust::raw_pointer_cast(people_child_indexes.data());

	infected_indexes_ptr = thrust::raw_pointer_cast(infected_indexes.data());
	infected_daily_kval_sum_ptr = thrust::raw_pointer_cast(infected_daily_kval_sum.data());

	daily_contact_infectors_ptr = thrust::raw_pointer_cast(daily_contact_infectors.data());
	daily_contact_victims_ptr = thrust::raw_pointer_cast(daily_contact_victims.data());
	daily_contact_kval_types_ptr = thrust::raw_pointer_cast(daily_contact_kval_types.data());
	daily_action_type_ptr = thrust::raw_pointer_cast(daily_action_type.data());

	workplace_offsets_ptr = thrust::raw_pointer_cast(workplace_offsets.data());
	workplace_people_ptr = thrust::raw_pointer_cast(workplace_people.data());
	workplace_max_contacts_ptr = thrust::raw_pointer_cast(workplace_max_contacts.data());

	household_offsets_ptr = thrust::raw_pointer_cast(household_offsets.data());
	household_people_ptr = thrust::raw_pointer_cast(household_people.data());

	errand_people_table_ptr = thrust::raw_pointer_cast(errand_people_table.data());
	errand_people_weekendHours_ptr = thrust::raw_pointer_cast(errand_people_weekendHours.data());
	errand_people_destinations_ptr = thrust::raw_pointer_cast(errand_people_destinations.data());

	errand_infected_locations_ptr = thrust::raw_pointer_cast(errand_infected_locations.data());
	errand_infected_weekendHours_ptr = thrust::raw_pointer_cast(errand_infected_weekendHours.data());
	errand_infected_ContactsDesired_ptr = thrust::raw_pointer_cast(errand_infected_ContactsDesired.data());

	errand_locationOffsets_multiHour_ptr = thrust::raw_pointer_cast(errand_locationOffsets_multiHour.data());
	errand_hourOffsets_weekend_ptr = thrust::raw_pointer_cast(errand_hourOffsets_weekend.data());

	status_counts_dev_ptr = thrust::raw_pointer_cast(status_counts.data());

	if(SIM_VALIDATION)
	{
		debug_contactsToActions_float1_ptr = thrust::raw_pointer_cast(debug_contactsToActions_float1.data());
		debug_contactsToActions_float2_ptr = thrust::raw_pointer_cast(debug_contactsToActions_float2.data());
		debug_contactsToActions_float3_ptr = thrust::raw_pointer_cast(debug_contactsToActions_float3.data());
		debug_contactsToActions_float4_ptr = thrust::raw_pointer_cast(debug_contactsToActions_float4.data());
	}

	if(PROFILE_SIMULATION)
	{
		hipDeviceSynchronize();
		profiler.endFunction(-1,1);
	}
}

void PandemicSim::daily_clearActionsArray()
{
	int size_to_clear = is_weekend() ? MAX_CONTACTS_WEEKEND * infected_count : MAX_CONTACTS_WEEKDAY * infected_count;
	hipMemsetAsync(daily_action_type_ptr, 0, sizeof(int) * size_to_clear,stream_secondary);
}


void PandemicSim::daily_countInfectedStats()
{
	//get pointers
	int * pandemic_counts_ptr = status_counts_dev_ptr;
	int * seasonal_counts_ptr = pandemic_counts_ptr + 8;

	//memset to 0
	hipMemsetAsync(pandemic_counts_ptr, 0, sizeof(int) * 16,stream_secondary);

	size_t dynamic_smemsize = 0;
	///	kernel_countInfectedStatus<<<COUNTING_GRID_BLOCKS, COUNTING_GRID_THREADS,smemsize, stream_countInfectedStatus>>>(
	kernel_countInfectedStatus<<<COUNTING_GRID_BLOCKS, COUNTING_GRID_THREADS, dynamic_smemsize, stream_secondary>>>(
		people_status_pandemic_ptr, people_status_seasonal_ptr, 
		number_people, 
		pandemic_counts_ptr, seasonal_counts_ptr);

	hipMemcpyAsync(&status_counts_today, pandemic_counts_ptr,sizeof(int) * 16,hipMemcpyDeviceToHost,stream_secondary);
}

void PandemicSim::daily_writeInfectedStats()
{
	int pandemic_recovered = status_counts_today[0];
	int pandemic_susceptible = status_counts_today[1];

	int pandemic_symptomatic = status_counts_today[2] + status_counts_today[3] + status_counts_today[4];
	int pandemic_asymptomatic = status_counts_today[5] + status_counts_today[6] + status_counts_today[7];
	int pandemic_infected = pandemic_symptomatic + pandemic_asymptomatic;

	int seasonal_recovered = status_counts_today[8];
	int seasonal_susceptible = status_counts_today[9];

	int seasonal_symptomatic = status_counts_today[10] + status_counts_today[11] + status_counts_today[12];
	int seasonal_asymptomatic = status_counts_today[13] + status_counts_today[14] + status_counts_today[15];
	int seasonal_infected = seasonal_symptomatic + seasonal_asymptomatic;


	if(SIM_VALIDATION)
	{
		int pandemic_total = pandemic_susceptible + pandemic_infected + pandemic_recovered;
		int seasonal_total = seasonal_susceptible + seasonal_infected + seasonal_recovered;

		debug_assert("pandemic_total does not equal number_people in infected_status func", number_people, pandemic_total);
		debug_assert("seasonal_total does not equal number_people in infected_status func", number_people, seasonal_total);

		if(current_day == 0)
		{
			debug_assert("initial_infected_pandemic does not match the observed count on first day",INITIAL_INFECTED_PANDEMIC, pandemic_infected);
			debug_assert("initial_infected_seasonal does not match the observed count on first day",INITIAL_INFECTED_SEASONAL, seasonal_infected);
		}
	}

	fprintf(f_outputInfectedStats,
		"%d,%d,%d,%d,%d,%d,%d,%d,%d,%d,%d\n",
		current_day,
		pandemic_susceptible,
		pandemic_infected,
		pandemic_symptomatic,
		pandemic_asymptomatic,
		pandemic_recovered,
		seasonal_susceptible,
		seasonal_infected,
		seasonal_symptomatic,
		seasonal_asymptomatic,
		seasonal_recovered);
}

void PandemicSim::setup_calculateInfectionData()
{
	//adjust the asymptomatic profiles downwards
	for(int i = 3; i < NUM_PROFILES; i++)
		for(int j = 0; j < CULMINATION_PERIOD; j++)
			VIRAL_SHEDDING_PROFILES_HOST[i][j] *= asymp_factor;

	//calculate reproduction factors
	for(int i = 0; i < STRAIN_COUNT; i++)
	{
		INFECTIOUSNESS_FACTOR_HOST[i] = BASE_REPRODUCTION_HOST[i] / ((1.0f - asymp_factor) * PERCENT_SYMPTOMATIC_HOST);
	}
}
/*
struct memReadFunctor_int
{
	int * memPtr;
	__device__ int operator() (int offset)
	{
		return memPtr[offset];
	}
};*/

struct memReadFunctor_float
{
	__device__ float operator () (int offset1, int offset2)
	{
		return VIRAL_SHEDDING_PROFILES_DEVICE[offset1][offset2];
	}
};


void PandemicSim::debug_helper()
{
	int elements = NUM_PROFILES * CULMINATION_PERIOD;
	thrust::device_vector<float> d_profiles(elements);
//	thrust::copy_n(VIRAL_SHEDDING_PROFILES_DEVICE,elements,d_profiles.begin());

	int profile =2;
	thrust::counting_iterator<int> count_it(0);
	thrust::constant_iterator<int> const_it(profile);
	memReadFunctor_float memrdObj;
	thrust::transform(const_it, const_it+10, count_it, d_profiles.begin(), memrdObj);

	thrust::host_vector<float> h_profiles = d_profiles;

	FILE * fprofiledata = fopen("../profile_data.csv","w");
	fprintf(fprofiledata,"profile,day,val\n");
		for(int day = 0; day < CULMINATION_PERIOD; day++)
		{
			int idx = (profile * CULMINATION_PERIOD) + day;
			fprintf(fprofiledata,"%d,%d,%f\n",profile,day, h_profiles[idx]);
		}
	
	fclose(fprofiledata);
}


void PandemicSim::setup_loadSeed()
{
	int core_seed;

	FILE * fSeed = fopen("seed.txt","r");
	if(fSeed == NULL)
	{
		debug_print("failed to open seed file");
		perror("Error opening seed file");
		exit(1);
	}

	fscanf(fSeed, "%d", &core_seed);
	fclose(fSeed);

	srand(core_seed);
	for(int i = 0; i < SEED_LENGTH; i++)
	{
		int generated_seed = rand();
		SEED_HOST[i] = generated_seed;
	}
}

void PandemicSim::setup_loadFourSeeds()
{
	//load 4 seeds from file
	FILE * fSeed = fopen("seed.txt","r");
	if(fSeed == NULL)
	{
		debug_print("failed to open seed file");
		perror("Error opening seed file");
		exit(1);
	}

	for(int i = 0; i < SEED_LENGTH; i++)
	{
		fscanf(fSeed, "%d", &(SEED_HOST[i]));
	}
	fclose(fSeed);
}

void PandemicSim::setup_setCudaTopology()
{
	cuda_householdTypeAssignmentKernel_blocks = cuda_blocks;
	cuda_householdTypeAssignmentKernel_threads = cuda_threads;

	cuda_peopleGenerationKernel_blocks = cuda_blocks;
	cuda_peopleGenerationKernel_threads = cuda_threads;

	cuda_makeWeekdayContactsKernel_blocks = cuda_blocks;
	cuda_makeWeekdayContactsKernel_threads = cuda_threads;

	cuda_makeWeekendContactsKernel_blocks = cuda_blocks;
	cuda_makeWeekendContactsKernel_threads = cuda_threads;

	cuda_contactsToActionsKernel_blocks = cuda_blocks;
	cuda_contactsToActionsKernel_threads = cuda_threads;

	cuda_doInfectionActionsKernel_blocks = cuda_blocks;
	cuda_doInfectionAtionsKernel_threads = cuda_threads;
}
