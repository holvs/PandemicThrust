#include "hip/hip_runtime.h"
#include "resource_logging.h"

#define RESOURCE_LOG_FILENAME "output_resource_log.csv"

#ifdef _MSC_VER
#define VISUAL_STUDIO 1
#else
#define VISUAL_STUDIO 0
#endif

FILE * fMemory = NULL;
size_t initial_free_bytes;
size_t initial_total_bytes;

size_t max_memory_used = 0;

hipEvent_t event_start, event_stop;

float p_scale=0.f, l_scale=0.f;
const char * sim_type, * sim_device;
int core_seed;

bool file_exists(const char * filename)
{
	std::ifstream ifile(filename);
	return ifile;
}

void logging_pollMemUsage_doSetup(bool log_memory_usage, bool outputFilesInParentDir)
{
	//create events
	hipEventCreate(&event_start);
	hipEventCreate(&event_stop);

	hipMemGetInfo(&initial_free_bytes, &initial_total_bytes);

	if(log_memory_usage){
		//do initial setup for memory log
		if(outputFilesInParentDir)
			fMemory = fopen("../output_mem.csv","w");
		else
			fMemory = fopen("output_mem.csv","w");
		fprintf(fMemory, "day,freeBytes,bytesUsed,totalBytes,megabytesUsed\n");

		if(VISUAL_STUDIO)
			fprintf(fMemory,"INITIAL,%Iu,0,%Iu,0\n");
		else
			fprintf(fMemory, "INITIAL,%zu,0,%zu,0\n", initial_free_bytes, initial_total_bytes);
	}

	//record the start event
	hipEventRecord(event_start);
}

void logging_pollMemoryUsage_takeSample(int day)
{
	size_t current_free_bytes, current_total_bytes;

	hipMemGetInfo(&current_free_bytes, &current_total_bytes);

	size_t bytes_used = initial_free_bytes - current_free_bytes;
	size_t megabytes_used = bytes_used >> 20;

	if(bytes_used > max_memory_used)
		max_memory_used = bytes_used;
	
	if(VISUAL_STUDIO)
		fprintf(fMemory,"%d,%Iu,%Iu,%Iu,%Iu\n",
			day, current_free_bytes, bytes_used, current_total_bytes, megabytes_used);
	else
		fprintf(fMemory, "%d,%zu,%zu,%zu,%zu\n",
			day, current_free_bytes, bytes_used, current_total_bytes, megabytes_used);
}

void logging_pollMemoryUsage_done()
{
	hipEventRecord(event_stop);
	hipEventSynchronize(event_stop);

	//calculate elapsed time
	float elapsed_milliseconds;
	hipEventElapsedTime(&elapsed_milliseconds, event_start, event_stop);
	float elapsed_seconds = (float) elapsed_milliseconds / 1000;

	size_t current_free_bytes, current_total_bytes;
	hipMemGetInfo(&current_free_bytes, &current_total_bytes);

	size_t bytes_used = initial_free_bytes - current_free_bytes;
	if(bytes_used > max_memory_used)
		max_memory_used = bytes_used;

	size_t max_megabytes_used = max_memory_used >> 20;

	FILE * fResourceLog;
	bool log_exists = file_exists(RESOURCE_LOG_FILENAME);
	
	if(log_exists)
	{
		fResourceLog = fopen(RESOURCE_LOG_FILENAME, "a");
	}
	else
	{
		fResourceLog= fopen(RESOURCE_LOG_FILENAME, "w");
		fprintf(fResourceLog, "sim_type,sim_device,people_sim_scale,location_sim_scale,seed,runtime_milliseconds,runtime_seconds,bytes_used,megabytes_used\n");
	}

	if(VISUAL_STUDIO)
		fprintf(fResourceLog,"%s,%s,%f,%f,%d,%f,%f,%Iu,%Iu\n",
			sim_type,sim_device,p_scale,l_scale,core_seed,elapsed_milliseconds,elapsed_seconds,max_memory_used,max_megabytes_used);
	else
		fprintf(fResourceLog, "%s,%s,%f,%f,%d,%f,%f,%zu,%zu\n",
			sim_type,sim_device,p_scale,l_scale,core_seed,elapsed_milliseconds,elapsed_seconds,max_memory_used,max_megabytes_used);
	fclose(fResourceLog);


	if(fMemory != NULL)
		fclose(fMemory);

	hipEventDestroy(event_start);
	hipEventDestroy(event_stop);
}

void logging_setSimData(float people_scale, float loc_scale, const char * sim_type_string, const char * device, int seed)
{
	p_scale = people_scale;
	l_scale = loc_scale;
	sim_type = sim_type_string;
	sim_device = device;
	core_seed = seed;
}
