#include "hip/hip_runtime.h"
#include "resource_logging.h"

#ifdef _MSC_VER
#define VISUAL_STUDIO 1
#else
#define VISUAL_STUDIO 0
#endif

FILE * fMemory = NULL;
size_t initial_free_bytes;
size_t initial_total_bytes;

size_t max_memory_used = 0;

hipEvent_t event_start, event_stop;

float p_scale=0.f, l_scale=0.f;

void logging_pollMemUsage_doSetup(bool log_memory_usage, bool outputFilesInParentDir)
{
	//create events
	hipEventCreate(&event_start);
	hipEventCreate(&event_stop);

	hipMemGetInfo(&initial_free_bytes, &initial_total_bytes);

	if(log_memory_usage){
		//do initial setup for memory log
		if(outputFilesInParentDir)
			fMemory = fopen("../output_mem.csv","w");
		else
			fMemory = fopen("output_mem.csv","w");
		fprintf(fMemory, "day,freeBytes,bytesUsed,totalBytes,megabytesUsed\n");

		if(VISUAL_STUDIO)
			fprintf(fMemory,"INITIAL,%Iu,0,%Iu,0\n");
		else
			fprintf(fMemory, "INITIAL,%zu,0,%zu,0\n", initial_free_bytes, initial_total_bytes);
	}

	//record the start event
	hipEventRecord(event_start);
}

void logging_pollMemoryUsage_takeSample(int day)
{
	size_t current_free_bytes, current_total_bytes;

	hipMemGetInfo(&current_free_bytes, &current_total_bytes);

	size_t bytes_used = initial_free_bytes - current_free_bytes;
	size_t megabytes_used = bytes_used >> 20;

	if(bytes_used > max_memory_used)
		max_memory_used = bytes_used;
	
	if(VISUAL_STUDIO)
		fprintf(fMemory,"%d,%Iu,%Iu,%Iu,%Iu\n",
			day, current_free_bytes, bytes_used, current_total_bytes, megabytes_used);
	else
		fprintf(fMemory, "%d,%zu,%zu,%zu,%zu\n",
			day, current_free_bytes, bytes_used, current_total_bytes, megabytes_used);
}

void logging_pollMemoryUsage_done()
{
	hipEventRecord(event_stop);
	hipEventSynchronize(event_stop);

	//calculate elapsed time
	float elapsed_milliseconds;
	hipEventElapsedTime(&elapsed_milliseconds, event_start, event_stop);
	float elapsed_seconds = (float) elapsed_milliseconds / 1000;

	size_t current_free_bytes, current_total_bytes;
	hipMemGetInfo(&current_free_bytes, &current_total_bytes);

	size_t bytes_used = initial_free_bytes - current_free_bytes;
	if(bytes_used > max_memory_used)
		max_memory_used = bytes_used;

	size_t max_megabytes_used = max_memory_used >> 20;


	FILE * fResourceLog = fopen("output_resource_log.csv", "w");
	fprintf(fResourceLog, "people_sim_scale,location_sim_scale,runtime_milliseconds,runtime_seconds,bytes_used,megabytes_used\n");

	if(VISUAL_STUDIO)
		fprintf(fResourceLog,"%f,%f,%f,%f,%Iu,%Iu\n",
			p_scale,l_scale,elapsed_milliseconds,elapsed_seconds,max_memory_used,max_megabytes_used);
	else
		fprintf(fResourceLog, "%f,%f,%f,%f,%zu,%zu\n",
			p_scale,l_scale,elapsed_milliseconds,elapsed_seconds,max_memory_used,max_megabytes_used);
	fclose(fResourceLog);


	if(fMemory != NULL)
		fclose(fMemory);

	hipEventDestroy(event_start);
	hipEventDestroy(event_stop);
}

void logging_setSimScale(float people_scale, float loc_scale)
{
	p_scale = people_scale;
	l_scale = loc_scale;

}
