#include "stdafx.h"

#include "PandemicSim.h"

//#include "indirect.h"
#include "resource_logging.h"

#if CUDA_PROFILER_ENABLE == 1
#include "hip/hip_runtime_api.h"
#endif

#ifdef _MSC_VER
#include <Windows.h>

void delay_start()
{
	int milliseconds = 1000 * MAIN_DELAY_SECONDS;
	Sleep(milliseconds);
}
#else
#include <unistd.h>

void delay_start()
{
	sleep(MAIN_DELAY_SECONDS);
}
#endif


int main()
{
	if(MAIN_DELAY_SECONDS > 0)
		delay_start();

	if(CUDA_PROFILER_ENABLE)
		hipProfilerStart();

	logging_pollMemUsage_doSetup(POLL_MEMORY_USAGE, OUTPUT_FILES_IN_PARENTDIR);

	PandemicSim sim;
	sim.setupSim();
	sim.runToCompletion();

	logging_pollMemoryUsage_done();

	if(CUDA_PROFILER_ENABLE)
		hipProfilerStop();

//	hipDeviceReset();

	return 0;
}
